#include "hip/hip_runtime.h"
//----------------------------------------------------------------------------------
//
// FLUIDS v.3 - SPH Fluid Simulator for CPU and GPU
// Copyright (C) 2012-2013. Rama Hoetzlein, http://fluids3.com
//
// BSD 3-clause:
// Redistribution and use in source and binary forms, with or without modification, 
// are permitted provided that the following conditions are met:
// 1. Redistributions of source code must retain the above copyright notice, this 
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this 
//    list of conditions and the following disclaimer in the documentation and/or 
//    other materials provided with the distribution.
// 3. Neither the name of the copyright holder nor the names of its contributors may 
//    be used to endorse or promote products derived from this software without specific 
//   prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY 
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES 
// OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT 
// OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) 
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
// TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
// EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//----------------------------------------------------------------------------------

#define CUDA_KERNEL
#include "fluid_system_cuda.cuh"

#include "cutil_math.h"			// cutil32.lib
#include <string.h>
#include <assert.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__constant__ FParams		fparam;			// CPU Fluid params
__constant__ FBufs			fbuf;			// GPU Particle buffers (unsorted). An FBufs struct holds an array of pointers. 
__constant__ FBufs			ftemp;			// GPU Particle buffers (sorted)
__constant__ FGenome		fgenome;		// GPU Genome for particle automata behaviour. Also holds morphogen diffusability.
__constant__ uint			gridActive;

#define SCAN_BLOCKSIZE		512

extern "C" __global__ void insertParticles ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	//-- debugging (pointers should match CUdeviceptrs on host side)
	// printf ( " pos: %012llx, gcell: %012llx, gndx: %012llx, gridcnt: %012llx\n", fbuf.bufC(FPOS), fbuf.bufC(FGCELL), fbuf.bufC(FGNDX), fbuf.bufC(FGRIDCNT) );

	register float3 gridMin =	fparam.gridMin;      // "register" is a compiler 'hint', to keep this variable in thread register
	register float3 gridDelta = fparam.gridDelta;    //  even if other variable have to be moved to slower 'local' memory  
	register int3 gridRes =		fparam.gridRes;      //  in the streaming multiprocessor's cache.
	register int3 gridScan =	fparam.gridScanMax;

	register int		gs;
	register float3		gcf;
	register int3		gc;	

	gcf = (fbuf.bufF3(FPOS)[i] - gridMin) * gridDelta; 
	gc = make_int3( int(gcf.x), int(gcf.y), int(gcf.z) );
	gs = (gc.y * gridRes.z + gc.z)*gridRes.x + gc.x;

	if ( gc.x >= 1 && gc.x <= gridScan.x && gc.y >= 1 && gc.y <= gridScan.y && gc.z >= 1 && gc.z <= gridScan.z ) {
		fbuf.bufI(FGCELL)[i] = gs;											// Grid cell insert.
		fbuf.bufI(FGNDX)[i] = atomicAdd ( &fbuf.bufI(FGRIDCNT)[ gs ], 1 );		// Grid counts.

		//gcf = (-make_float3(poff,poff,poff) + fbuf.bufF3(FPOS)[i] - gridMin) * gridDelta;
		//gc = make_int3( int(gcf.x), int(gcf.y), int(gcf.z) );
		//gs = ( gc.y * gridRes.z + gc.z)*gridRes.x + gc.x;
	} else {
		fbuf.bufI(FGCELL)[i] = GRID_UNDEF;		
	}
}

// Counting Sort - Full (deep copy)
extern "C" __global__ void countingSortFull ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;		// particle index				
	if ( i >= pnum ) return;

	// Copy particle from original, unsorted buffer (msortbuf),
	// into sorted memory location on device (mpos/mvel)
	uint icell = ftemp.bufI(FGCELL) [ i ];	

	if ( icell != GRID_UNDEF ) {	  
		// Determine the sort_ndx, location of the particle after sort		
		uint indx =  ftemp.bufI(FGNDX)  [ i ];		
	    int sort_ndx = fbuf.bufI(FGRIDOFF) [ icell ] + indx ;	// global_ndx = grid_cell_offet + particle_offset	
		//printf ( "%d: cell: %d, off: %d, ndx: %d\n", i, icell, fbuf.bufI(FGRIDOFF)[icell], indx );
		float3 zero; zero.x=0;zero.y=0;zero.z=0;
		// Transfer data to sort location
		fbuf.bufI (FGRID) [ sort_ndx ] =	sort_ndx;			// full sort, grid indexing becomes identity		
		fbuf.bufF3(FPOS) [sort_ndx] =		ftemp.bufF3(FPOS) [i];
		fbuf.bufF3(FVEL) [sort_ndx] =		ftemp.bufF3(FVEL) [i];
		fbuf.bufF3(FVEVAL)[sort_ndx] =		ftemp.bufF3(FVEVAL) [i];
		fbuf.bufF3(FFORCE)[sort_ndx] =	zero;// old:	ftemp.bufF3(FFORCE) [i];  
                                            // fbuf.bufF3(FFORCE)[ i ] += force; in contributeForce() requires value setting to 0
		fbuf.bufF (FPRESS)[sort_ndx] =		ftemp.bufF(FPRESS) [i];
		fbuf.bufF (FDENSITY)[sort_ndx] =	ftemp.bufF(FDENSITY) [i];
		fbuf.bufI (FCLR) [sort_ndx] =		ftemp.bufI(FCLR) [i];
		fbuf.bufI (FGCELL) [sort_ndx] =		icell;
		fbuf.bufI (FGNDX) [sort_ndx] =		indx;		
        
        // add extra data for morphogenesis
        for (int a=0;a<BONDS_PER_PARTICLE*2;a++){
            fbuf.bufI (FELASTIDX) [sort_ndx*BONDS_PER_PARTICLE*2 + a] =	ftemp.bufI(FELASTIDX) [i*BONDS_PER_PARTICLE*2 + a]; //sort_ndx= grid_cell_offet + particle_offset   , i=particle index
        }
        fbuf.bufI (FPARTICLE_ID) [sort_ndx] =	ftemp.bufI(FPARTICLE_ID) [i];
        fbuf.bufI (FMASS_RADIUS) [sort_ndx] =	ftemp.bufI(FMASS_RADIUS) [i];
        
        fbuf.bufI (FNERVEIDX) [sort_ndx] =	ftemp.bufI(FNERVEIDX) [i];
        fbuf.bufI (FCONC) [sort_ndx] =		ftemp.bufI(FCONC) [i];
        fbuf.bufI (FEPIGEN) [sort_ndx] =	ftemp.bufI(FEPIGEN) [i];  
	}
} 

extern "C" __device__ float contributePressure ( int i, float3 p, int cell )  
// pressure due to particles in 'cell'. NB for each particle there are 27 cells in which interacting particles might be.
{			
	if ( fbuf.bufI(FGRIDCNT)[cell] == 0 ) return 0.0;                                  // If the cell is empty, skip it.

	float3 dist;
	float dsq, c, sum = 0.0;
	register float d2 = fparam.psimscale * fparam.psimscale;
	register float r2 = fparam.r2 / d2;
	
	int clast = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];     // off set of this cell in the list of particles,  PLUS  the count of particles in this cell.

	for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++ ) {   // For particles in this cell.
		int pndx = fbuf.bufI(FGRID) [cndx];                                       // index of this particle
		dist = p - fbuf.bufF3(FPOS) [pndx];                                       // float3 distance between this particle, and the particle for which the loop has been called.
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);                    // scalar distance squared
		if ( dsq < r2 && dsq > 0.0) {                                             // IF in-range && not the same particle. 
			c = (r2 - dsq)*d2;                                                           //(NB this means all unused particles can be stored at one point)
			sum += c * c * c;				
		} 
	}
	
	return sum;                                                     // NB a scalar value for pressure contribution, at the current particle, due to particles in this cell.
}
			
extern "C" __global__ void computePressure ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                // particle index
	if ( i >= pnum ) return;

	// Get search cell
	int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	uint gc = fbuf.bufI(FGCELL) [i];                                       // get grid cell of the current particle.
	if ( gc == GRID_UNDEF ) return;                                        // IF particle not in the simulation
	gc -= nadj;

	// Sum Pressures
	float3 pos = fbuf.bufF3(FPOS) [i];
	float sum = 0.0;
	for (int c=0; c < fparam.gridAdjCnt; c++) {                                    
		sum += contributePressure ( i, pos, gc + fparam.gridAdj[c] );
	}
	__syncthreads();
		
	// Compute Density & Pressure
	sum = sum * fparam.pmass * fparam.poly6kern;
	if ( sum == 0.0 ) sum = 1.0;
	fbuf.bufF(FPRESS)  [ i ] = ( sum - fparam.prest_dens ) * fparam.pintstiff;
	fbuf.bufF(FDENSITY)[ i ] = 1.0f / sum;
}

extern "C" __device__ float3 contributeForce ( int i, float3 ipos, float3 iveleval, float ipress, float idens, int cell, uint particleID,
                                               uint elastIdx[BONDS_PER_PARTICLE * 2], uint bond[BONDS_PER_PARTICLE][2], 
                                               uchar intact[BONDS_PER_PARTICLE]
                                               /*uint particleID[BONDS_PER_PARTICLE], uint  BondType[BONDS_PER_PARTICLE]*/)
{			
	if ( fbuf.bufI(FGRIDCNT)[cell] == 0 ) return make_float3(0,0,0);               // If the cell is empty, skip it.

	float dsq, c, pterm;	
	float3 dist, eterm, force = make_float3(0,0,0);
	int j;
    

	int clast = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];

	for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++ ) {          // For particles in this cell.
		j = fbuf.bufI(FGRID)[ cndx ];				                                     
		dist = ( ipos - fbuf.bufF3(FPOS)[ j ] );                                  // dist in cm (Rama's comment)
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);                    // scalar distance squared
		if ( dsq < fparam.rd2 && dsq > 0) {                                       // IF in-range && not the same particle
			dsq = sqrt(dsq * fparam.d2);                                          // sqrt(dist^2 * sim_scale^2))
			c = ( fparam.psmoothradius - dsq ); 
			pterm = fparam.psimscale * -0.5f * c * fparam.spikykern * ( ipress + fbuf.bufF(FPRESS)[ j ] ) / dsq;			
            
            //  elastic force due to bonds
            uint rest_len = REST_LENGTH * fparam.pradius;
            for(int a=1; a < BONDS_PER_PARTICLE ; a++){ // for this other particle, check list of bonds for particle of this thread
                if(particleID == elastIdx[a]){      // bond present
                    float abs_dist = sqrt(dsq);
                    if( abs_dist < bond[a][1] ){    // bond intact
                        eterm = dist * ( (rest_len - abs_dist) * bond[a][0] / abs_dist ); // exerts force
                        intact[a]+=1;                // mark bond intact n.b. this catches double counting as well as out of range bonds.
                    }
                    break;                          // bond processed => break out of for loop 
                }
            }            
            
			force += (eterm + pterm * dist + fparam.vterm * ( fbuf.bufF3(FVEVAL)[ j ] - iveleval )) * c * idens * (fbuf.bufF(FDENSITY)[ j ] ); 
            /*
            // force due to pressure gradient PLUS viscosity.
            //   pterm(-ve scalar) * distance(float3) => repulsion
            //   viscosity * (vel(float3)[j] - vel(float3)[i]) * c * density[i] * desity[j]  => drag
               
                //uint b = fbuf.bufI(FELASTIDX)[i+a]   ;//[i][a];  
//                 if(fbuf.bufI(FELASTIDX)[j] == b){
//                     float abs_dist = sqrt(dsq);
//                     eterm = dist * ( (rest_len - abs_dist) * YoungsModulus / abs_dist );  // elastic_force  // NB need to split abs_force by xyz of 
//                     fbuf.bufI(FELASTIDX)[j]  |= TWO_POW_31;                       // bitwise inclusive OR, mark bond as intact, NB this is a uint,
//                 }
            */
            /*
            // fbuf.bufI(FELASTIDX)[0]   is 24bits particle ID => 2^24 = 16,777,216 particles, and 8 bits bond type => 2^8 = 256 bond types.
            //                              look up bond type in texture memory => modulus. Use fixed bond length.
            // fbuf.bufI(FELASTIDX)[0]   is 24 bits particle ID of bonded particle, and 8 bits of other data. Includes 'broken bond' flag.
            
            
            //  for(int a=1; a < BONDS_PER_PARTICLE ; a++)
            //  { 
            //      unit b = m_Fluid.gpu(FELASTIDX)[i][a];     
            //      if(m_Fluid.gpu(FELASTIDX)[j][0] == b)
            //      {
            //          contribute_elastic_force()
            //          m_Fluid.gpu(FELASTIDX)[i][a]  *= (-1);     //mark bond as intact  ? is this a uint !?
            //      }
            //
            //  if(  m_Fluid.gpu(FELASTIDX)[0] == 
            */
        }
	}
	return force;
}


extern "C" __global__ void computeForce ( int pnum)
{			
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	          // particle index				
	if ( i >= pnum ) return;

	// Get search cell	
	uint gc = fbuf.bufI(FGCELL)[ i ];
	if ( gc == GRID_UNDEF ) return;						              // particle out-of-range
	gc -= (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;

	// Sum Pressures	
	register float3 force;                                            // request to compiler to store in a register for speed.
	force = make_float3(0,0,0);		
    
    uint elastIdx[BONDS_PER_PARTICLE];
    uint bond[BONDS_PER_PARTICLE][2];
    for (int a=0;a<BONDS_PER_PARTICLE;a++){                             // copy FELASTIDX to thread memory for particle i.
        elastIdx[a] = fbuf.bufI(FELASTIDX)[i*BONDS_PER_PARTICLE*2 + a*2];// particle IDs   i*BONDS_PER_PARTICLE*2 + a  //  [i][a*2]
        uint temp = fbuf.bufI(FELASTIDX)[i*BONDS_PER_PARTICLE*2 + a*2 +1]; //[i][a*2]
        bond[a][0] = temp & TWO_POW_24_MINUS_1;     // modulus          // '&' bitwise AND is bit masking.  
        bond[a][1] = (temp >> 24);                  // elastic limit    // '>>' Bit shift can deliver high bits to bottom
    }
    uchar intact[BONDS_PER_PARTICLE];
    for (int a=0;a<BONDS_PER_PARTICLE;a++){ intact[a]=0;}               // ledger for chcking bonds are intact.
    
    uint particleID = fbuf.bufI(FPARTICLE_ID)[i];   // ID of this particle
    
/*    
//    // get ElastIdx[4], particle IDs and young's modulus. // NB need yield strain and other info too.
//     uint BondType[BONDS_PER_PARTICLE];                                               // BondType[0] can store flag bit for broken bonds. 
//     uint particleID[BONDS_PER_PARTICLE];
// 
//     for (int i=0;i<BONDS_PER_PARTICLE;i++){
//         particleID[i] = fbuf.bufI(FELASTIDX)[i]  & TWO_POW_24 ;     // '&' bitwise AND is bit masking.  '>>' Bit shift can deliver high bits to bottom.
//         BondType[i] = fbuf.bufI(FELASTIDX)[i];
//         BondType[i] >> 24;                               // Young's modulus = pow(2,(BondType/6.4)) => 0-256 log spread from gel to diamond.
//     }
*/    
    
	for (int c=0; c < fparam.gridAdjCnt; c++) {
		force += contributeForce ( i, fbuf.bufF3(FPOS)[ i ], fbuf.bufF3(FVEVAL)[ i ], fbuf.bufF(FPRESS)[ i ], fbuf.bufF(FDENSITY)[ i ], gc + fparam.gridAdj[c], particleID, elastIdx, bond , intact); /*particleID, BondType*/
	}
	fbuf.bufF3(FFORCE)[ i ] += force;  //  += req for elastic bonds. NB need to reset to zero in  CountingSortFull(..)
	
	for (int a=0;a<BONDS_PER_PARTICLE;a++){                            // remove broken bonds
        if(intact[a]!=1){fbuf.bufII(FELASTIDX)[i*BONDS_PER_PARTICLE*2 + a*2 +1]=0;} // nb particle_ID = 0 must be "NO_PARTICLE"  // [i][a*2]
    }
}


extern "C" __global__ void randomInit ( int seed, int numPnts )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= numPnts ) return;

	// Initialize particle random generator	
	hiprandState_t* st = (hiprandState_t*) (fbuf.bufC(FSTATE) + i*sizeof(hiprandState_t));
	hiprand_init ( seed + i, 0, 0, st );		
}

#define CURANDMAX		2147483647

extern "C" __global__ void emitParticles ( float frame, int emit, int numPnts )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= emit ) return;

	hiprandState_t* st = (hiprandState_t*) (fbuf.bufC(FSTATE) + i*sizeof(hiprandState_t));
	uint v = hiprand( st);
	uint j = v & (numPnts-1);
	float3 bmin = make_float3(-170,10,-20);
	float3 bmax = make_float3(-190,60, 20);

	float3 pos = make_float3(0,0,0);	
	pos.x = float( v & 0xFF ) / 256.0;
	pos.y = float((v>>8) & 0xFF ) / 256.0;
	pos.z = float((v>>16) & 0xFF ) / 256.0;
	pos = bmin + pos*(bmax-bmin);	
	
	fbuf.bufF3(FPOS)[j] = pos;
	fbuf.bufF3(FVEVAL)[j] = make_float3(0,0,0);
	fbuf.bufF3(FVEL)[j] = make_float3(5,-2,0);
	fbuf.bufF3(FFORCE)[j] = make_float3(0,0,0);	
	
}

__device__ uint getGridCell ( float3 pos, uint3& gc )
{	
	gc.x = (int)( (pos.x - fparam.gridMin.x) * fparam.gridDelta.x);			// Cell in which particle is located
	gc.y = (int)( (pos.y - fparam.gridMin.y) * fparam.gridDelta.y);
	gc.z = (int)( (pos.z - fparam.gridMin.z) * fparam.gridDelta.z);		
	return (int) ( (gc.y*fparam.gridRes.z + gc.z)*fparam.gridRes.x + gc.x);	
}

extern "C" __global__ void sampleParticles ( float* brick, uint3 res, float3 bmin, float3 bmax, int numPnts, float scalar )
{
	float3 dist;
	float dsq;
	int j, cell;	
	register float r2 = fparam.r2;
	register float h2 = 2.0*r2 / 8.0;		// 8.0=smoothing. higher values are sharper

	uint3 i = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;
	if ( i.x >= res.x || i.y >= res.y || i.z >= res.z ) return;
	
	float3 p = bmin + make_float3(float(i.x)/res.x, float(i.y)/res.y, float(i.z)/res.z) * (bmax-bmin);
	//float3 v = make_float3(0,0,0);
	float v = 0.0;

	// Get search cell
	int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	uint3 gc;
	uint gs = getGridCell ( p, gc );
	if ( gc.x < 1 || gc.x > fparam.gridRes.x-fparam.gridSrch || gc.y < 1 || gc.y > fparam.gridRes.y-fparam.gridSrch || gc.z < 1 || gc.z > fparam.gridRes.z-fparam.gridSrch ) {
		brick[ (i.y*int(res.z) + i.z)*int(res.x) + i.x ] = 0.0;
		return;
	}

	gs -= nadj;	

	for (int c=0; c < fparam.gridAdjCnt; c++) {
		cell = gs + fparam.gridAdj[c];		
		if ( fbuf.bufI(FGRIDCNT)[cell] != 0 ) {				
			for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell]; cndx++ ) {
				j = fbuf.bufI(FGRID)[cndx];
				dist = p - fbuf.bufF3(FPOS)[ j ];
				dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
				if ( dsq < fparam.rd2 && dsq > 0 ) {
					dsq = sqrt(dsq * fparam.d2);					
					//v += fbuf.mvel[j] * (fparam.gausskern * exp ( -(dsq*dsq)/h2 ) / fbuf.mdensity[ j ]);
					v += fparam.gausskern * exp ( -(dsq*dsq)/h2 );
				}
			}
		}
	}
	__syncthreads();

	brick[ (i.z*int(res.y) + i.y)*int(res.x) + i.x ] = v * scalar;
	//brick[ (i.z*int(res.y) + i.y)*int(res.x) + i.x ] = length(v) * scalar;
}

extern "C" __global__ void computeQuery ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	// Get search cell
	int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	uint gc = fbuf.bufI(FGCELL) [i];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= nadj;

	// Sum Pressures
	float sum = 0.0;
	for (int c=0; c < fparam.gridAdjCnt; c++) {
		sum += 1.0;
	}
	__syncthreads();
	
}

		
extern "C" __global__ void advanceParticles ( float time, float dt, float ss, int numPnts )
{		
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= numPnts ) return;
	
	if ( fbuf.bufI(FGCELL)[i] == GRID_UNDEF ) {
		fbuf.bufF3(FPOS)[i] = make_float3(-1000,-1000,-1000);
		fbuf.bufF3(FVEL)[i] = make_float3(0,0,0);
		return;
	}
			
	// Get particle vars
	register float3 accel, norm;
	register float diff, adj, speed;
	register float3 pos = fbuf.bufF3(FPOS)[i];
	register float3 veval = fbuf.bufF3(FVEVAL)[i];

	// Leapfrog integration						
	accel = fbuf.bufF3(FFORCE)[i];
	accel *= fparam.pmass;	
		
	// Boundaries
	// Y-axis
	
	diff = fparam.pradius - (pos.y - (fparam.pboundmin.y + (pos.x-fparam.pboundmin.x)*fparam.pground_slope )) * ss;
	if ( diff > EPSILON ) {
		norm = make_float3( -fparam.pground_slope, 1.0 - fparam.pground_slope, 0);
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	diff = fparam.pradius - ( fparam.pboundmax.y - pos.y )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3(0, -1, 0);
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	// X-axis
	diff = fparam.pradius - (pos.x - (fparam.pboundmin.x + (sin(time*fparam.pforce_freq)+1)*0.5 * fparam.pforce_min))*ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 1, 0, 0);
		adj = (fparam.pforce_min+1) * fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	diff = fparam.pradius - ( (fparam.pboundmax.x - (sin(time*fparam.pforce_freq)+1)*0.5*fparam.pforce_max) - pos.x)*ss;
	if ( diff > EPSILON ) {
		norm = make_float3(-1, 0, 0);
		adj = (fparam.pforce_max+1) * fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	// Z-axis
	diff = fparam.pradius - (pos.z - fparam.pboundmin.z ) * ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 0, 0, 1 );
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	diff = fparam.pradius - ( fparam.pboundmax.z - pos.z )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 0, 0, -1 );
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
		
	// Gravity
	accel += fparam.pgravity;

	// Accel Limit
	speed = accel.x*accel.x + accel.y*accel.y + accel.z*accel.z;
	if ( speed > fparam.AL2 ) {
		accel *= fparam.AL / sqrt(speed);
	}

	// Velocity Limit
	float3 vel = fbuf.bufF3(FVEL)[i];
	speed = vel.x*vel.x + vel.y*vel.y + vel.z*vel.z;
	if ( speed > fparam.VL2 ) {
		speed = fparam.VL2;
		vel *= fparam.VL / sqrt(speed);
	}

	// Ocean colors
	/*uint clr = fbuf.bufI(FCLR)[i];
	if ( speed > fparam.VL2*0.2) {
		adj = fparam.VL2*0.2;		
		clr += ((  clr & 0xFF) < 0xFD ) ? +0x00000002 : 0;		// decrement R by one
		clr += (( (clr>>8) & 0xFF) < 0xFD ) ? +0x00000200 : 0;	// decrement G by one
		clr += (( (clr>>16) & 0xFF) < 0xFD ) ? +0x00020000 : 0;	// decrement G by one
		fbuf.bufI(FCLR)[i] = clr;
	}
	if ( speed < 0.03 ) {		
		int v = int(speed/.01)+1;
		clr += ((  clr & 0xFF) > 0x80 ) ? -0x00000001 * v : 0;		// decrement R by one
		clr += (( (clr>>8) & 0xFF) > 0x80 ) ? -0x00000100 * v : 0;	// decrement G by one
		fbuf.bufI(FCLR)[i] = clr;
	}*/
	
	//-- surface particle density 
	//fbuf.mclr[i] = fbuf.mclr[i] & 0x00FFFFFF;
	//if ( fbuf.mdensity[i] > 0.0014 ) fbuf.mclr[i] += 0xAA000000;

	// Leap-frog Integration
	float3 vnext = accel*dt + vel;					// v(t+1/2) = v(t-1/2) + a(t) dt		
	fbuf.bufF3(FVEVAL)[i] = (vel + vnext) * 0.5;	// v(t+1) = [v(t-1/2) + v(t+1/2)] * 0.5			
	fbuf.bufF3(FVEL)[i] = vnext;
	fbuf.bufF3(FPOS)[i] += vnext * (dt/ss);			// p(t+1) = p(t) + v(t+1/2) dt		
}


extern "C" __global__ void prefixFixup(uint *input, uint *aux, int len)     // merge *aux into *input  
{
	unsigned int t = threadIdx.x;
	unsigned int start = t + 2 * blockIdx.x * SCAN_BLOCKSIZE;
	if (start < len)					input[start] += aux[blockIdx.x];      
	if (start + SCAN_BLOCKSIZE < len)   input[start + SCAN_BLOCKSIZE] += aux[blockIdx.x];
}

extern "C" __global__ void prefixSum(uint* input, uint* output, uint* aux, int len, int zeroff) // sum *input, write to *output
{
	__shared__ uint scan_array[SCAN_BLOCKSIZE << 1];
	unsigned int t1 = threadIdx.x + 2 * blockIdx.x * SCAN_BLOCKSIZE;
	unsigned int t2 = t1 + SCAN_BLOCKSIZE;

	// Pre-load into shared memory
	scan_array[threadIdx.x] = (t1<len) ? input[t1] : 0.0f;
	scan_array[threadIdx.x + SCAN_BLOCKSIZE] = (t2<len) ? input[t2] : 0.0f;
	__syncthreads();

	// Reduction
	int stride;
	for (stride = 1; stride <= SCAN_BLOCKSIZE; stride <<= 1) {
		int index = (threadIdx.x + 1) * stride * 2 - 1;
		if (index < 2 * SCAN_BLOCKSIZE)
			scan_array[index] += scan_array[index - stride];
		__syncthreads();
	}

	// Post reduction
	for (stride = SCAN_BLOCKSIZE >> 1; stride > 0; stride >>= 1) {
		int index = (threadIdx.x + 1) * stride * 2 - 1;
		if (index + stride < 2 * SCAN_BLOCKSIZE)
			scan_array[index + stride] += scan_array[index];
		__syncthreads();
	}
	__syncthreads();

	// Output values & aux
	if (t1 + zeroff < len)	output[t1 + zeroff] = scan_array[threadIdx.x];
	if (t2 + zeroff < len)	output[t2 + zeroff] = (threadIdx.x == SCAN_BLOCKSIZE - 1 && zeroff) ? 0 : scan_array[threadIdx.x + SCAN_BLOCKSIZE];
	if (threadIdx.x == 0) {
		if (zeroff) output[0] = 0;
		if (aux) aux[blockIdx.x] = scan_array[2 * SCAN_BLOCKSIZE - 1];
	}
}

