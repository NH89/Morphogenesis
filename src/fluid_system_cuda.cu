#include "hip/hip_runtime.h"
//----------------------------------------------------------------------------------
//
// FLUIDS v.3 - SPH Fluid Simulator for CPU and GPU
// Copyright (C) 2012-2013. Rama Hoetzlein, http://fluids3.com
//
// BSD 3-clause:
// Redistribution and use in source and binary forms, with or without modification, 
// are permitted provided that the following conditions are met:
// 1. Redistributions of source code must retain the above copyright notice, this 
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this 
//    list of conditions and the following disclaimer in the documentation and/or 
//    other materials provided with the distribution.
// 3. Neither the name of the copyright holder nor the names of its contributors may 
//    be used to endorse or promote products derived from this software without specific 
//   prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY 
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES 
// OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT 
// OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) 
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
// TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
// EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//----------------------------------------------------------------------------------

#define CUDA_KERNEL
#include "fluid_system_cuda.cuh"
#include <cfloat>
#include <cstdint>
#include "cutil_math.h"			// cutil32.lib
#include <string.h>
#include <assert.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__constant__ FParams		fparam;			// CPU Fluid params
__constant__ FBufs			fbuf;			// GPU Particle buffers (unsorted). An FBufs struct holds an array of pointers. 
__constant__ FBufs			ftemp;			// GPU Particle buffers (sorted)
__constant__ FGenome		fgenome;		// GPU Genome for particle automata behaviour. Also holds morphogen diffusability.
//__constant__ FBondParams    fbondparams;    // GPU copy of remodelling parameters. 
__constant__ uint			gridActive;

#define SCAN_BLOCKSIZE		512
//#define FLT_MIN  0.000000001              // set here as 2^(-30)
//#define UINT_MAX 65535

//if(fparam.debug>2) => device printf

extern "C" __global__ void insertParticles ( int pnum )                                         // decides which bin each particle belongs in.
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index
	if ( i >= pnum ) return;

	//-- debugging (pointers should match CUdeviceptrs on host side)
	// printf ( " pos: %012llx, gcell: %012llx, gndx: %012llx, gridcnt: %012llx\n", fbuf.bufC(FPOS), fbuf.bufC(FGCELL), fbuf.bufC(FGNDX), fbuf.bufC(FGRIDCNT) );
  //  if (fparam.debug>2 && i==0)printf("\ninsertParticles(): pnum=%u\n",pnum);

	register float3 gridMin =	fparam.gridMin;                                  // "register" is a compiler 'hint', to keep this variable in thread register
	register float3 gridDelta = fparam.gridDelta;                                //  even if other variable have to be moved to slower 'local' memory  
	register int3 gridRes =		fparam.gridRes;                                  //  in the streaming multiprocessor's cache.
	register int3 gridScan =	fparam.gridScanMax;
    register int gridTot =      fparam.gridTotal;

	register int		gs;
	register float3		gcf;
	register int3		gc;	

	gcf = (fbuf.bufF3(FPOS)[i] - gridMin) * gridDelta;                           // finds bin as a float3
	gc = make_int3( int(gcf.x), int(gcf.y), int(gcf.z) );                        // crops to an int3
	gs = (gc.y * gridRes.z + gc.z)*gridRes.x + gc.x;                             // linearizes to an int for a 1D array of bins
	
if(fparam.debug>2 && i==pnum-1) printf("\ninsertParticles()1: gridTot=%i,  i=%u: gc.x=%i, gc.y=%i, gc.z=%i, gs=%i \t gridScan.x=%i, gridScan.y=%i, gridScan.z=%i, gridTot=%u,\t gridDelta=(%f,%f,%f) gridMin=(%f,%f,%f) gridRes=(%i,%i,%i)", 
    gridTot, i, gc.x, gc.y, gc.z, gs,  gridScan.x, gridScan.y, gridScan.z, gridTot, gridDelta.x, gridDelta.y, gridDelta.z,  gridMin.x, gridMin.y, gridMin.z, gridRes.x, gridRes.y, gridRes.z );

	if ( gc.x >= 1 && gc.x <= gridScan.x && gc.y >= 1 && gc.y <= gridScan.y && gc.z >= 1 && gc.z <= gridScan.z ) {
		fbuf.bufI(FGCELL)[i] = gs;											     // Grid cell insert.
		fbuf.bufI(FGNDX)[i] = atomicAdd ( &fbuf.bufI(FGRIDCNT)[ gs ], 1 );       // Grid counts.         //  ## counts particles in this bin.
                                                                                                         //  ## add counters for dense lists. ##############
        // for each gene, if active, then atomicAdd bin count for gene
        for(int gene=0; gene<NUM_GENES; gene++){ // NB data ordered FEPIGEN[gene][particle] AND +ve int values -> active genes.
            //if(fparam.debug>2 && i==0)printf("\n");
            if (fbuf.bufI(FEPIGEN) [i + gene*fparam.maxPoints] >0 ){  // "if((int)fbuf.bufI(FEPIGEN)" may clash with INT_MAX
                atomicAdd ( &fbuf.bufI(FGRIDCNT_ACTIVE_GENES)[gene*gridTot  + gs ], 1 );
                //if(fparam.debug>2 && (gene==6||gene==9) /*i<10*/) printf("\ninsertParticles()2: i=,%u, gene=,%u, gs=,%u, fbuf.bufI(FGRIDCNT_ACTIVE_GENES)[ gene*gridTot  + gs ]=,%u",
                //    i, gene, gs, fbuf.bufI(FGRIDCNT_ACTIVE_GENES)[ gene*gridTot  + gs ]);
            }
            // could use a small array of uints to store gene activity as bits. This would reduce the reads, but require bitshift and mask to read. 
            //if(fparam.debug>2 && i==0)printf("\ninsertParticles()3: fbuf.bufI(FEPIGEN) [i*NUM_GENES + gene]=%u  gene=%u  i=%u,",fbuf.bufI(FEPIGEN)[gene*pnum + i/* i*NUM_GENES + gene*/], gene ,i  );
        }
        //if(fparam.debug>2 && i==0)printf("\n");
	} else {
		fbuf.bufI(FGCELL)[i] = GRID_UNDEF;  // gridTot;//    // m_GridTotal  
		//if(i>pnum-10)fbuf.bufI(FGNDX)[i] = atomicAdd ( &fbuf.bufI(FGRIDCNT)[ gridTot-1 ], 1 );  // NB limit on the number of atomic operations on one variable.
        //if(fparam.debug>2)printf("\ninsertParticles()4: i=%i GRID_UNDEF, gc.x=%i, gc.y=%i, gc.z=%i,  ",i, gc.x, gc.y, gc.z);
	}
}


extern "C" __global__ void prefixFixup(uint *input, uint *aux, int len)                         // merge *aux into *input  
{
	unsigned int t = threadIdx.x;
	unsigned int start = t + 2 * blockIdx.x * SCAN_BLOCKSIZE;
	if (start < len)					input[start] += aux[blockIdx.x];     
	if (start + SCAN_BLOCKSIZE < len)   input[start + SCAN_BLOCKSIZE] += aux[blockIdx.x];
    
}

extern "C" __global__ void prefixSum(uint* input, uint* output, uint* aux, int len, int zeroff) // sum *input, write to *output
{
	__shared__ uint scan_array[SCAN_BLOCKSIZE << 1];
	unsigned int t1 = threadIdx.x + 2 * blockIdx.x * SCAN_BLOCKSIZE;
	unsigned int t2 = t1 + SCAN_BLOCKSIZE;

	// Pre-load into shared memory
	scan_array[threadIdx.x] = (t1<len) ? input[t1] : 0.0f;
	scan_array[threadIdx.x + SCAN_BLOCKSIZE] = (t2<len) ? input[t2] : 0.0f;
	__syncthreads();

	// Reduction
	int stride;
	for (stride = 1; stride <= SCAN_BLOCKSIZE; stride <<= 1) {
		int index = (threadIdx.x + 1) * stride * 2 - 1;
		if (index < 2 * SCAN_BLOCKSIZE)
			scan_array[index] += scan_array[index - stride];
		__syncthreads();
	}

	// Post reduction
	for (stride = SCAN_BLOCKSIZE >> 1; stride > 0; stride >>= 1) {
		int index = (threadIdx.x + 1) * stride * 2 - 1;
		if (index + stride < 2 * SCAN_BLOCKSIZE)
			scan_array[index + stride] += scan_array[index];
		__syncthreads();
	}
	__syncthreads();

	// Output values & aux
	if (t1 + zeroff < len)	output[t1 + zeroff] = scan_array[threadIdx.x];
	if (t2 + zeroff < len)	output[t2 + zeroff] = (threadIdx.x == SCAN_BLOCKSIZE - 1 && zeroff) ? 0 : scan_array[threadIdx.x + SCAN_BLOCKSIZE];
	if (threadIdx.x == 0) {
		if (zeroff) output[0] = 0;
		if (aux) aux[blockIdx.x] = scan_array[2 * SCAN_BLOCKSIZE - 1];
	}
}

extern "C" __global__ void tally_denselist_lengths(int num_lists, int fdense_list_lengths, int fgridcnt, int fgridoff )
{
    uint list = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                                  // which dense list is being tallied.
	if ( list >= num_lists ) return;
    register int gridTot =      fparam.gridTotal;
    fbuf.bufI(fdense_list_lengths)[list] = fbuf.bufI(fgridcnt)[(list+1)*gridTot -1] + fbuf.bufI(fgridoff)[(list+1)*gridTot -1];
    
    //if(fparam.debug>2)printf("\ntally_denselist_lengths: gridTot=%u, fbuf.bufI(%i)[%i] = %u, &fdense_list_lengths)[list]=%p \t",
    //       gridTot, fdense_list_lengths, list, fbuf.bufI(fdense_list_lengths)[list], &fbuf.bufI(fdense_list_lengths)[list] );
}

extern "C" __global__ void countingSortFull ( int pnum )                                // Counting Sort - Full (deep copy)
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                             // particle index
	if ( i >= pnum ) return;
    if (fparam.debug>2 && i==0)printf("\ncountingSortFull(): pnum=%u\n",pnum);
	// Copy particle from original, unsorted buffer (msortbuf),
	// into sorted memory location on device (mpos/mvel)
	uint icell = ftemp.bufI(FGCELL) [ i ];                                              // icell is bin into which i is sorted in fbuf.*

	//if ( icell == GRID_UNDEF ) printf("\nicell == GRID_UNDEF, i=,%u,",i);   
	
	if ( icell != GRID_UNDEF ) {	                                                    // This line would eliminate out of range particles from the model, inc. NULL particles.
		// Determine the sort_ndx, location of the particle after sort		
        uint indx =  ftemp.bufI(FGNDX)  [ i ];                                          // indx is off set within new cell
        int sort_ndx = fbuf.bufI(FGRIDOFF) [ icell ] + indx ;                           // global_ndx = grid_cell_offet + particle_offset	
		float3 zero; zero.x=0;zero.y=0;zero.z=0;
        
        // Make dense lists for (i) available genes (ii) active genes (iii) diffusion particles (iv) active/reserve particles. ######################
        // NB req new FGNDX & FGRIDOFF for each of (i-iv).
        // Write (1) list of current array lengths, (2) arrays containing  [sort_ndx] of relevant particles.
        // In use kernels read the array to access correct particle.
        // If there is data only used by such kernels, then it should be stored in a dense array.  
        
		// Transfer data to sort location
		fbuf.bufI (FGRID)   [sort_ndx] =	sort_ndx;                                   // full sort, grid indexing becomes identity		
		fbuf.bufF3(FPOS)    [sort_ndx] =	ftemp.bufF3(FPOS)    [i];
		fbuf.bufF3(FVEL)    [sort_ndx] =	ftemp.bufF3(FVEL)    [i];
		fbuf.bufF3(FVEVAL)  [sort_ndx] =	ftemp.bufF3(FVEVAL)  [i];
		fbuf.bufF3(FFORCE)  [sort_ndx] =    zero;                                       // fbuf.bufF3(FFORCE)[ i ] += force; in contributeForce() requires value setting to 0 
		fbuf.bufF (FPRESS)  [sort_ndx] =	ftemp.bufF(FPRESS)   [i];
		fbuf.bufF (FDENSITY)[sort_ndx] =	ftemp.bufF(FDENSITY) [i];
        fbuf.bufI (FAGE)    [sort_ndx] =	ftemp.bufI(FAGE)     [i];
		fbuf.bufI (FCLR)    [sort_ndx] =	ftemp.bufI(FCLR)     [i];
		fbuf.bufI (FGCELL)  [sort_ndx] =	icell;
		fbuf.bufI (FGNDX)   [sort_ndx] =	indx;
        float3 pos = ftemp.bufF3(FPOS) [i];
        // add extra data for morphogenesis
        // track the sort index of the other particle
        for (int a=0;a<BONDS_PER_PARTICLE;a++){
            // FELASTIDX: [0]current index, [1]elastic limit, [2]restlength, [3]modulus, [4]damping coeff, [5]particle ID, [6]bond index, [7]stress integrator, [8]change-type binary indicator
            uint j = ftemp.bufI(FELASTIDX) [i*BOND_DATA + a*DATA_PER_BOND];             // NB i,j are valid only in ftemp.*
            uint j_sort_ndx = UINT_MAX;
            uint jcell = GRID_UNDEF;
   
            if (j<pnum){
                jcell       = ftemp.bufI(FGCELL) [ j ];                                 // jcell is bin into which j is sorted in fbuf.*
                uint jndx   = UINT_MAX;
                if ( jcell != GRID_UNDEF ) {                                            // avoid out of bounds array reads
                    jndx    =  ftemp.bufI(FGNDX)  [ j ];      
                    if((fbuf.bufI(FGRIDOFF) [ jcell ] + jndx) <pnum){
                        j_sort_ndx = fbuf.bufI(FGRIDOFF) [ jcell ] + jndx ;             // new location in the list of the other particle
                    }
                }                                                                       // set modulus and length to zero if ( jcell != GRID_UNDEF ) // No longer done.
            }
            fbuf.bufI (FELASTIDX) [sort_ndx*BOND_DATA + a*DATA_PER_BOND]  = j_sort_ndx; // NB if (j>=pnum) j_sort_ndx = UINT_MAX; preserves non-bonds
            for (int b=1;b<5/*DATA_PER_BOND*/;b++){                                     // copy [1]elastic limit, [2]restlength, [3]modulus, [4]damping coeff, etc // no longer (iff unbroken)
                fbuf.bufF (FELASTIDX) [sort_ndx*BOND_DATA + a*DATA_PER_BOND +b] = ftemp.bufF (FELASTIDX) [i*BOND_DATA + a*DATA_PER_BOND + b]; // uints
            }                                                                           // old: copy the modulus & length
            fbuf.bufI (FELASTIDX) [sort_ndx*BOND_DATA + a*DATA_PER_BOND +5] = ftemp.bufI (FELASTIDX) [i*BOND_DATA + a*DATA_PER_BOND + 5];   //[5]partID, uint
            fbuf.bufI (FELASTIDX) [sort_ndx*BOND_DATA + a*DATA_PER_BOND +6] = ftemp.bufI (FELASTIDX) [i*BOND_DATA + a*DATA_PER_BOND + 6];   //[6]bond index, uint
            fbuf.bufF (FELASTIDX) [sort_ndx*BOND_DATA + a*DATA_PER_BOND +7] = ftemp.bufF (FELASTIDX) [i*BOND_DATA + a*DATA_PER_BOND + 7];   //[7]stress integrator, float
            fbuf.bufI (FELASTIDX) [sort_ndx*BOND_DATA + a*DATA_PER_BOND +8] = ftemp.bufI (FELASTIDX) [i*BOND_DATA + a*DATA_PER_BOND + 8];   //[8]change-type, uint
        }
        for (int a=0;a<BONDS_PER_PARTICLE;a++){                                         // The list of bonds from other particles 
            uint k = ftemp.bufI(FPARTICLEIDX) [i*BONDS_PER_PARTICLE*2 + a*2];           // NB i,j are valid only in ftemp.*
            uint b = ftemp.bufI(FPARTICLEIDX) [i*BONDS_PER_PARTICLE*2 + a*2 +1];
            uint kndx, kcell, ksort_ndx = UINT_MAX; 
            if (k<pnum){                                                                // (k>=pnum) => bond broken // crashes when j=0 (as set in demo), after run().
                kcell         = ftemp.bufI(FGCELL) [ k ];                               // jcell is bin into which j is sorted in fbuf.*
                if ( kcell   != GRID_UNDEF ) {
                    kndx      = ftemp.bufI(FGNDX)  [ k ];  
                    ksort_ndx = fbuf.bufI(FGRIDOFF)[ kcell ] + kndx ;            
                }
            }
            fbuf.bufI (FPARTICLEIDX) [sort_ndx*BONDS_PER_PARTICLE*2 + a*2]      =  ksort_ndx; // ftemp.bufI(FPARTICLEIDX) [i*BONDS_PER_PARTICLE + a]
            fbuf.bufI (FPARTICLEIDX) [sort_ndx*BONDS_PER_PARTICLE*2 + a*2 +1]   =  b;
            ftemp.bufI(FPARTICLEIDX) [i       *BONDS_PER_PARTICLE*2 + a*2]      = UINT_MAX;   // set ftemp copy for use as a lock when inserting new bonds in ComputeForce(..)
        }
        //if (fparam.debug>2)printf("\n(sort_ndx=%u, i=%u)", sort_ndx, i);
        
        fbuf.bufI (FPARTICLE_ID) [sort_ndx] =	ftemp.bufI(FPARTICLE_ID) [i];
        fbuf.bufI (FMASS_RADIUS) [sort_ndx] =	ftemp.bufI(FMASS_RADIUS) [i];
        fbuf.bufI (FNERVEIDX)    [sort_ndx] =	ftemp.bufI(FNERVEIDX)    [i];
        
        uint* fbuf_epigen  = &fbuf.bufI(FEPIGEN)[sort_ndx];
        uint* ftemp_epigen = &ftemp.bufI(FEPIGEN)[i];
        for (int a=0;a<NUM_GENES;a++)  fbuf_epigen[pnum*a]  = ftemp_epigen[pnum*a];  // NB launched with pnum=mMaxPoints=fparam.maxPoints
        
        float* fbuf_conc  = &fbuf.bufF(FCONC)[sort_ndx * NUM_TF];
        float* ftemp_conc = &ftemp.bufF(FCONC)[i * NUM_TF];
        for (int a=0;a<NUM_TF;a++)     fbuf_conc[a] = ftemp_conc[a]; 
            //fbuf.bufF (FCONC)[sort_ndx * NUM_TF + a] = ftemp.bufF(FCONC)[i * NUM_TF + a];
            //__syncwarp();
	}
}

/*
extern "C" __global__ void countingSortEPIGEN ( int pnum )    
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                             // particle index
	if ( i >= pnum ) return;
    if (fparam.debug>2 && i==0)printf("\ncountingSortFull(): pnum=%u\n",pnum);
	// Copy particle from original, unsorted buffer (msortbuf),
	// into sorted memory location on device (mpos/mvel)
	uint icell = ftemp.bufI(FGCELL) [ i ];                                              // icell is bin into which i is sorted in fbuf.*

	//if ( icell == GRID_UNDEF ) printf("\nicell == GRID_UNDEF, i=,%u,",i);   
	
	if ( icell != GRID_UNDEF ) {	                                                    // This line would eliminate out of range particles from the model, inc. NULL particles.
		// Determine the sort_ndx, location of the particle after sort		
        uint indx =  ftemp.bufI(FGNDX)  [ i ];                                          // indx is off set within new cell
        int sort_ndx = fbuf.bufI(FGRIDOFF) [ icell ] + indx ;                           // global_ndx = grid_cell_offet + particle_offset	

        uint* fbuf_epigen  = &fbuf.bufI(FEPIGEN)[sort_ndx];
        uint* ftemp_epigen = &ftemp.bufI(FEPIGEN)[i];
        
        for (int a=0;a<NUM_GENES;a++)   {
            fbuf_epigen[pnum*a]  = ftemp_epigen[pnum*a];
            if(sort_ndx>pnum) printf("\ncountingSortEPIGEN: sort_ndx=,%u, i=,%u, pnum=,%u, a=,%u, ftemp_epigen[pnum*a]=,%u,  ",
                   sort_ndx, i, pnum, a, ftemp_epigen[pnum*a]
                  );
        }
    }
}
*/


extern "C" __global__ void countingSortDenseLists ( int pnum )
{
    unsigned int bin = threadIdx.x + blockIdx.x * SCAN_BLOCKSIZE/2;
    register int gridTot =      fparam.gridTotal;
    if (fparam.debug>2 && bin==0) printf("\n\n######countingSortDenseLists###### bin==0  gridTot=%u, fbuf.bufI (FGRIDOFF)[bin]=%u \n",gridTot, fbuf.bufI (FGRIDOFF)[0]);
	if ( bin >= gridTot ) return;                                    // for each bin, for each particle, for each gene, 
                                                                     // if gene active, then write to dense list 
    uint count = fbuf.bufI (FGRIDCNT)[bin];
    //if (fparam.debug>2 && bin%10000==0)printf("|");
    if (count==0) return;                                            // return here means that IFF all bins in this threadblock are empty,
    /*
    //if (fparam.debug>2 && count>0&&bin%10000==0)printf("\n\ncountingSortDenseLists: (count>100) bin=%u\n\n",bin);                           // then this multiprocessor is free for the next threadblock.
    //if (fparam.debug>2 && bin%100==0)printf("!count=%u,",count);                                                                 // NB Faster still would be a list of occupied bins.
    //if (fparam.debug>2 && count>27)printf("\ncount=%u,bin=%u\t",count,bin);
    
    uint grdoff_ =0;
    if(bin>0)grdoff_ =fbuf.bufI (FGRIDOFF)[bin-1];
    */
    uint grdoffset = fbuf.bufI (FGRIDOFF)[bin];
    uint gene_counter[NUM_GENES]={0};
    /*
    int step = grdoff_-grdoffset;
    if (fparam.debug>2 && bin>0 && step>27)  printf("\nbin=%u, gridoff step = %u, grdoff_=%u,  grdoffset=%u \t",bin, step, grdoff_, grdoffset );
    if (fparam.debug>2 && grdoffset>2200 && grdoffset<22100) printf("\ngrdoffset=%u  ",grdoffset);
    */
    register uint* lists[NUM_GENES];
    for (int gene=0; gene<NUM_GENES;gene++) lists[gene]=fbuf.bufII(FDENSE_LISTS)[gene]; // This element entry is a pointer
    
    register uint* offsets[NUM_GENES];
    for (int gene=0; gene<NUM_GENES;gene++) offsets[gene]=&fbuf.bufI(FGRIDOFF_ACTIVE_GENES)[gene * gridTot];   // The address of this element
    
    if (grdoffset+count > pnum){    printf("\n\n!!Overflow: (grdoffset+count > pnum), bin=%u \n",bin);     return;}
    
    for(uint particle=grdoffset; particle<grdoffset+count; particle++){
        if (fparam.debug>2 && particle>=22000 && particle<20030) printf("\nparticle==%u, ",particle);
        for(int gene=0; gene<NUM_GENES; gene++){
            /*
            if (gene==2 && particle%100==0) printf("\n offsets[gene][bin] + gene_counter[gene] =%u , particle=%u , fbuf.bufI(FEPIGEN) [particle + pnum*gene]=%u\t", 
                offsets[gene][bin] + gene_counter[gene] , particle, fbuf.bufI(FEPIGEN) [particle + pnum*gene]);
            */
            if(  /*(int)*/fbuf.bufI(FEPIGEN) [particle + pnum*gene] >0 ) {    // NB launched with pnum=mMaxPoints=fparam.maxPoints      // if (this gene is active in this particle)
                lists[gene][ offsets[gene][bin] + gene_counter[gene] ]=particle;
                gene_counter[gene]++;
                //if (fparam.debug>2 )printf("*");
                /*
                 * if (gene>2/_*particle<10&&gene==2*_/)printf("\ncountingSortDenseLists()1:  particle=,%u, gene=,%u, bin=,%u, grdoffset=,%u, count=,%u, address=,%p, \t offsets[gene][bin]=,%u, gene_counter[gene]=,%u, fbuf.bufI(FEPIGEN) [particle + pnum*gene]=%u ",
                    particle, gene, bin, grdoffset, count,
                    &lists[gene][ offsets[gene][bin] + gene_counter[gene] ],
                    offsets[gene][bin],
                    gene_counter[gene],
                    fbuf.bufI(FEPIGEN) [particle + pnum*gene]//UINT_MAX//
                                                   );
                */
                if (fparam.debug>2 && gene_counter[gene]>fbuf.bufI(FGRIDCNT_ACTIVE_GENES)[gene*gridTot +bin] )   
                    printf("\n Overflow: particle=,%u, ID=,%u, gene=,%u, bin=,%u, gene_counter[gene]=,%u, fbuf.bufI (FGRIDCNT_ACTIVE_GENES)[gene*gridTot +bin]=,%u \t\t",
                           particle, fbuf.bufI(FPARTICLE_ID)[particle], gene, bin, gene_counter[gene], fbuf.bufI (FGRIDCNT_ACTIVE_GENES)[gene*gridTot +bin]);
                    /*
                    //else printf("\n Non-overflow: particle=%u, ID=%u, gene=%u, bin=%u, gene_counter[gene]=%u, fbuf.bufI (FGRIDCNT_ACTIVE_GENES)[gene*gridTot +bin]=%u \t\t",
                    //       particle, fbuf.bufI(FPARTICLE_ID)[particle], gene, bin, gene_counter[gene], fbuf.bufI (FGRIDCNT_ACTIVE_GENES)[gene*gridTot +bin]);
                    */
            }else if (fparam.debug>2 && gene==2 && particle%1000==0)printf("*");
        }
    }
/* 
     * debug chk 
    if (fparam.debug>2){
    uint particle=grdoffset, gene=2;
            if(particle<10 && gene==2) {
                lists[gene][ offsets[gene][bin] + gene_counter[gene] ]=particle;   
                printf("\ncountingSortDenseLists: gene=%u, bin=%u, lists[gene][ offsets[gene][bin] + gene_counter[gene] ] = %u,  offsets[gene][bin]=%u,  gene_counter[gene]=%u ", 
                       gene, bin, lists[gene][ offsets[gene][bin] + gene_counter[gene] ],  offsets[gene][bin], gene_counter[gene]++ );
                gene_counter[gene]++;
            } 
    }
*/
}

extern "C" __global__ void countingSortChanges ( int pnum )
{
    uint bin = bin = threadIdx.x + blockIdx.x * SCAN_BLOCKSIZE/2;  //__mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index
    {   // debug chk
   /* 
    uint bin2=bin;
    float3 gridDelta = fparam.gridDelta;                                //  even if other variable have to be moved to slower 'local' memory  
    int3 gridRes =		fparam.gridRes;   
	float3 gc, binPos;
    gc.x=bin2%gridRes.x;
    bin2-=gridRes.x;
    bin2=bin2/gridRes.x;
    gc.z=bin2%gridRes.z;
    gc.y=bin2/gridRes.z;
    binPos=gc/gridDelta;
   */
  // if (fparam.debug>2 && threadIdx.x==0) printf("\nblockIdx.x=,%u \t",blockIdx.x);
    
    //unsigned int bin = threadIdx.x + blockIdx.x * SCAN_BLOCKSIZE/2;     // NB have to searach all particles => use main list bins. 
    }
    register int gridTot =      fparam.gridTotal;
	if ( bin >= gridTot ) return;                                    // for each bin, for each particle, for each change_list, 
                                                                     // if change_list active, then write to dense list 
    uint count = fbuf.bufI (FGRIDCNT/*_CHANGES*/)[bin];
    //if (count==0) return; 
    {   // debug chk
    /*if (fparam.debug>2 && threadIdx.x==0 && blockIdx.x%32==0)*///if(count!=0)printf("\ncountingSortChanges: bin=%u, gridTot=%u, count=%u, blockIdx.x=%u,  blockDim.x=%u, threadIdx.x=%u \t",bin, gridTot, count, blockIdx.x , blockDim.x, threadIdx.x );
    //if (fparam.debug>2 &&  bin==471311 /*blockIdx.x<100 && bin%32==0*/)printf("\n\n###countingSortChanges: bin=,%u, binPos=(,%f,%f,%f,) gridTot=,%u, count=,%u, blockIdx.x=,%u,  blockDim.x=,%u, threadIdx.x=,%u \t\n",
    //     bin, binPos.x, binPos.y, binPos.z, gridTot, count, blockIdx.x , blockDim.x, threadIdx.x );
    }
                                               // return here means that if all bins in this threadblock are empty,
                                                                     // then this multiprocessor is free for the next threadblock.
  //if (fparam.debug>2)printf("\ncountingSortChanges: bin=%u, count=%u \t",bin,count);
    uint grdoffset = fbuf.bufI (FGRIDOFF)[bin];
    uint change_list_counter[NUM_CHANGES]={0};                       // holds off set within the change bin for this change type, for the particles added so far.  
    
    register uint* lists[NUM_CHANGES];
    for (int change_list=0; change_list<NUM_CHANGES;change_list++) lists[change_list]=fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list];           // This element entry is a pointer
    
    //if(fparam.debug>2 && bin == 1/*change_list>6*/) for (int change_list=0; change_list<NUM_CHANGES;change_list++) printf("\nPointer to lists[%u] = %p,",change_list, lists[change_list]);
    if (count==0) return; 
    
    register uint list_length[NUM_CHANGES];
    for (uint change_list=0; change_list<NUM_CHANGES;change_list++) list_length[change_list]=fbuf.bufI(FDENSE_BUF_LENGTHS_CHANGES)[change_list];/*FDENSE_LIST_LENGTHS_CHANGES*/
/*
    if (bin==0){
        printf("\n");
        for(uint k=0; k<9; k++){
            printf("\n##countingSortChanges1: k=%u, list_length[%u]=%u, fbuf.bufI(FDENSE_BUF_LENGTHS_CHANGES/_*FDENSE_LIST_LENGTHS_CHANGES*_/)[change_list]=%u,  &fbuf.bufI(FDENSE_BUF_LENGTHS_CHANGES /_*FDENSE_LIST_LENGTHS_CHANGES*_/)[%u]=%p \t",
                k, k, list_length[k], fbuf.bufI(FDENSE_BUF_LENGTHS_CHANGES/_*FDENSE_LIST_LENGTHS_CHANGES*_/)[k], k, &fbuf.bufI(FDENSE_BUF_LENGTHS_CHANGES/_*FDENSE_LIST_LENGTHS_CHANGES*_/)[k]);
        }
    }
*/ 
    register uint* offsets[NUM_CHANGES];
    for (int change_list=0; change_list<NUM_CHANGES; change_list++)   offsets[change_list] = &fbuf.bufI(FGRIDOFF_CHANGES)[change_list * gridTot];   // The address of this element
/*
  //if (fparam.debug>2)printf("\ncountingSortChanges: grdoffset=%u, count=%u, pnum=%u \t",grdoffset, count, pnum);
*/
    if (grdoffset+count > pnum){ /* if (fparam.debug>2){printf("\n\n!!Overflow,  countingSortChanges: (grdoffset+count > pnum), bin=%u \n",bin);}  */   return;}
    
    for(uint particle=grdoffset; particle<grdoffset+count; particle++){                                                             // loop through particleIDx in bin in main particle list
/*
 * if(particle==grdoffset){
    float3 pos = fbuf.bufF3(FPOS)[particle];
    uint ID = fbuf.bufI(FPARTICLE_ID)[particle];
    printf("\ncountingSortChanges: bin=%u, particle=%u, ID=%u\t pos.x=%f, pos.y=%f, pos.z=%f",bin, particle, ID, pos.x, pos.y, pos.z);
    }
*/
        for(uint bond=0; bond<BONDS_PER_PARTICLE; bond++){                                                                          // loop through bonds on particle
            uint change = fbuf.bufI(FELASTIDX) [particle*BOND_DATA + bond*DATA_PER_BOND + 8];                                       // binary change indicator per bond.
          //if (fparam.debug>2)printf("\ncountingSortChanges: change=%u \t",change);
            if(change) {
                for (uint change_type=1, change_list=0; change_list<NUM_CHANGES; change_type*=2, change_list++){                    // loop through change indicator  
                  /*  
                   //printf("\nparticle=,%u, change_list=,%u, countingSortChanges: change=,%u, change_type=,%u, (change & change_type)=,%u, \t",
                   //       particle, change_list, change,change_type, (change & change_type) ); 
                   */ 
                    if(change & change_type){                                                                                       // bit mask to ID change type due to this bond
                        //if (fparam.debug>2)printf("\n\ncountingSortChanges: particle=%u, bond=%u \n\n",particle,bond);
                        lists[change_list] [( offsets[change_list][bin] + change_list_counter[change_list] )]                               = particle;   // write particleIdx to change list
                        lists[change_list] [( offsets[change_list][bin] + change_list_counter[change_list] + list_length[change_list] )]    = bond;       // write bondIdx to change list
                        
                        /*
                        if(change_list==1) printf("\ncountingSortChanges, change_list==1: particle=%u, bond=%u, particle_index=%u \t",
                            lists[change_list] [( offsets[change_list][bin] + change_list_counter[change_list] )],
                            lists[change_list] [( offsets[change_list][bin] + change_list_counter[change_list] + list_length[change_list] )],
                            offsets[change_list][bin] + change_list_counter[change_list]
                        );
                        */
                        
                         /*
                         //printf("[%u](%u,%u),",change_list,particle, bond);  // && threadIdx.x==0   if(change_list>6  )
                         //if(change_list==7 && particle==0) printf("\n\n[7](0,%u),", bond);
                         //if(change_list==8) printf("\n\n[8](%u,%u),",particle, bond);
                         */
                        {   // debug chk
                        /*
                        printf("\ncountingSortChanges: change_list=%u, \tlists[change_list]=%p, \tlist_length[change_list]=%u, \t&lists[change_list][particle]=%p, \t&lists[change_list][bond]=%p     \t", 
                               change_list, lists[change_list], list_length[change_list],
                               &lists[change_list][ (offsets[change_list][bin] + change_list_counter[change_list]) ], 
                               &lists[change_list][ (offsets[change_list][bin] + change_list_counter[change_list] + list_length[change_list]) ]
                              );
                        */
                        /*
                        if (particle==0){
                            printf("\ncountingSortChanges2, : lists[0]=%p, lists[1]=%p, &lists[0][0]=%p, &lists[0][1]=%p \n", lists[0] , lists[1], &lists[0][0], &lists[0][1] ); 
                            printf("\ncountingSortChanges3, :change_list=%u, bin=%u,  offsets[%u][%u]=%u, change_list_counter[%u]=%u, list_length[%u]=%u \n",
                                   change_list, bin, change_list, bin, offsets[change_list][bin], change_list, change_list_counter[change_list], change_list, list_length[change_list] );
                            for(int k=0; k<9; k++)
                                printf("\ncountingSortChanges4, :list_length[%u]=%u, fbuf.bufI(FDENSE_LIST_LENGTHS_CHANGES)[change_list]=%u,\t",
                                    k, list_length[k], fbuf.bufI(FDENSE_LIST_LENGTHS_CHANGES)[k]);
                        }
                        */
                        //if (fparam.debug>2 && change_type==2)printf("\ncountingSortChanges, : particle=%u, bond=%u, change=%u, change_type=%u, list_length[%u]=%u,  (offsets[change_list][bin] + change_list_counter[change_list])=%u  \t", 
                        //   particle, bond, change, change_type, change_list, list_length[change_list],  (offsets[change_list][bin] + change_list_counter[change_list]) );
                        /*
                        if (particle==0){
                            printf("\ncountingSortChanges2:  ");
                            for(int k=0; k<NUM_CHANGES; k++){
                                printf("\nlists[%u]=%p,  list_length[%u]=%u,  step=%ld", k, lists[k], k, list_length[k], (lists[k+1]-lists[k])/2  );
                            }
                        }
                        */
                       /* 
                       if (fparam.debug>2 && particle<10/_*00*_/) printf("\ncountingSortChanges()1: debug chk: particle=%u, bond=%u, change=%u, change_list=%u, change_list_counter[change_list]=%u, offsets[change_list][bin]=%u \t\t fbuf.bufI(FGRIDCNT_CHANGES)[ 0*gridTot + fbuf.bufI(FGCELL)[particle] ] =%u, fbuf.bufI(FGCELL)[particle]=%u, \t\t particleIndx=%u, bondIndx=%u \t", 
                            particle, bond, change, change_list, change_list_counter[change_list], offsets[change_list][bin],
                            fbuf.bufI(FGRIDCNT_CHANGES)[ 0*gridTot + fbuf.bufI(FGCELL)[particle] ],
                            fbuf.bufI(FGCELL)[particle],
                            lists[change_list][ (offsets[change_list][bin] + change_list_counter[change_list]) ],
                            lists[change_list][ (offsets[change_list][bin] + change_list_counter[change_list] + list_length[change_list]) ]   // NB only heal : change_list=0
                        );
                        */
                        }
                        change_list_counter[change_list]++;
                    }
                }
            }
        }
    }
 {   // debug chk
  //      for(uint particle=grdoffset; particle<grdoffset+count; particle++){ // ? has found particle in change list, _not_ index in main list  ?     // loop through particles in bin
  //      for(uint bond=0; bond<BONDS_PER_PARTICLE; bond++){                                                                                      // loop through bonds on particle
  //          uint change = fbuf.bufI(FELASTIDX) [particle*BOND_DATA + bond*DATA_PER_BOND + 8];                                                  // binary change indicator per bond.
          //if (fparam.debug>2)printf("\ncountingSortChanges: change=%u \t",change);
  //          if(change) {
  //              for (uint change_type=1, change_list=0; change_list<NUM_CHANGES; change_type*=2, change_list++){                               // loop through change indicator  
                   //if (fparam.debug>2)printf("\ncountingSortChanges: change=%u, change_type=%u, (change & change_type)=%u \t",change,change_type, (change & change_type) ); 
                    
  //                  if(change & change_type){                                                                                                  // bit mask to ID change type due to this bond
                        //if (fparam.debug>2)printf("\n\ncountingSortChanges: particle=%u, bond=%u \n\n",particle,bond);
                      //  lists[change_list][ (offsets[change_list][bin] + change_list_counter[change_list]) ]=particle;                         // write particleIdx to change list
                      //  lists[change_list][ (offsets[change_list][bin] + change_list_counter[change_list] + list_length[change_list]) ]=bond;  // write bondIdx to change list
                        /*
                        printf("\ncountingSortChanges: change_list=%u, \tlists[change_list]=%p, \tlist_length[change_list]=%u, \t&lists[change_list][particle]=%p, \t&lists[change_list][bond]=%p     \t", 
                               change_list, lists[change_list], list_length[change_list],
                               &lists[change_list][ (offsets[change_list][bin] + change_list_counter[change_list]) ], 
                               &lists[change_list][ (offsets[change_list][bin] + change_list_counter[change_list] + list_length[change_list]) ]
                              );
                        */
                        /*
                        if (particle==0){
                            printf("\ncountingSortChanges2, : lists[0]=%p, lists[1]=%p, &lists[0][0]=%p, &lists[0][1]=%p \n", lists[0] , lists[1], &lists[0][0], &lists[0][1] ); 
                            printf("\ncountingSortChanges3, :change_list=%u, bin=%u,  offsets[%u][%u]=%u, change_list_counter[%u]=%u, list_length[%u]=%u \n",
                                   change_list, bin, change_list, bin, offsets[change_list][bin], change_list, change_list_counter[change_list], change_list, list_length[change_list] );
                            for(int k=0; k<9; k++)
                                printf("\ncountingSortChanges4, :list_length[%u]=%u, fbuf.bufI(FDENSE_LIST_LENGTHS_CHANGES)[change_list]=%u,\t",
                                    k, list_length[k], fbuf.bufI(FDENSE_LIST_LENGTHS_CHANGES)[k]);
                        }
                        */
                        //if (fparam.debug>2 && change_type==2)printf("\ncountingSortChanges, : particle=%u, bond=%u, change=%u, change_type=%u, list_length[%u]=%u,  (offsets[change_list][bin] + change_list_counter[change_list])=%u  \t", 
                        //   particle, bond, change, change_type, change_list, list_length[change_list],  (offsets[change_list][bin] + change_list_counter[change_list]) );
                        /*
                        if (particle==0){
                            printf("\ncountingSortChanges2:  ");
                            for(int k=0; k<NUM_CHANGES; k++){
                                printf("\nlists[%u]=%p,  list_length[%u]=%u,  step=%ld", k, lists[k], k, list_length[k], (lists[k+1]-lists[k])/2  );
                            }
                        }
                        */
                        
                        /*
                         * printf("\ncountingSortChanges()2: debug chk: particle=%u, bond=%u, change=%u, change_list=%u, bin=%u, \t\t offsets[change_list][bin+1] - offsets[change_list][bin]=%u,  fbuf.bufI(FGRIDCNT_CHANGES)[ 0*gridTot + fbuf.bufI(FGCELL)[particle] ] =%u, fbuf.bufI(FGCELL)[particle]=%u, \t\t change_list_counter[change_list]=%u, list_length[change_list]=%u, particleIndx=%u, bondIndx=%u \t", 
                            particle, bond, change, change_list, bin, offsets[change_list][bin+1] - offsets[change_list][bin],
                            fbuf.bufI(FGRIDCNT_CHANGES)[ 0*gridTot + fbuf.bufI(FGCELL)[particle] ],
                            fbuf.bufI(FGCELL)[particle],
                            change_list_counter[change_list], list_length[change_list],
                            lists[change_list][ (offsets[change_list][bin] + change_list_counter[change_list]) ],
                            lists[change_list][ (offsets[change_list][bin] + change_list_counter[change_list] + list_length[change_list]) ]   // NB only heal : change_list=0
                        );*/
    //                }
    //            }
    //        }
    //    }
    //}
    /*
    for(uint particle=grdoffset; particle<grdoffset+count; particle++){ // ? has found particle in change list, _not_ index in main list  ?     // loop through particles in bin
        for(uint bond=0; bond<BONDS_PER_PARTICLE; bond++){                                                                                      // loop through bonds on particle
            uint change = fbuf.bufI(FELASTIDX) [particle*BOND_DATA + bond*DATA_PER_BOND + 8];
            if(change==1) {
                for (uint counter=0;counter<change_list_counter[0];counter++){
                    if (fparam.debug>2)printf("\ncountingSortChanges()2: debug chk: particle=%u, bond=%u, change=%u, particleIndx=%u, bondIndx=%u \t", 
                       particle, bond, change,
                       lists[0][ (offsets[0][bin] + change_list_counter[0]) ],
                       lists[0][ (offsets[0][bin] + change_list_counter[0] + list_length[0]) ]                                                  // NB only heal : change_list=0
                    );
                }
            }else if (fparam.debug>2)printf("\n#countingSortChanges(): debug chk: particle=%u, bond=%u, change=%u \t",particle, bond, change );
            
        }
    }// end debug chk
    */
 }
}

extern "C" __device__ float contributePressure ( int i, float3 p, int cell, float &sum_p6k )  
// pressure due to particles in 'cell'. NB for each particle there are 27 cells in which interacting particles might be.
{			
	if ( fbuf.bufI(FGRIDCNT)[cell] == 0 ) return 0.0;                       // If the cell is empty, skip it.

	float3 dist;
	float dsq, r, q, b, c, sum = 0.0;//, sum_p6k = 0.0;
	register float d2 = fparam.psimscale * fparam.psimscale;                // max length in simulation space
	register float r2 = fparam.r2 / d2;                                     // = m_FParams.psmoothradius^2 / m_FParams.psimscale^2
    register float H  = fparam.H;                                           // = m_FParams.psmoothradius / m_FParams.psimscale;
    register float sr = fparam.psmoothradius;
	
	int clast = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];      // off set of this cell in the list of particles,  PLUS  the count of particles in this cell.

	for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++ ) {    // For particles in this cell.
		int pndx = fbuf.bufI(FGRID) [cndx];                                 // index of this particle
		dist = p - fbuf.bufF3(FPOS) [pndx];                                 // float3 distance between this particle, and the particle for which the loop has been called.
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);              // scalar distance squared
        
        // From https://github.com/DualSPHysics/DualSPHysics/wiki/3.-SPH-formulation#31-smoothing-kernel 
        /*
         * q=r/h, where r=dist between particles, h=smoothing length
         * 
         * W(r,h) = alpha_D(1-q/2)**4 *(2q+1) for 0<=q<=2
         * 
         * where alpha_D = 21/(16*Pi*h**3)  , the normalization kernel in 3D,
         * i.e. 1/integral_(0,2){kernel * area of a sphere}dr 
         * 
         */
        
		if ( dsq < r2 && dsq > 0.0) {                                       // if(in-range && not the same particle) ie unused particles can be stored at one point.
            r=sqrt(dsq);
            q=r/sr;                                                         //r/H; i.e ss:=1
            b=(1-q/2.0);
            b*=b; 
            b*=b;
            sum  += b*(2*q +1);//(H+4*r);                                   // Wendland C^2 quintic kernel for 3 dimensions.
            /*
            if (i<10)printf("\n contribPressure()1: i=,%u, ,j=,%u,\t ,r=sqrt(dsq)=,%f, ,H=sr/ss=,%f, q=r/H=,%f, ,b=(1-q/2.0)^3,%f,\t ,pressure= 1-q/2.0)^3*(2*q +1)=,%f  ",i, pndx, r, H, q, b, b*(2*q +1) );

			c = (r2 - dsq)*d2;
			sum_p6k += c * c * c;
            if (i<10)printf("\ncontribPressure()2: i=,%u, ,j=,%u, r2=sr^2/ss^2=,%f, dsq=,%f, d2=ss^2=,%f,\t\t,c=(r2-dsq)*d2=,%f, ,,,,pressure_p6k=c^3=,%f, ", i, pndx,  r2, dsq, d2, c,  c*c*c );
            */
		}
	}
	return sum;                                                             // NB a scalar value for pressure contribution, at the current particle, due to particles in this cell.
}
			
extern "C" __global__ void computePressure ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                 // particle index
	if ( i >= pnum ) return;

	// Get search cell
	int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	uint gc = fbuf.bufI(FGCELL) [i];                                        // get grid cell of the current particle.
	if ( gc == GRID_UNDEF ) return;                                         // IF particle not in the simulation
	gc -= nadj;

	// Sum Pressures
	float3 pos = fbuf.bufF3(FPOS) [i];
	float sum = 0.0, sum_p6k = 0.0;
	for (int c=0; c < fparam.gridAdjCnt; c++) {                                    
		sum += contributePressure ( i, pos, gc + fparam.gridAdj[c], sum_p6k );
	}
	__syncthreads();
    
	// Compute Density & Pressure
    float old_sum=sum,  old_sum_p6k=sum_p6k;
    float rest_dens = 0.0015;/*fparam.prest_dens*/
	sum = sum * fparam.pmass * fparam.wendlandC2kern;
	//sum_p6k = sum_p6k * fparam.pmass * fparam.poly6kern;
    
	if ( sum == 0.0 ) sum = 1.0;
	fbuf.bufF(FPRESS)  [ i ] = ( sum - rest_dens ) * fparam.pintstiff;   // pressure = (diff from rest density) * stiffness
	fbuf.bufF(FDENSITY)[ i ] = 1.0f / sum;
    /*
    if (i<10)printf("\n computePressure()2: i=,%u, ,old_sum=,%f, ,old_sum_p6k=,%-20.20f, ,sum*=pmass*wendlandC2kern=,%.32f, ,sum_p6k*=pmass*poly6kern=,%f,\t ,wendlandC2kern=,%f, poly6kern=,%f, ,pmass=,%f, ,prest_dens=,%f, ,pintstiff=,%f,\t ,Pressure=(sum-prest_dens)*pintstiff=,%f  ", 
        i, old_sum, old_sum_p6k, sum, sum_p6k, fparam.wendlandC2kern, fparam.poly6kern, fparam.pmass, rest_dens, fparam.pintstiff, fbuf.bufF(FPRESS)[i]  );
    */
}

extern "C" __global__ void computeGeneAction ( int pnum, int gene, uint list_len )  //NB here pnum is for the dense list NB Must zero ftemp.bufI(FEPIGEN) and ftemp.bufI(FCONC) before calling.
{
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                                         // particle index
    if ( i >= list_len ) return;
    uint particle_index = fbuf.bufII(FDENSE_LISTS)[gene][i];
    /*if (particle_index >= pnum){
        printf("\ncomputeGeneAction: (particle_index >= pnum),  gene=%u, i=%u, list_len=%u, particle_index=%u, pnum=%u .\t",
            gene, i, list_len, particle_index, pnum);
    } */   
    int delay = (int)fbuf.bufI(FEPIGEN)[gene*fparam.maxPoints + particle_index];                                // Change in _epigenetic_ activation of this particle
    //printf("\nDelay=%i, particle_index=%u\t", delay, particle_index);
    if (0 < delay && delay < INT_MAX){                                                                           // (FEPIGEN==INT_MAX) => active & not counting down.
        fbuf.bufI(FEPIGEN)[gene*fparam.maxPoints + particle_index]--;                                           // (FEPIGEN<1) => inactivated @ insertParticles(..)
        if (delay==1  &&  gene<NUM_GENES && fbuf.bufI(FEPIGEN)[ (gene+1)*fparam.maxPoints + particle_index ] )  // If next gene is active, start count down to inactivate it.
            fbuf.bufI(FEPIGEN)[(gene+1)*fparam.maxPoints + particle_index] = fgenome.delay[gene+1] ;            // Start countdown to silence next gene.
    }                                                                                               // (fgenome.delay[gene+1]==INT_MAX) => barrier to spreading inactivation.
    uint sensitivity[NUM_GENES];                                                                    // TF sensitivities : broadcast to threads
    #pragma unroll                                                                                  // speed up by eliminating loop logic.
    for(int j=0;j<NUM_GENES;j++) sensitivity[j]= fgenome.sensitivity[gene][j];                      // for each gene, its sensitivity to each TF or morphogen
    /*if(i==list_len-1)printf("\ncomputeGeneAction Chk : gene=%u, i=%u, list_len=%u, particle_index=%u, pnum=%u ,  sensitivity[15]=%u.\t",
            gene, i, list_len, particle_index, pnum, sensitivity[15]); */                             // debug chk 
    float activity=0;                                                                               // compute current activity of gene
    #pragma unroll
    for (int tf=0;tf<NUM_TF;tf++){                                                                  // read FCONC
        if(sensitivity[tf]!=0){                                                                     // skip reading unused fconc[]
            activity +=  sensitivity[tf] * fbuf.bufI(FCONC)[particle_index + fparam.maxPoints*tf];
        }                                                           
    }
    // Compute actions                                             // Non-difusible TFs inc instructions to particle modification kernel wrt behaviour (cell type). 
    int numTF =  fgenome.secrete[gene][2*NUM_TF];                  // (i) secrete sparse list of TFs  => atomicAdd(ftemp...) to allow async gene kernels.
    for (int j=0;j<numTF;j++){
        int tf = fgenome.secrete[gene][j*2];
        int secretion_rate = fgenome.secrete[gene][j*2 + 1];
        atomicAdd( &ftemp.bufI(FCONC)[particle_index*NUM_TF +tf], secretion_rate*activity);
    }
    int numLRNA = fgenome.activate[gene][2*NUM_GENES];             // (ii) secrete spare list long RNA => activate other genes.  NB threshold.
    for (int j=0;j<numLRNA;j++){
        int other_gene = fgenome.activate[gene][j*2];
        int threshold = fgenome.activate[gene][j*2 + 1];
        if(threshold<activity)
        atomicAdd( &ftemp.bufI(FEPIGEN)[other_gene*fparam.maxPoints + particle_index], 1);   // what should be the initial state of other_gene when activated ?
    }
}

extern "C" __global__ void tallyGeneAction ( int pnum, int gene, uint list_length ){// called by ComputeGenesCUDA () after computeGeneAction (..) & synchronize().
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                            // particle index
    if ( particle_index >= list_length ) return;                                                    // pnum should be length of list.
    // ## TODO convert i to particle index _iff_ not called for all particles : use a special dense list for "living tissue", made at same time as gene lists
    uint i = fbuf.bufII(FDENSE_LISTS)[2][particle_index];                                           // call for dense list of living cells (gene'2'living/telomere (has genes))
    if ( i >= pnum ) return; 
    
    float * fbufFCONC = &fbuf.bufF(FCONC)[i*NUM_TF];
    float * ftempFCONC = &ftemp.bufF(FCONC)[i*NUM_TF];
    uint * fbufFEPIGEN = &fbuf.bufI(FEPIGEN)[i]; //*NUM_GENES                                             // TODO FEPIGEN is a uint here. May need to pack binaries for spread & stop. See paper.
    uint * ftempFEPIGEN = &ftemp.bufI(FEPIGEN)[i]; //*NUM_GENES   // ## need to zero ftemp after counting sort full
    
    for(int j=0; j<NUM_TF;j++)      fbufFCONC[j] += ftempFCONC[j];  // *fparam.maxPoints
    for(int j=0; j<NUM_GENES;j++) fbufFEPIGEN[j*fparam.maxPoints] += ftempFEPIGEN[j*fparam.maxPoints];
}


extern "C" __global__ void computeNerveActivation ( int pnum ) //TODO computeNerveActivation    // initially simple sparse random connections + STDP, later neurogenesis
{                                                                 // NB (i) sensors concetrated in hands & feet (ii)stimuls from womb wall 
    
}

extern "C" __global__ void computeMuscleContraction ( int pnum ) //TODO computeMuscleContraction  // read attached nerve, compute force  
{
    
}

extern "C" __global__ void assembleMuscleFibresOutGoing ( int pnum, uint list, uint list_length ) // used for muscle, elastic ligg, and tendon
{
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                            // particle index
    if ( particle_index >= list_length )return;
    uint i = fbuf.bufII(FDENSE_LISTS)[list][particle_index];
    if (i > fparam.maxPoints) return;
    
    printf("\nassembleMuscleFibresOutGoing() chk1: i=%u   ",i  );
    
    ///////////// Swap outgoing bond indicies 
    // Find highest stress incoming bond
    float maxStress         = 0.0;
    uint maxStressBondIdx   = 0;
    float stress;
    char* bond_char_ptr = &fbuf.bufC(FELASTIDX)[i*BOND_DATA];
    uint* bond_uint_ptr = &fbuf.bufI(FELASTIDX)[i*BOND_DATA];                                       //fbuf.bufI(FELASTIDX)[i*BOND_DATA + bond*DATA_PER_BOND +  ] ;
    float*bond_flt_ptr  = &fbuf.bufF(FELASTIDX)[i*BOND_DATA];                                       //FELASTIDX   [0]current index, [1]elastic limit, [2]restlength, [3]modulus,
                                                                                                                //[4]damping coeff, [5]particle ID,   [6]bond index, 
                                                                                                                //[7]stress integrator,  [8]change-type binary indicator
    for(int bond=0; bond<BONDS_PER_PARTICLE; bond++){                                               // find highest stress bond
        float stress = bond_flt_ptr[7 + bond*DATA_PER_BOND] ;
        if (stress>maxStress){
            maxStress = stress;
            maxStressBondIdx = bond;        // TODO chk vs null bonds or particles
        }
    }
    printf("\nassembleMuscleFibresOutGoing() chk2: i=%u   ",i  );
    if (maxStressBondIdx!=0){
        printf("\nassembleMuscleFibresOutGoing()  called for :  i=%u   ",i );
        // store high stress bond
        uint bytes = sizeof(uint)*DATA_PER_BOND;
        char temp[sizeof(uint)*DATA_PER_BOND /* DATA_PER_BOND*4 */] = {0};                       // NB sensitive to size of uint and float
        uint bondStep = maxStressBondIdx*DATA_PER_BOND;
        //memcpy(&bond_char_ptr[bondStep], temp, bytes);
        uint  currIdx           = bond_uint_ptr[0 + bondStep];  // could be done faster with a memcpy()  of bytes to a void or char pointer
        float elastLim          = bond_flt_ptr [1 + bondStep];  // NB existing memcpy causes data corruption, probable indexing error.
        float restLength        = bond_flt_ptr [2 + bondStep];
        float modulus           = bond_flt_ptr [3 + bondStep];
        float dampingCoeff      = bond_flt_ptr [4 + bondStep];
        uint  particleID        = bond_uint_ptr[5 + bondStep]; 
        uint  bondIndex         = bond_uint_ptr[6 + bondStep]; 
        float stressIntegrator  = bond_flt_ptr [7 + bondStep];
        uint  changeIndicator   = bond_uint_ptr[8 + bondStep];
        printf("\nassembleMuscleFibresOutGoing() chk1 called for :  i=%u   ",i );
        
        // move low stress bond
        //memcpy(bond_char_ptr, &bond_char_ptr[bondStep*sizeof(uint)], bytes);
        uint otherParticle          = bond_uint_ptr [0];
        uint otherParticleBondIDx   = bond_uint_ptr [6];
        if(otherParticle < fparam.maxPoints && otherParticleBondIDx < BONDS_PER_PARTICLE){
            bond_uint_ptr[0 + bondStep] = bond_uint_ptr[0] ;
            bond_flt_ptr [1 + bondStep] = bond_flt_ptr [1];
            bond_flt_ptr [2 + bondStep] = bond_flt_ptr [2];
            bond_flt_ptr [3 + bondStep] = fgenome.param[1][fgenome.default_modulus];  // change modulus elastic fibre type
            bond_flt_ptr [4 + bondStep] = bond_flt_ptr [4];
            bond_uint_ptr[5 + bondStep] = bond_uint_ptr[5]; 
            bond_uint_ptr[6 + bondStep] = bond_uint_ptr[6]; 
            bond_flt_ptr [7 + bondStep] = bond_flt_ptr [7];
            bond_uint_ptr[8 + bondStep] = bond_uint_ptr[8];

            // update reciprocal record
            printf("\nassembleMuscleFibresOutGoing() chk2 called for :  i=%u  otherParticle=%u  otherParticleBondIDx=%u ", i, otherParticle, otherParticleBondIDx );
            printf(".\n");//flush hopefully...
    
            fbuf.bufI(FELASTIDX)[otherParticle*BOND_DATA + otherParticleBondIDx*DATA_PER_BOND + 6]  = 0;  
        }
        printf("\nassembleMuscleFibresOutGoing() chk3 called for :  i=%u   ",i );    
        
        // write high stress bond
        //memcpy(temp, bond_char_ptr, bytes);
        bond_uint_ptr[0] = currIdx;
        bond_flt_ptr [1] = elastLim;
        bond_flt_ptr [2] = restLength;
        bond_flt_ptr [3] = fgenome.param[0][fgenome.collagen];   //modulus;    // change modulus collagen fibre type
        bond_flt_ptr [4] = dampingCoeff;
        bond_uint_ptr[5] = particleID; 
        bond_uint_ptr[6] = bondIndex; 
        bond_flt_ptr [7] = stressIntegrator;
        bond_uint_ptr[8] = changeIndicator;
        printf("\nassembleMuscleFibresOutGoing() chk4 called for :  i=%u   ",i );
        
        // update reciprocal record
        otherParticle          = bond_uint_ptr [0];
        otherParticleBondIDx   = bond_uint_ptr [6];
        if(otherParticle < fparam.maxPoints && otherParticleBondIDx < BONDS_PER_PARTICLE)
            fbuf.bufI(FELASTIDX)[otherParticle*BOND_DATA + otherParticleBondIDx*DATA_PER_BOND + 6]  = maxStressBondIdx;  
        
    printf("\nassembleMuscleFibresOutGoing() chk5 called for :  i=%u   ",i );
    }
    printf("\nassembleMuscleFibresOutGoing() chk3: i=%u   ",i  );
    
}

    
extern "C" __global__ void assembleMuscleFibresInComing ( int pnum, uint list, uint list_length ) // used for muscle, elastic ligg, and tendon
{
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                            // particle index
    if ( particle_index >= list_length )return;
    uint i = fbuf.bufII(FDENSE_LISTS)[list][particle_index];
    if (i > fparam.maxPoints) return;
    
    printf("\nassembleMuscleFibresInComing() chk1: i=%u   ",i  );
    
    ///////////// Swap outgoing bond indicies 
    
    // find highest stress incoming bond
    uint incomingParticleIdx      ;//  = fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE];
    uint incomingParticleBondIDx  ;//  = fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE +1];
    
    float maxStress         = 0.0;
    uint maxStressBondIdx   = 0;
    
    
    
    
    for(int bond=0; bond<BONDS_PER_PARTICLE; bond++){                                                                                   // find highest stress bond
        incomingParticleIdx        = fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE + bond*2];
        incomingParticleBondIDx    = fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE + bond*2 +1];
        
        
        if ( incomingParticleIdx < fparam.maxPoints  && incomingParticleBondIDx < BONDS_PER_PARTICLE ) {                                // chk vs null bonds or particles
            float stress = fbuf.bufF(FELASTIDX)[incomingParticleIdx*BOND_DATA +  incomingParticleBondIDx*DATA_PER_BOND + 7]; 
            if (stress>maxStress){
                maxStress = stress;
                maxStressBondIdx = bond; 
            }
        }
    }
    
    
    printf("\nassembleMuscleFibresInComing() chk4: i=%u  maxStressBondIdx=%u ",i ,maxStressBondIdx );
    if (maxStressBondIdx!=0 ){
        // store high stress bond
        incomingParticleIdx        = fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE + maxStressBondIdx*2];
        incomingParticleBondIDx    = fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE + maxStressBondIdx*2 +1];
        
        printf("\nassembleMuscleFibresInComing() chk4.1: i=%u  incomingParticleIdx=%u,  incomingParticleBondIDx=%u ",i ,incomingParticleIdx, incomingParticleBondIDx );
        
        uint lowStressIncomingParticleIdx          = fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE ];
        uint lowStressIncomingParticleBondIDx      = fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE +1];
        
        printf("\nassembleMuscleFibresInComing() chk4.2: i=%u  lowStressIncomingParticleIdx=%u,  lowStressIncomingParticleBondIDx=%u ",i ,lowStressIncomingParticleIdx, lowStressIncomingParticleBondIDx );
        
        // move low stress bond
        fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE + maxStressBondIdx*2]       =  lowStressIncomingParticleIdx;
        fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE + maxStressBondIdx*2 +1]    =  lowStressIncomingParticleBondIDx;
         
        // update reciprocal record
        if(lowStressIncomingParticleIdx<fparam.maxPoints && lowStressIncomingParticleBondIDx<BONDS_PER_PARTICLE){
            fbuf.bufF(FELASTIDX)[lowStressIncomingParticleIdx*BOND_DATA +  lowStressIncomingParticleBondIDx*DATA_PER_BOND + 6] =  maxStressBondIdx;
        }
        
        
        // write high stress bond
        //if(incomingParticleIdx>fparam.maxPoints || incomingParticleBondIDx>BONDS_PER_PARTICLE){
        //    incomingParticleIdx=UINT_MAX;
        //    incomingParticleBondIDx=UINT_MAX;
        //}
        fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE ]      =  incomingParticleIdx;
        fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE +1]    =  incomingParticleBondIDx;
        
       
        // update reciprocal record
        if(incomingParticleIdx<fparam.maxPoints && incomingParticleBondIDx<BONDS_PER_PARTICLE){
            fbuf.bufF(FELASTIDX)[incomingParticleIdx*BOND_DATA +  incomingParticleBondIDx*DATA_PER_BOND + 6] =  0;
        }
    }
    printf("\nassembleMuscleFibresInComing() chk5: i=%u   ",i  );
    
    /////////// connect contractile fibres (bond[1])  // replace bond[1], and leave other particles to heal.
    // if (tendon) return;
    // hipDeviceSynchronize(); // is this correct ?
    
    
    
    
    
    // if(muscle) connect nerves
    // if (elastic ligg) return;
    
    
    
    
    
}


extern "C" __global__ void assembleFasciaFibres ( int pnum, uint list, uint list_length ) // used for muscle and elastic ligg
{
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                            // particle index
    if ( particle_index >= list_length )return;
    uint i = fbuf.bufII(FDENSE_LISTS)[list][particle_index];
    if (i > fparam.maxPoints) return;
    
    float maxStress0         = 0.0;
    uint maxStressBondIdx0   = 0;
    float maxStress1         = 0.0;
    uint maxStressBondIdx1   = 0;
    float stress0 = 0.0, stress1 = 0.0;
    char* bond_char_ptr = &fbuf.bufC(FELASTIDX)[i*BOND_DATA];
    uint* bond_uint_ptr = &fbuf.bufI(FELASTIDX)[i*BOND_DATA];                                       //fbuf.bufI(FELASTIDX)[i*BOND_DATA + bond*DATA_PER_BOND +  ] ;
    float*bond_flt_ptr  = &fbuf.bufF(FELASTIDX)[i*BOND_DATA];                                       //FELASTIDX   [0]current index, [1]elastic limit, [2]restlength, [3]modulus,
                                                                                                                //[4]damping coeff, [5]particle ID,   [6]bond index, 
                                                                                                                //[7]stress integrator,  [8]change-type binary indicator
    for(int bond=0; bond<BONDS_PER_PARTICLE; bond++){                                               // find highest stress bond
        float stress = bond_flt_ptr[7 + bond*DATA_PER_BOND] ;
        if (stress>maxStress0){
            maxStress0 = stress;
            maxStressBondIdx0 = bond;
            
            
        }else if (stress>maxStress1){
            maxStress1 = stress;
            maxStressBondIdx1 = bond;
        
        }
        
    }
    //  if (maxStressBondIdx0==0) return;
    // Swap bond indicies 
    // store high stress bond
    uint bytes = sizeof(uint)*DATA_PER_BOND;
    char temp[DATA_PER_BOND*4] = {0};                       // NB sensitive to size of uint and float
    uint  bondStep = maxStressBondIdx0*DATA_PER_BOND;
    
    memcpy(&bond_char_ptr[bondStep], temp, bytes);
    /*
    uint  currIdx           = bond_uint_ptr[0 + bondStep];  // could be done faster with a memcpy()  of bytes to a void or char pointer
    float elastLim          = bond_flt_ptr [1 + bondStep];
    float restLength        = bond_flt_ptr [2 + bondStep];
    float modulus           = bond_flt_ptr [3 + bondStep];
    float dampingCoeff      = bond_flt_ptr [4 + bondStep];
    uint  particleID        = bond_uint_ptr[5 + bondStep]; 
    uint  bondIndex         = bond_uint_ptr[6 + bondStep]; 
    float stressIntegrator  = bond_flt_ptr [7 + bondStep];
    uint  changeIndicator   = bond_uint_ptr[8 + bondStep];
    */
    
    // move low stress bond
    memcpy(bond_char_ptr, &bond_char_ptr[bondStep*sizeof(uint)], bytes);
    /*
    bond_uint_ptr[0 + bondStep] = bond_uint_ptr[0] ;
    bond_flt_ptr [1 + bondStep] = bond_flt_ptr [1];
    bond_flt_ptr [2 + bondStep] = bond_flt_ptr [2];
    bond_flt_ptr [3 + bondStep] = bond_flt_ptr [3];  // change modulus elastic fibre type
    bond_flt_ptr [4 + bondStep] = bond_flt_ptr [4];
    bond_uint_ptr[5 + bondStep] = bond_uint_ptr[5]; 
    bond_uint_ptr[6 + bondStep] = bond_uint_ptr[6]; 
    bond_flt_ptr [7 + bondStep] = bond_flt_ptr [7];
    bond_uint_ptr[8 + bondStep] = bond_uint_ptr[8];
    */
//    bond_flt_ptr [3 + bondStep] =    ;  // change modulus elastic fibre type
    // update reciprocal record
    
    
    
    // write high stress bond
    memcpy(temp, bond_char_ptr, bytes);
    /*
    bond_uint_ptr[0] = currIdx;
    bond_flt_ptr [1] = elastLim;
    bond_flt_ptr [2] = restLength;
    bond_flt_ptr [3] = modulus;                 // change modulus collagen fibre type
    bond_flt_ptr [4] = dampingCoeff;
    bond_uint_ptr[5] = particleID; 
    bond_uint_ptr[6] = bondIndex; 
    bond_flt_ptr [7] = stressIntegrator;
    bond_uint_ptr[8] = changeIndicator;
    */
//    bond_flt_ptr [3] =  ;   // change modulus to  collagen fibre type. Leave elastlim and restlength to computeBondChanges().
    // update reciprocal record
    
    
    
    // again for 2nd fibre
    
}


extern "C" __global__ void computeBondChanges ( int pnum, uint list_length )// Given the action of the genes, compute the changes to particle properties & splitting/combining 
{                                                                                                   // Also "inserts changes" 
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                            // particle index
    if ( particle_index >= list_length ) {/*if (fparam.debug>2)printf("\tcomputeBondChanges:particle_index %u>= %u list_length.\t",particle_index, list_length);*/ return;}                                                    // pnum should be length of list.
    // ## TODO convert i to particle index _iff_ not called for all particles : use a special dense list for "living tissue", made at same time as gene lists
    uint i = fbuf.bufII(FDENSE_LISTS)[2][particle_index];                                           // call for dense list of living cells (gene'2'living/telomere (has genes))
    //if ( i >= pnum || i==0 ) {printf("\tcomputeBondChanges:i %u>=%u pnum\t",i,pnum);   return;} 
    
    if ( i >= pnum ) {printf("\tcomputeBondChanges:i %u>=%u pnum\t",i,pnum);   return;} 
    if ( i==0 ) {printf("\tcomputeBondChanges:i=%u,  pnum=%u, fparam.maxPoints=%u \t",i, pnum, fparam.maxPoints);}

    float * fbufFCONC = &fbuf.bufF(FCONC)[i*NUM_TF];
    //float * ftempFCONC = &ftemp.bufF(FCONC)[i*NUM_TF];
    uint  * fbufFEPIGEN = &fbuf.bufI(FEPIGEN)[i];   /*  *NUM_GENES  */                              // TODO FEPIGEN is a uint here. May need to pack binaries for spread & stop. See paper.
    //uint  * ftempFEPIGEN = &ftemp.bufI(FEPIGEN)[i*NUM_GENES];    // ## need to zero ftemp after counting sort full
    
    //for(int j=0; j<NUM_TF;j++)      fbufFCONC[j] += ftempFCONC[j];                                  // list of transcription factor conc for this particle
    //for(int j=0; j<NUM_GENES;j++) fbufFEPIGEN[j] += ftempFEPIGEN[j];                                // list of epigenetic activations for this particle 
                                                                                                    // NB modification were writtent to ftemp, now added to fbuf here.
    /*
    // read FCONC, FNERVEIDX, , FPRESS, FDENSITY

    // define FELASTIDX  14  // [0]current index, [1]elastic limit, [2]restlength, [3]modulus, [4]damping coeff, [5]particle ID, [6]bond index [7]stress integrator
    
    //(i)modify  FCONC, FNERVEIDX, FPRESS, FDENSITY, FMASS_RADIUS,  FELASTIDX [1]elastic limit, [2]restlength, [3]modulus, [4]damping coeff
    
    ///// NB this is the new "freeze", responsible for bond formation.
    // Read FCONC[0] "growth", FELASTIDX[all], FEPIGEN[2-10 & 14,15] 
    
    // Equation to modify spring parameters :   NB limit to modulus -> duplicate particles & fibres
    //(i) principal fibre (tendons)                 i.e. spring[0]                      
    //(ii) 1st 3 fibres (fibrocyte, elastocyte)     i.e. spring[0-2]
    //(iii) all fibres (cartilage & bone)           i.e. spring[0-BONDS_PER_PARTICLE]
    */
    uint* bond_uint_ptr = &fbuf.bufI(FELASTIDX)[i*BOND_DATA];                                       //fbuf.bufI(FELASTIDX)[i*BOND_DATA + bond*DATA_PER_BOND +  ] ;
    float*bond_flt_ptr  = &fbuf.bufF(FELASTIDX)[i*BOND_DATA];                                       //FELASTIDX   [0]current index, [1]elastic limit, [2]restlength, [3]modulus,
                                                                                                                //[4]damping coeff, [5]particle ID,   [6]bond index, 
                                                                                                                //[7]stress integrator,  [8]change-type binary indicator
    register int gridTot = fparam.gridTotal;
    /*
    // hold as a texture or similar. // part of fparams ? 
    // 3 materials for bonds - elastin, collagen, apatite - depend on (i)tissue type (ii) additional bonds 
    
    // -- special genes, for simulation efficiency
    // 0 active particles
    // 1 solid  (has springs)
    // 2 living/telomere (has genes)
    
    // -- behavioural celltypes
    // 3 fat
    // 4 nerves 
    // 5 fibrocyte
    // 6 tendon
    // 7 muscle
    // 8 cartilage
    // 9 bone
    // 10 elastic_lig
    
    // 11 fixed particles
    // 12 externally actuated 
    */
    uint bond_type[BONDS_PER_PARTICLE] = {0};                                                       //  0=elastin, 1=collagen, 2=apatite
    // calculate material type for bond
    for (int bond=0; bond<BONDS_PER_PARTICLE; bond++) bond_type[bond] = 2*(fbufFEPIGEN[9*fparam.maxPoints]>0/*bone*/);
    if (fbufFEPIGEN[6*fparam.maxPoints]>0/*tendon*/||fbufFEPIGEN[7*fparam.maxPoints]>0/*muscle*/||fbufFEPIGEN[10*fparam.maxPoints]>0/*elast lig*/) {bond_type[0] = 1; bond_type[3] = 1;}
    for (int bond=0; bond<BONDS_PER_PARTICLE; bond++) bond_type[bond] = 1*(fbufFEPIGEN[6*fparam.maxPoints]>0/*cartilage*/);
    
    //if (fparam.debug>2 && i%1000==0)printf(",%u,",i);
    
    for (uint bond=0; bond<BONDS_PER_PARTICLE;bond++, bond_uint_ptr+=DATA_PER_BOND, bond_flt_ptr+=DATA_PER_BOND ){
        if (bond_flt_ptr[2]>0){                                                                     // NB (rest_length==0) => bond broken, do not modify.
            float strain_integrator = bond_flt_ptr[7];
            float * param_ptr = fgenome.param[bond_type[bond]];
            float restln_multiplier   = (strain_integrator - param_ptr[fgenome.elongation_threshold]) * param_ptr[fgenome.elongation_factor];
            float strength_multiplier = (strain_integrator - param_ptr[fgenome.strength_threshold])   * param_ptr[fgenome.strengthening_factor];
            float integ_elong_thresh = (strain_integrator - param_ptr[fgenome.elongation_threshold]);
            float integ_stren_thresh = (strain_integrator - param_ptr[fgenome.strength_threshold]);
            
            bond_flt_ptr[2]/*rest length*/ +=  bond_flt_ptr[2] * (strain_integrator - param_ptr[fgenome.elongation_threshold]) * param_ptr[fgenome.elongation_factor];
            bond_flt_ptr[3]/*modulus*/     +=  bond_flt_ptr[3] * (strain_integrator - param_ptr[fgenome.strength_threshold])   * param_ptr[fgenome.strengthening_factor];
            /*
            if (fparam.debug>2 && fbuf.bufI(FPARTICLE_ID)[i]<10){
                //printf("\ncomputeBondChanges(): ParticleID=%u,  bond=%u, bond_type=%u, fbufFEPIGEN[9]=%2.2f, [6]=%2.2f, [7]=%2.2f, [10]=%2.2f,  rest_length=%f,  modulus=%f\t, strain_integrator=%f, elongation_threshold=%f,\t integ-elong_thresh=%f elongation_factor=%f, \t restln_multiplier=%f \t\t strength_threshold=%f, integ_stren_thresh=%f, strengthening_factor=%f, strength_multiplier=%f",
                //   fbuf.bufI(FPARTICLE_ID)[i], bond, bond_type[bond], fbufFEPIGEN[9], fbufFEPIGEN[6], fbufFEPIGEN[7], fbufFEPIGEN[10], bond_flt_ptr[2], bond_flt_ptr[3], 
                //       strain_integrator, param_ptr[fgenome.elongation_threshold], integ_elong_thresh, param_ptr[fgenome.elongation_factor], restln_multiplier, param_ptr[fgenome.strength_threshold], integ_stren_thresh, param_ptr[fgenome.strengthening_factor], strength_multiplier );
            }
            */
        }
        // "insert changes"
        uint * fbufFGRIDCNT_CHANGES = fbuf.bufI(FGRIDCNT_CHANGES);
        int m = 1 + ((bond==0)&&(fbufFEPIGEN[7*fparam.maxPoints]>0/*muscle*/||fbufFEPIGEN[10*fparam.maxPoints]>0));        
                                                                                                    // i.e. if (bond==0 && fbufFEPIGEN[7]>0/*muscle*/) m=2 else m=1;
                                                                                                    // NB two different lists for each change, for (muscle & elastic ligg  vs other tissues)
        bond_uint_ptr[8]=0;                                                                         // Need to zero the indicator.
        /*
        //if (fparam.debug>2 && i%1000==0)if(bond_flt_ptr[2]!=0.0)printf(",");  //("\tcomputeBondChanges:(bond_flt_ptr[2]!=0.0): =i%u \t", i);
        //if (fparam.debug>2 && i%1000==0)if(!(bond < 3 || fbufFEPIGEN[8]>0 ||  fbufFEPIGEN[9]>0))printf("'");   //("\tcomputeBondChanges:!(bond < 3 || fbufFEPIGEN[8]>0 ||  fbufFEPIGEN[9]>0): =i%u \t", i);   
        */
        // NB heal all bonds as if mesenchyme, then remodel later. This is needed to hold tissue together.
        if (bond_flt_ptr[2]==0.0 /*&& (bond < 3 || fbufFEPIGEN[8]>0 ||  fbufFEPIGEN[9]>0/_*cartilage OR bone*_/)*/  ){  // bond_flt_ptr[2]=restlength==0.0 => bond broken 
           /*   
            // && bond_uint_ptr[0]/_*other particle*_/<pnum/_*bond broken*_/
            //TODO what happens when bond broken vs never existed ?  NB information about direction of broken bond.
             if (fparam.debug>2 && i<10  &&  fbufFGRIDCNT_CHANGES[0*gridTot+fbuf.bufI(FGCELL)[i]]<10  )
                printf("\ncomputeBondChanges()1: i=,%u, particle_index=,%u,  bond_uint_ptr[8]=,%u, fbufFGRIDCNT_CHANGES=,%u, address=,%p, ",
                    i, particle_index , bond_uint_ptr[8], fbufFGRIDCNT_CHANGES[ 0*gridTot  + fbuf.bufI(FGCELL)[i] ],
                    &fbuf.bufII(FDENSE_LISTS)[2][particle_index]
                );
            
            //if (fparam.debug>2)printf(".");
            */
            atomicAdd ( &fbufFGRIDCNT_CHANGES[ 0*gridTot  + fbuf.bufI(FGCELL)[i] ], 1 );            //add to heal list //NB device-wide atomic
            bond_uint_ptr[8]+=1;                                                                    // FELASTIDX [8]change-type binary indicator NB accumulates all changes for this bond
            
            if (bond>BONDS_PER_PARTICLE)//(fbuf.bufI(FPARTICLE_ID)[i]<10) 
                printf("\nError :computeBondChanges:add to heal list: i=%u, ParticleID=%u, bond=%u, bond_uint_ptr[0]=%u, fbufFEPIGEN[8*fparam.maxPoints]=%u, fbufFEPIGEN[9*fparam.maxPoints]=%u "
                ,i,fbuf.bufI(FPARTICLE_ID)[i],bond,bond_uint_ptr[0],fbufFEPIGEN[8*fparam.maxPoints],fbufFEPIGEN[9*fparam.maxPoints]);
            /*
            if(fbufFGRIDCNT_CHANGES[0*gridTot+fbuf.bufI(FGCELL)[i]]<50  && i<10)
                printf("\ncomputeBondChanges()2: i=%u, particle_index=%u,  bond_uint_ptr[8]=%u, fbufFGRIDCNT_CHANGES=%u ",
                    i, particle_index , bond_uint_ptr[8], fbufFGRIDCNT_CHANGES[ 0*gridTot  + fbuf.bufI(FGCELL)[i] ]);
                
          //  if (fparam.debug>2 && i==0)printf("\ncomputeBondChanges()2: i==0, particle_index=%u,  bond_uint_ptr[8]=%u, fbufFGRIDCNT_CHANGES=%u ",
          //      particle_index , bond_uint_ptr[8], fbufFGRIDCNT_CHANGES[ 0*gridTot  + fbuf.bufI(FGCELL)[i] ])
          */
            break;                                                                                  // First, heal one bond per timestep. Remodel only after freeze.
        }else if(fparam.freeze==false){                                                                                      // prevents clash with heal.
            if (bond_flt_ptr[2]>fgenome.param[bond_type[bond]][fgenome.max_rest_length]) {  
                atomicAdd ( &fbufFGRIDCNT_CHANGES[ m*gridTot  + fbuf.bufI(FGCELL)[i] ], 1 );        //add to elongate list , store particleIdx & bond 
                bond_uint_ptr[8]+=2*m;
            }
            if (bond_flt_ptr[2]<fgenome.param[bond_type[bond]][fgenome.min_rest_length]) {  
                atomicAdd ( &fbufFGRIDCNT_CHANGES[ (2+m)*gridTot  + fbuf.bufI(FGCELL)[i] ], 1 );    //add to shorten list
                bond_uint_ptr[8]+=8*m;
            }
            if (bond_flt_ptr[3]>fgenome.param[bond_type[bond]][fgenome.max_modulus])     {  
                atomicAdd ( &fbufFGRIDCNT_CHANGES[ (4+m)*gridTot  + fbuf.bufI(FGCELL)[i] ], 1 );    //add to strengthen list 
                bond_uint_ptr[8]+=32*m;
            }
            if (bond_flt_ptr[3]<fgenome.param[bond_type[bond]][fgenome.min_modulus])     {  
                atomicAdd ( &fbufFGRIDCNT_CHANGES[ (6+m)*gridTot  + fbuf.bufI(FGCELL)[i] ], 1 );    //add to weaken list
                bond_uint_ptr[8]+=128*m;
            }
        }
        // bond_uint_ptr[8]+=2^n; is ELASTIDX for binary change indicator per bond. 
    }
}

//////   Particle modification kernels called together. Must make sure that they cannot clash. NB atomic operations. 
extern "C" __device__ void addParticle (uint parent_Idx, uint &new_particle_Idx)                    // Template for stregthening & lengthening kernels
{   
    //printf("\naddParticle()1:  parent_Idx=%u, new_particle_Idx=%u, fbuf.bufI(FPARTICLE_ID)[new_particle_Idx]=%u", parent_Idx, new_particle_Idx, fbuf.bufI(FPARTICLE_ID)[new_particle_Idx] );
    
    atomicCAS(&fbuf.bufI(FPARTICLE_ID)[new_particle_Idx /*_otherParticleBondIndex*/], UINT_MAX, parent_Idx);
    if(fbuf.bufI(FPARTICLE_ID)[new_particle_Idx]==parent_Idx){// TODO set a unique particle ID. 
    
    //int particle_Idx = atomicAdd(&fparam.pnumActive, 1);                              // fparam.pnumActive = mActivePoints from PrefixSumCellsCUDA, set in CountingSortFullCUDA
                                                                                      // NOT safe to use fbuf.bufI(FGRIDOFF)[fparam.gridTotal] as active particle count!
        //if (fparam.debug>2)printf("\naddParticle()2:  parent_Idx=%u, new_particle_Idx=%u", parent_Idx, new_particle_Idx);
    
    //if (particle_Idx >= 0  &&  particle_Idx < fparam.pnum) {
    //    new_particle_Idx                            = particle_Idx;
        fbuf.bufF3(FVEVAL)[new_particle_Idx]        = fbuf.bufF3(FVEVAL)[parent_Idx]; // NB could use average with next row. Prob not needed, because old bond is stretched.
        fbuf.bufF3(FVEL)[new_particle_Idx]          = fbuf.bufF3(FVEL)[parent_Idx];
        fbuf.bufF3(FFORCE)[new_particle_Idx]        = fbuf.bufF3(FFORCE)[parent_Idx];
        fbuf.bufI(FMASS_RADIUS)[new_particle_Idx]   = fbuf.bufI(FMASS_RADIUS)[parent_Idx];
        fbuf.bufI(FAGE)[new_particle_Idx]           = fparam.frame;
        fbuf.bufI(FCLR)[new_particle_Idx]           = fbuf.bufI(FCLR)[parent_Idx];
        fbuf.bufI(FNERVEIDX)[new_particle_Idx]      = fbuf.bufI(FNERVEIDX)[parent_Idx];
        
        //for (int tf=0;tf<NUM_TF;tf++)                   fbuf.bufF(FCONC)[new_particle_Idx*NUM_TF+tf]          = fbuf.bufF(FCONC)[parent_Idx*NUM_TF+tf];
        float* fbuf_conc  = &fbuf.bufF(FCONC)[new_particle_Idx * NUM_TF];
        float* fbuf_parent_conc = &fbuf.bufF(FCONC)[parent_Idx * NUM_TF];
        for (int a=0;a<NUM_TF;a++)     fbuf_conc[a] = fbuf_parent_conc[a]; 
        
        //for (int gene=0;gene<NUM_GENES;gene++)          fbuf.bufI(FEPIGEN)[new_particle_Idx*NUM_GENES+gene]   = fbuf.bufI(FEPIGEN)[parent_Idx*NUM_GENES+gene];
        uint* fbuf_epigen  = &fbuf.bufI(FEPIGEN)[new_particle_Idx];
        uint* fbuf_parent_epigen = &fbuf.bufI(FEPIGEN)[parent_Idx];
        for (int a=0;a<NUM_GENES;a++)  fbuf_epigen[fparam.maxPoints*a]  = fbuf_parent_epigen[fparam.maxPoints*a];
        
        //if (fparam.debug>2)printf("\naddParticle()3:  parent_Idx=%u, new_particle_Idx=%u, fbuf.bufI(FAGE)[new_particle_Idx]=%u,  fparam.maxPoints=%u, \"muscle\"=fbuf.bufI(FEPIGEN)[new_particle_Idx+7*fparam.maxPoints]=%u ", 
        //    parent_Idx, new_particle_Idx, fbuf.bufI(FAGE)[new_particle_Idx], fparam.maxPoints, fbuf.bufI(FEPIGEN)[new_particle_Idx+7*fparam.maxPoints]
        //);
        // TODO should FEPIGEN be float, int or uint?
    } else new_particle_Idx=UINT_MAX;               // else failed.
    __syncwarp;
    //if (fparam.debug>2)printf("\naddParticle()4:  parent_Idx=%u  ", parent_Idx);
    //__syncthreads;
}

extern "C" __device__ void removeParticle (uint particle_Idx)                                                       // Template for weakening & shortening kernels
{   //  active particle count : done automatically by insert_particles(..)
    //  sets values to null particle, => will be sorted to reserve section of particle list in next time step.
    if (fparam.debug>2)printf("\nremoveParticle() particle_Idx=%u \t",particle_Idx);
    fbuf.bufF3(FPOS)[particle_Idx]      = fparam.pboundmax;
    fbuf.bufF3(FVEVAL)[particle_Idx]    = make_float3(0,0,0);
    fbuf.bufF3(FVEL)[particle_Idx]      = make_float3(0,0,0);
    fbuf.bufF3(FFORCE)[particle_Idx]    = make_float3(0,0,0);
    for (int incomingBondIdx=0; incomingBondIdx<BONDS_PER_PARTICLE; incomingBondIdx++){                             // Remove reciprocal data for incoming bonds
        uint jIdx       = fbuf.bufI(FPARTICLEIDX)[particle_Idx*BONDS_PER_PARTICLE*2 + incomingBondIdx*2];
        uint bondIdx    = fbuf.bufI(FPARTICLEIDX)[particle_Idx*BONDS_PER_PARTICLE*2 + incomingBondIdx*2 +1];
        if(jIdx!=UINT_MAX){
            uint *ptr_elastidx =  &fbuf.bufI(FELASTIDX)[jIdx*BOND_DATA + bondIdx*DATA_PER_BOND];
            for (int j=0;j<DATA_PER_BOND;j++)  ptr_elastidx[j] = UINT_MAX;
        fbuf.bufI(FPARTICLEIDX)[particle_Idx*BONDS_PER_PARTICLE*2 + incomingBondIdx*2]      = UINT_MAX;
        fbuf.bufI(FPARTICLEIDX)[particle_Idx*BONDS_PER_PARTICLE*2 + incomingBondIdx*2 +1]   = UINT_MAX;
        }
    }
    for (int outgoingBondIdx=0; outgoingBondIdx<BONDS_PER_PARTICLE; outgoingBondIdx++){                             // Remove reciprocal data for outgoing bonds
        uint jIdx       = fbuf.bufI(FELASTIDX)[particle_Idx*DATA_PER_BOND + outgoingBondIdx*BONDS_PER_PARTICLE +0];
        uint bondIdx    = fbuf.bufI(FELASTIDX)[particle_Idx*DATA_PER_BOND + outgoingBondIdx*BONDS_PER_PARTICLE +6];
        if(jIdx!=UINT_MAX){
            fbuf.bufI(FPARTICLEIDX)[jIdx*BONDS_PER_PARTICLE*2 + bondIdx*2]      = UINT_MAX;
            fbuf.bufI(FPARTICLEIDX)[jIdx*BONDS_PER_PARTICLE*2 + bondIdx*2 +1]   = UINT_MAX;
        }
    }
    uint *ptr_elastidx =  &fbuf.bufI(FELASTIDX)[particle_Idx*BOND_DATA];                                            // Null FELASTIDX
    for (int j=0;j<BOND_DATA;j++)  ptr_elastidx[j] = UINT_MAX;
    
    uint *ptr_epigen = &fbuf.bufI(FEPIGEN)[particle_Idx];                                                 // Zero FEPIGEN
    for (int gene=0;gene<NUM_GENES;gene++)  ptr_epigen[gene*fparam.maxPoints]=0;
    
    float *ptr_tf = &fbuf.bufF(FCONC)[particle_Idx*NUM_TF];                                                         // Zero FCONC
    for (int tf=0;tf<NUM_TF;tf++) ptr_tf[tf]=0;
}

extern "C" __device__ void find_potential_bonds (int i, float3 ipos, int cell, uint _bonds[BONDS_PER_PARTICLE][2], float _bond_dsq[BONDS_PER_PARTICLE], float max_len_sq)
{                                                                                                           // Triangulated cubic bond selection...
	if ( fbuf.bufI(FGRIDCNT)[cell] == 0 ) return;                                                           // If the cell is empty, skip it.
	float dsq;//, sdist;//, c;
	float3 dist = make_float3(0,0,0), eterm  = make_float3(0,0,0), force = make_float3(0,0,0);
	uint j;
	int clast = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];                                      // index of last particle in this cell
    for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++ ) {                                    // For particles in this cell.
		j = fbuf.bufI(FGRID)[ cndx ];
		dist = ( ipos - fbuf.bufF3(FPOS)[ j ] );                                                            // dist in cm (Rama's comment)
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);                                              // scalar distance squared
		if ( dsq < max_len_sq && dsq > 0) {                                                                 // IF in-range && not the same particle
            //sdist = sqrt(dsq * fparam.d2);                                                                // smoothing distance = sqrt(dist^2 * sim_scale^2))
			//c = ( fparam.psmoothradius - sdist ); 
            bool known = false;
            uint bond_index = UINT_MAX;
            for (int a=0; a<BONDS_PER_PARTICLE; a++){                                                       // chk if known, i.e. already bonded 
                    if (fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + a*2] == i        ) known = true;   // particle 'j' has a bond to particle 'i'
                    if (fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + a*2] == UINT_MAX ) bond_index = a; // particle 'j' has an empty bond 'a' : picks last empty bond
                    if (_bonds[a][0] == j )known = true; // needed?                                         // particle 'i' already has a bond to particle 'j'  
                                                                                                            // not req?, _bonds starts empty && only touch 'j' once
            }
            if (known == false && bond_index<UINT_MAX){       
                    //int bond_direction = 1*(dist.x-dist.y+dist.z>0.0) + 2*(dist.x+dist.y-dist.z>0.0);     // booleans divide bond space into quadrants of x>0.
                    float approx_zero    = 0.02*fparam.rd2;
                    int   bond_direction = ((dist.x+dist.y+dist.z)>0) * (1*(dist.x*dist.x>approx_zero) + 2*(dist.y*dist.y>approx_zero) + 4*(dist.z*dist.z>approx_zero)) -1; 
                                                                                                            // booleans select +ve quadrant x,y,z axes and their planar diagonals
                    //if (fparam.debug>2)printf("\ni=%u, bond_direction=%i, dist=(%f,%f,%f), dsq=%f, approx_zero=%f", i, bond_direction, dist.x, dist.y, dist.z, dsq, approx_zero);
                    if(0<=bond_direction && bond_direction<BONDS_PER_PARTICLE && dsq<_bond_dsq[bond_direction]){ //if new candidate bond is shorter, for this quadrant. 
                                                                                                            //lacks a candidate bond _bonds[bond_direction][1]==0
                        _bonds[bond_direction][0] = j;                                                      // index of other particle
                        _bonds[bond_direction][1] = bond_index;                                             // FPARTICLEIDX vacancy index of other particle
                        _bond_dsq[bond_direction] = dsq;                                                    // scalar distance squared 
                    }
            }                                                                                               // end of collect potential bonds
        }                                                                                                   // end of: IF in-range && not the same particle
    }                                                                                                       // end of loop round particles in this cell
}


extern "C" __global__ void initialize_FCURAND_STATE (int pnum)  // designed to use to bootstrap itself. Set j=0 from host, call kernel repeatedly for 256^n threads, n=0-> to pnum threads.
{
    unsigned long long sequence=0, offset=1;//, seed=0;
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                 // particle index
    /*
    if(pnum==0 && i==0){ 
        seed = clock64();
        if (fparam.debug>2)printf("\ninitialize_FCURAND_STATE(): seed=%llu,\t  &fbuf.bufCuRNDST(FCURAND_STATE)[0]=%p .\t",seed,&fbuf.bufCuRNDST(FCURAND_STATE)[0]);  // getting (nil) a null pointer.
        hiprand_init(seed, sequence, offset, &fbuf.bufCuRNDST(FCURAND_STATE)[0]);
    }
    */
	if ( i >= pnum ) return;
    /*
    int j = i/256;
    // atomic lock, to ensure separate random numbers
    seed = hiprand(&fbuf.bufCuRNDST(FCURAND_STATE)[j]);
    
    seed = seed << 32; 
    seed += hiprand(&fbuf.bufCuRNDST(FCURAND_STATE)[j]);
    */
    hiprand_init(fbuf.bufI(FCURAND_SEED)[i], sequence, offset, &fbuf.bufCuRNDST(FCURAND_STATE)[i]);
    uint rnd_nm=hiprand(&fbuf.bufCuRNDST(FCURAND_STATE)[i]);
    
    //if (fparam.debug>2)printf("\n(i=%u,seed=%i, &fbuf.bufCuRNDST(FCURAND_STATE)[i]=%p, rnd_nm=%u),",i,fbuf.bufI(FCURAND_SEED)[i] , &fbuf.bufCuRNDST(FCURAND_STATE)[i], rnd_nm);
}


extern "C" __device__ void find_potential_bond (int i, float3 ipos, uint _thisParticleBonds[BONDS_PER_PARTICLE], float3 tpos, int gc, uint &_otherParticleIdx, uint &_otherParticleBondIdx, float &_bond_dsq, float max_len)                                                                                      // Used when just one bond, near a target location "tpos" is sought.
{
    int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
    gc -= nadj; 
    int cell;
    _bond_dsq=max_len*max_len;//FLT_MAX; Better to use breaking length.  (FELASTIDX)[1]elastic limit : depends on type of new bond. Could set _bond_dsq when calling find_potential_bond().
    float max_len_sq = max_len*max_len;
    uint rnd_nmbr = hiprand(&fbuf.bufCuRNDST(FCURAND_STATE)[i]);                                                 // NB bitshift and mask to get rand bool to choose bond
    /*
    float3 old_tpos=tpos;
    */
    tpos.x += max_len/float(4+(rnd_nmbr&7))     *(-1*float(1&(rnd_nmbr>>3))  );                                 // shift tpos by a random step < max_len, randomises bond.
    tpos.y += max_len/float(4+((rnd_nmbr>>4)&7))*(-1*float(1&(rnd_nmbr>>7))  );
    tpos.z += max_len/float(4+((rnd_nmbr>>8)&7))*(-1*float(1&(rnd_nmbr>>11)) );
/*
    //printf("\ni=%u, &fbuf.bufCuRNDST(FCURAND_STATE)[i]=%p, rnd_nmbr=%u, (1&(rnd_nmbr>>12)=%u, (-1*int(1&(rnd_nmbr>>12))=%d,  float(-1*int(rnd_nmbr&64)=%f  ",i, &fbuf.bufCuRNDST(FCURAND_STATE)[i], rnd_nmbr, 1&(rnd_nmbr>>12), -1*int(1&(rnd_nmbr>>12)), (-1*float(rnd_nmbr&64))  );  // (-1*(rnd_nmbr&64),
*/
/*
     printf("\nold_tpos=(%f,%f,%f), tpos=(%f,%f,%f), max_len=%f.\trnd_nmbr=%u, \trnd_nmbr&7=%u, \t(1&(rnd_nmbr>>3)=%u, \tfloat(1&(rnd_nmbr>>3))*2-1=%f, \t(4+((rnd_nmbr<<3)&7))*(float(1&(rnd_nmbr>>3))*2-1)=%f", 
           old_tpos.x,old_tpos.y,old_tpos.z, tpos.x,tpos.y,tpos.z, max_len, 
           rnd_nmbr, rnd_nmbr&7, 1&(rnd_nmbr>>3), float(1&(rnd_nmbr>>3))*2-1, float(4+(rnd_nmbr&7))*(float(1&(rnd_nmbr>>3))*2-1)
          );
*/
    for (int c=0; c < fparam.gridAdjCnt; c++) { 
        cell = gc + fparam.gridAdj[c];
        float dsq;
        float3 dist = make_float3(0,0,0);
        uint j;
        int clast = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];                                      // index of last particle in this cell
        for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++ ) {                                    // For particles in this cell.
            j = fbuf.bufI(FGRID)[ cndx ];
            dist = ( ipos - fbuf.bufF3(FPOS)[ j ] );                                                            // dist in cm (Rama's comment)
            dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);                                              // scalar distance squared
            if ( dsq < max_len_sq && dsq > 0) {  // probably wasteful, if tpos is in range.                     // IF in-range && not the same particle
                dist = ( tpos - fbuf.bufF3(FPOS)[ j ] );                                                        // dist to target location
                dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);                                          // scalar distance squared
                if(dsq<_bond_dsq){                                                                              // If closer to tpos than current candidate
                    bool known      = false;
                    uint bond_index = UINT_MAX;
                    for (int a=0; a<BONDS_PER_PARTICLE; a++){                                                   // chk if known, i.e. already bonded 
                        if (fbuf.bufI(FELASTIDX)[i*BOND_DATA*2 + a*DATA_PER_BOND] == j   ) known = true;
                        //if (fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + a*2] == i        ) known = true;   // particle 'j' has a bond to particle 'i'
                        /*if (fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + a*2] == UINT_MAX )*/ bond_index = a; // particle 'j' has an empty bond 'a' : picks last empty bond
                        if (_thisParticleBonds[a] == j )known = true;                                           // particle 'i' already has a bond to particle 'j'  
                    }
                    if (known == false && bond_index<UINT_MAX){
                        _otherParticleIdx = j;                                                                  // index of other particle
                        _otherParticleBondIdx = bond_index;                                                     // FPARTICLEIDX vacancy index of other particle
                        _bond_dsq = dsq;                                                                        // scalar distance squared 
                    }
                }                                                                                               // end of collect potential bonds
            }                                                                                                   // end of: IF in-range && not the same particle
        }                                                                                                       // end of loop round particles in this cell
    }
}


extern "C" __device__ void breakBond (uint thisParticleIdx, uint otherParticleIdx, uint bondIdx, uint otherParticleBondIdx){
    //FBondParams *params_ =  &fgenome.fbondparams[bondType];
    uint*   uint_ptr = &fbuf.bufI(FELASTIDX)[thisParticleIdx*BOND_DATA + bondIdx*DATA_PER_BOND];
    float* float_ptr = &fbuf.bufF(FELASTIDX)[thisParticleIdx*BOND_DATA + bondIdx*DATA_PER_BOND];
    uint_ptr [0]    = UINT_MAX;                                                                             //[0]current index, 
    float_ptr[1]    = 0.0;                                                                                  //[1]elastic limit, 
    float_ptr[2]    = 0.0;                                                                                  //[2]restlength, 
    float_ptr[3]    = 0.0;                                                                                  //[3]modulus, 
    float_ptr[4]    = 0.0;                                                                                  //[4]damping coeff, 
    uint_ptr [5]    = UINT_MAX;                                                                             //[5]particle ID,   
    uint_ptr [6]    = UINT_MAX;                                                                             //[6]bond index 
    float_ptr[7]    = 0;                                                                                    //[7]stress integrator 
    uint_ptr [8]    = 0;                                                                                    //[8]change-type 
                                                                                                            // Connect new particle incoming bonds
    fbuf.bufI(FPARTICLEIDX)[otherParticleIdx*2*BONDS_PER_PARTICLE + otherParticleBondIdx*2]       = UINT_MAX;                  // particle Idx
    fbuf.bufI(FPARTICLEIDX)[otherParticleIdx*2*BONDS_PER_PARTICLE + otherParticleBondIdx*2 +1]    = UINT_MAX;                  // bond Idx 
    
    /*if(thisParticleIdx<20) printf("\nmakeBond: bondtype=%u, default_rest_length=%f, %f, thisParticleIdx=%u, otherParticleIdx=%u, otherParticleBondIdx=%u  \t", 
           bondType, fgenome.param[bondType][fgenome.default_rest_length],  fbuf.bufF(FELASTIDX)[thisParticleIdx*BOND_DATA + bondIdx*DATA_PER_BOND +2], thisParticleIdx, otherParticleIdx, otherParticleBondIdx );
    */
}

extern "C" __device__ void makeBond (uint thisParticleIdx, uint otherParticleIdx, uint bondIdx, uint otherParticleBondIdx, uint bondType /* elastin, collagen, apatite */){
    //FBondParams *params_ =  &fgenome.fbondparams[bondType];
    uint*   uint_ptr = &fbuf.bufI(FELASTIDX)[thisParticleIdx*BOND_DATA + bondIdx*DATA_PER_BOND];
    float* float_ptr = &fbuf.bufF(FELASTIDX)[thisParticleIdx*BOND_DATA + bondIdx*DATA_PER_BOND];
    uint_ptr [0]    = otherParticleIdx;                                                                     //[0]current index, 
    float_ptr[1]    = fgenome.param[bondType][fgenome.elastLim];                                            //[1]elastic limit, 
    float_ptr[2]    = fgenome.param[bondType][fgenome.default_rest_length];                                 //[2]restlength, 
    float_ptr[3]    = fgenome.param[bondType][fgenome.default_modulus];                                     //[3]modulus, 
    float_ptr[4]    = fgenome.param[bondType][fgenome.default_damping];                                     //[4]damping coeff, 
    uint_ptr [5]    = fbuf.bufI(FPARTICLE_ID)[otherParticleIdx];                                            //[5]particle ID,   
    uint_ptr [6]    = otherParticleBondIdx;                                                                 //[6]bond index 
    float_ptr[7]    = 0;                                                                                    //[7]stress integrator 
    uint_ptr [8]    = 0;                                                                                    //[8]change-type 
                                                                                                            // Connect new particle incoming bonds
    fbuf.bufI(FPARTICLEIDX)[otherParticleIdx*2*BONDS_PER_PARTICLE + otherParticleBondIdx*2]       = thisParticleIdx;           // particle Idx
    fbuf.bufI(FPARTICLEIDX)[otherParticleIdx*2*BONDS_PER_PARTICLE + otherParticleBondIdx*2 +1]    = bondIdx;                   // bond Idx 
    
    /*if(thisParticleIdx<20) printf("\nmakeBond: bondtype=%u, default_rest_length=%f, %f, thisParticleIdx=%u, otherParticleIdx=%u, otherParticleBondIdx=%u  \t", 
           bondType, fgenome.param[bondType][fgenome.default_rest_length],  fbuf.bufF(FELASTIDX)[thisParticleIdx*BOND_DATA + bondIdx*DATA_PER_BOND +2], thisParticleIdx, otherParticleIdx, otherParticleBondIdx );
    */
}


extern "C" __device__ int atomicMakeBond(uint thisParticleIndx,  uint otherParticleIdx, uint bondIdx, uint otherParticleBondIndex, uint bond_type){
    int _otherParticleBondIndex = otherParticleIdx*2*BONDS_PER_PARTICLE + otherParticleBondIndex*2;//BONDS_PER_PARTICLE*2*otherParticleIdx + 2*bondIdx;
{// debug
    //printf("\natomicMakeBond1: ftemp.bufI(FPARTICLEIDX)[_otherParticleBondIndex]=%u \t",ftemp.bufI(FPARTICLEIDX)[_otherParticleBondIndex]);
    //do {} while( atomicCAS(&fbuf.bufI(FPARTICLEIDX)[_otherParticleBondIndex], UINT_MAX, thisParticleIndx) );                                               // lock ////// ###### //  if (not locked) write zero to 'ftemp' to lock.
    
    //printf("\natomicMakeBond2: ftemp.bufI(FPARTICLEIDX)[_otherParticleBondIndex]=%u, \tfbuf.bufI(FPARTICLEIDX)[_otherParticleBondIndex]=%u \t"
    //    ,ftemp.bufI(FPARTICLEIDX)[_otherParticleBondIndex], fbuf.bufI(FPARTICLEIDX)[_otherParticleBondIndex]);
    
    /*if (fbuf.bufI(FPARTICLEIDX)[_otherParticleBondIndex]==UINT_MAX)*/  
    //fbuf.bufI(FPARTICLEIDX)[_otherParticleBondIndex]  = thisParticleIndx;                                   //  if (bond is unoccupied) write to 'fbuf' to assign this bond
    //ftemp.bufI(FPARTICLEIDX)[_otherParticleBondIndex] = UINT_MAX;                                                                            // release lock // ######
}
    atomicCAS(&fbuf.bufI(FPARTICLEIDX)[_otherParticleBondIndex], UINT_MAX, thisParticleIndx);
    if (fbuf.bufI(FPARTICLEIDX)[_otherParticleBondIndex] == thisParticleIndx){                                                               // if (this bond is assigned) write bond data
        makeBond ( thisParticleIndx, otherParticleIdx /*candidate_target_pIDx*/, bondIdx, otherParticleBondIndex, bond_type);
        return 0;
    }else return 1;
}

extern "C" __device__ int findBondAxis(float3 pos, uint j ){
    float3 dist     = ( pos - fbuf.bufF3(FPOS)[ j ] );                                                             // dist in cm (Rama's comment)                                   
    float distxsq   = dist.x*dist.x, distysq=dist.y*dist.y, distzsq=dist.z*dist.z;
    float dsq       = distxsq + distysq + distzsq;                                                                  // scalar distance squared
            
            //printf("\n(B:particle=%u,j=%u,dist=(%f,%f,%f),dsq=%f),", particle, j, dist.x, dist.y, dist.z, dsq );
    int axis =  1*(distxsq>distysq && distxsq>distzsq) + 2*(distysq>=distxsq && distysq>distzsq) +3*(distzsq>=distxsq && distzsq>=distysq);   
    if ((axis==1 && dist.x>0.0) || (axis==2 && dist.y>0.0) || (axis==3 && dist.z>0.0)) axis +=2; else axis--;    // sort by longest axis +/-ve 
    
    return axis;
}


extern "C" __device__ void find_closest_particle_per_axis(uint particle, float3 pos, uint neighbours[6]){       // Used by "insertNewParticle()"
    // identify which bin to search  NB particle is new => not yet inserted into a cell
	register float3 gridMin   =	fparam.gridMin;                 // "register" is a compiler 'hint', to keep this variable in thread register
	register float3 gridDelta = fparam.gridDelta;               //  even if other variable have to be moved to slower 'local' memory  
	register int3   gridRes   =	fparam.gridRes;                 //  in the streaming multiprocessor's cache.
    int		gs;
	float3	gcf;
	int3	gc;
	gcf = (pos - gridMin) * gridDelta;                          // finds bin as a float3
	gc  = make_int3( int(gcf.x), int(gcf.y), int(gcf.z) );      // crops to an int3
	gs  = (gc.y * gridRes.z + gc.z)*gridRes.x + gc.x;           // linearizes to an int for a 1D array of bins
	
	float neighbours_dsq[6] = {FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX};
    
    //printf("\n\n neighbours_dsq=(%f,%f,%f,%f,%f,%f)  \n", neighbours_dsq[0], neighbours_dsq[1], neighbours_dsq[2], neighbours_dsq[3], neighbours_dsq[4], neighbours_dsq[5]  );
	
	for (int c=0; c < fparam.gridAdjCnt; c++) {                                                                 // For adjacent cells
        uint cell = gs + fparam.gridAdj[c];
        if ( fbuf.bufI(FGRIDCNT)[cell] == 0 ) continue;                                                         // If the cell is empty, skip it.
        //printf(" (A:particle=%u,c=%u,cell=%u),",particle, c, cell);
        float dsq = FLT_MAX;
        float3 dist = make_float3(0,0,0);
        uint j = UINT_MAX;
        int clast = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];                                      // index of last particle in this cell
        
        for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++ ) {                                    // For particles in this cell.
            j = fbuf.bufI(FGRID)[ cndx ];
            if (j==particle)continue;
            
            dist = ( pos - fbuf.bufF3(FPOS)[ j ] );                                                             // dist in cm (Rama's comment)                                   
            float distxsq=dist.x*dist.x, distysq=dist.y*dist.y, distzsq=dist.z*dist.z;
            dsq = distxsq + distysq + distzsq;                                                                  // scalar distance squared
            
            //printf("\n(B:particle=%u,j=%u,dist=(%f,%f,%f),dsq=%f),", particle, j, dist.x, dist.y, dist.z, dsq );
            
            int axis =  1*(distxsq>distysq && distxsq>distzsq) + 2*(distysq>=distxsq && distysq>distzsq) +3*(distzsq>=distxsq && distzsq>=distysq);
            
            if ((axis==1 && dist.x>0.0) || (axis==2 && dist.y>0.0) || (axis==3 && dist.z>0.0)) axis +=2; else axis--;    // sort by longest axis +/-ve 
            
            //printf("\n(C:particle=%u, j=%u, axis=%i, dsq=%f, neighbours_dsq[axis]=%f ),",particle,j,axis,dsq,neighbours_dsq[axis] );
            if ( dsq>0 && dsq < neighbours_dsq[axis]) {                                                                  // IF in-range && not the same particle
                neighbours_dsq[axis] = dsq;
                neighbours[axis] = j;
                //printf("\n\n(D:particle=%u,dsq=%f,j=%u)\n",particle,dsq,j );
            }                                                                                                   // end of: IF in-range && not the same particle
        }                                                                                                       // end of loop round particles in this cell
    }
    //printf("\nfind_closest_particle_per_axis: particle=%u, pos=(%f,%f,%f), neighbours=(%u, %u, %u, %u, %u, %u), neighbours_dsq=(%f, %f, %f, %f, %f, %f)  ", 
    //       particle, pos.x,pos.y,pos.z, neighbours[0], neighbours[2], neighbours[2], neighbours[3], neighbours[4], neighbours[5], neighbours_dsq[0], neighbours_dsq[1], neighbours_dsq[2], neighbours_dsq[3], neighbours_dsq[4], neighbours_dsq[5]   );
}


extern "C" __device__ void find_bonds_to_redistribute(uint new_particle_Idx, float3 newParticlePos, uint neighbours[6], uint neighboursBondIdx[6], uint neighbours2[6]){
    float neighbours_dsq[6] = {FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX};                                                                        // Used by "insertNewParticle()"
    
    //if (fparam.debug>2 /*&& (threadIdx.x==0 || particle_index==0)*/ ) printf("\nfind_bonds_to_redistribute()1.0:  new_particle_Idx=%u, neighbours=(%u,%u,%u,%u,%u,%u) ", 
    //    new_particle_Idx, neighbours[0], neighbours[1], neighbours[2], neighbours[3], neighbours[4], neighbours[5]);
    
    for (int neighbour=0; neighbour<6;neighbour++){
        for (int bond =0; bond<BONDS_PER_PARTICLE; bond++){
            
            //printf("\nfind_bonds_to_redistribute()1.1:  neighbours[neighbour]=%u",neighbours[neighbour]);
            if(neighbours[neighbour]>fparam.maxPoints) continue;// not a valid particle
            uint otherParticle = fbuf.bufI(FELASTIDX)[neighbours[neighbour]*BOND_DATA + bond*DATA_PER_BOND];
            
            //if (fparam.debug>2 /*&& (threadIdx.x==0 || particle_index==0)*/ ) printf("\nfind_bonds_to_redistribute()1.2:  otherParticle=%u",otherParticle);
            if (otherParticle>fparam.maxPoints) continue;
            int chk =0;
            for (; chk<6; chk++) if (otherParticle==neighbours[chk] || otherParticle==neighbours2[chk]) chk =7; // not one of neighbours[6] or neighbours2[6]
            if (chk==7) continue;
            float3 dist = fbuf.bufF3(FPOS)[otherParticle] - newParticlePos ;
            
            float dsq = dist.x*dist.x+dist.y*dist.y+dist.z*dist.z;
            //if (fparam.debug>2  ) printf("\nfind_bonds_to_redistribute()1.3: otherParticle=%u, dsq=%f, neighbours_dsq[neighbour]=%f, neighbour=%u ", 
            //    otherParticle, dsq, neighbours_dsq[neighbour], neighbour);
            
            if (dsq < neighbours_dsq[neighbour]){
                neighbours_dsq[neighbour] = dsq;
                neighbours2[neighbour] = otherParticle;
                neighboursBondIdx[neighbour] = bond;
            }
        }
    }
    //if (fparam.debug>2 /*&& (threadIdx.x==0 || particle_index==0)*/ ) printf("\nfind_bonds_to_redistribute()1.4:  new_particle_Idx=%u,  neighbours2=(%u,%u,%u,%u,%u,%u)", 
    //    new_particle_Idx, neighbours2[0], neighbours2[1], neighbours2[2], neighbours2[3], neighbours2[4], neighbours2[5]);
}

extern "C" __device__ void makeBondIndxMap( uint parentParticleIndx, int bondInxMap[6]){ // A tractable way to approximately map rotation of bonds wrt the world frame.
    uint bond0otherPartlicleIdx = fbuf.bufI(FELASTIDX)[parentParticleIndx*BOND_DATA];                           // Used by "insertNewParticle()"
    uint bond1otherPartlicleIdx = fbuf.bufI(FELASTIDX)[parentParticleIndx*BOND_DATA+DATA_PER_BOND];
    uint bond2otherPartlicleIdx = fbuf.bufI(FELASTIDX)[parentParticleIndx*BOND_DATA+2*DATA_PER_BOND];
    float3 pos      = fbuf.bufF3(FPOS)[parentParticleIndx]; 
    float3 bond0    = fbuf.bufF3(FPOS)[bond0otherPartlicleIdx] - pos;
    float3 bond1    = fbuf.bufF3(FPOS)[bond1otherPartlicleIdx] - pos;
    float3 bond2    = fbuf.bufF3(FPOS)[bond2otherPartlicleIdx] - pos;
/*
    // int axis =  1*(distxsq<distysq && distxsq<distzsq) + 2*(distysq<=distxsq && distysq<distzsq) +3*(distzsq<=distxsq && distzsq<=distysq);
    // if ((axis==1 && dist.x) || (axis==2 && dist.y) || (axis==3 && dist.z)) axis +=2; else axis--;       // sort by longest axis +/-ve 
*/
    float distxsq=bond0.x*bond0.x,  distysq=bond0.y*bond0.y,  distzsq=bond0.z*bond0.z;
    float dsq = distxsq + distysq + distzsq;         
    int axis0 = 1*(distxsq>distysq && distxsq>distzsq) + 2*(distysq>=distxsq && distysq>distzsq) +3*(distzsq>=distxsq && distzsq>=distysq);
    
    distxsq   = bond1.x*bond1.x*(axis0!=1), distysq=bond1.y*bond1.y*(axis0!=2), distzsq=bond1.z*bond1.z*(axis0!=3);
    int axis1 = 1*(distxsq>distysq && distxsq>distzsq) + 2*(distysq>=distxsq && distysq>distzsq) +3*(distzsq>=distxsq && distzsq>=distysq);
    
    int axis2 = 1*((axis0!=1)&&(axis1!=1)) + 2*((axis0!=2)&&(axis1!=2)) + 3*((axis0!=3)&&(axis1!=3));
    
    if ((axis0==1 && bond0.x) || (axis0==2 && bond0.y) || (axis0==3 && bond0.z)){
        bondInxMap[0] = axis0 +2;
        bondInxMap[3] = axis0 -1;
    }else{
        bondInxMap[0] = axis0 -1;
        bondInxMap[3] = axis0 +2;
    }
    
    if ((axis1==1 && bond1.x) || (axis1==2 && bond1.y) || (axis1==3 && bond1.z)){
        bondInxMap[1] = axis1 +2;
        bondInxMap[4] = axis1 -1;
    }else{
        bondInxMap[1] = axis1 -1;
        bondInxMap[4] = axis1 +2;
    }
    
    if ((axis2==1 && bond2.x) || (axis2==2 && bond2.y) || (axis2==3 && bond2.z)){
        bondInxMap[2] = axis0 +2;
        bondInxMap[5] = axis0 -1;
    }else{
        bondInxMap[2] = axis0 -1;
        bondInxMap[5] = axis0 +2;
    }
    printf("\nmakeBondIndxMap: parentParticleIndx=%u, bondInxMap=(%i,%i,%i,%i,%i,%i) ", parentParticleIndx, bondInxMap[0], bondInxMap[1], bondInxMap[2], bondInxMap[3], bondInxMap[4], bondInxMap[5]   );
}


extern "C" __device__ void redistribute_bonds(uint new_particle_Idx, float3 newParticlePos, uint neighbours[6], uint neighboursBondIdx[6], uint neighbours2[6]){
    // for particle removal, given list of bonds ... 
    // for each bond 
    
    
}


extern "C" __device__ int insertNewParticle(uint new_particle_Idx, float3 newParticlePos, uint parentParticleIndx, uint bondIdx, uint secondParticleIdx, uint otherParticleBondIndex, uint bond_type[BONDS_PER_PARTICLE]){
    printf ("\ninsertNewParticle1: new_particle_Idx=%u,", new_particle_Idx);                                    // Inserts particle at newParticlePos AND redistributes bonds with neighbours.
  //  addParticle(parentParticleIndx, new_particle_Idx);                                                         // Used by lengthen_tissue(), also for strengthen_tissue(), & muscle...
    
    // cut the old bond here 
    //breakBond (uint thisParticleIdx, uint otherParticleIdx, uint bondIdx, uint otherParticleBondIdx)
    breakBond(parentParticleIndx, secondParticleIdx, bondIdx, otherParticleBondIndex);   
    // may still need to be atomic, and close to original readng, to avoid alteration by oter threads.
    
    // //makeBond (uint thisParticleIdx, uint otherParticleIdx, uint bondIdx, uint otherParticleBondIdx, uint bondType /* elastin, collagen, apatite */)
    if(new_particle_Idx>fparam.maxPoints||secondParticleIdx>fparam.maxPoints||bondIdx>BONDS_PER_PARTICLE||otherParticleBondIndex>BONDS_PER_PARTICLE)return 1;
    makeBond (parentParticleIndx, new_particle_Idx, bondIdx, otherParticleBondIndex*2*BONDS_PER_PARTICLE +0 /*+ otherParticleBondIndex*2 */, bond_type[bondIdx] );
    makeBond (new_particle_Idx, secondParticleIdx, 0/*bondIdx*/, otherParticleBondIndex*2*BONDS_PER_PARTICLE + otherParticleBondIndex*2, bond_type[bondIdx] );
    // NB These two lines(above) replace the parent bond, IFF all indices are valid.
    
    fbuf.bufF3(FPOS)[new_particle_Idx] = newParticlePos;
    uint neighbours[6]          = {UINT_MAX,UINT_MAX,UINT_MAX,UINT_MAX,UINT_MAX,UINT_MAX}, 
         neighboursBondIdx[6]   = {UINT_MAX,UINT_MAX,UINT_MAX,UINT_MAX,UINT_MAX,UINT_MAX}, 
         neighbours2[6]         = {UINT_MAX,UINT_MAX,UINT_MAX,UINT_MAX,UINT_MAX,UINT_MAX};
    
    find_closest_particle_per_axis(new_particle_Idx, newParticlePos, neighbours);
    //find_bonds_to_redistribute(new_particle_Idx, newParticlePos, neighbours, neighboursBondIdx, neighbours2);
    
    //neighbours[bondIdx]= parentParticleIndx;
    //neighboursBondIdx[bondIdx] = bondIdx;
    //neighbours2[otherParticleBondIndex] = otherParticleBondIndex;
    
    int ret1=0, ret2=0, ret3=0;
    int bondInxMap[6]={0,1,2,3,4,5};// no change map    // map parent particle orientation // UINT_MAX,UINT_MAX,UINT_MAX,UINT_MAX,UINT_MAX,UINT_MAX
    //makeBondIndxMap( parentParticleIndx, bondInxMap);
    
    // need to find the axis of the inherited bonds & swap in bondInxMap
    uint InheritedBondAxis = findBondAxis(newParticlePos, secondParticleIdx); 
    bondInxMap[InheritedBondAxis] = 0; // NB inherited bond is on bondIndex 0.
    
    
    printf("\ninsertNewParticle1.1: new_particle_Idx=%u, \tneighbours[]=(%u,%u,%u,%u,%u,%u), \tneighboursBondIdx[]=(%u,%u,%u,%u,%u,%u), \tneighbours2[]=(%u,%u,%u,%u,%u,%u), \tbondInxMap[]=(%u,%u,%u,%u,%u,%u) ",
       new_particle_Idx,
       neighbours[0],neighbours[1],neighbours[2],neighbours[3],neighbours[4],neighbours[5],  neighboursBondIdx[1],neighboursBondIdx[2],neighboursBondIdx[2],neighboursBondIdx[3],neighboursBondIdx[4],neighboursBondIdx[5],
       neighbours2[0],neighbours2[1],neighbours2[2],neighbours2[3],neighbours2[4],neighbours2[5],
       bondInxMap[0],bondInxMap[1],bondInxMap[2],bondInxMap[3],bondInxMap[4],bondInxMap[5]
    );
                                                                            // ? how to insert the bond being lengthened or strengthened ? 
                                                                            // should occur implicitly due to orientation & placement wrt parent particle.
    for (int bond=1; bond<6; bond++){
        if (neighboursBondIdx[bondInxMap[bond]]<BONDS_PER_PARTICLE 
            && neighbours[bondInxMap[bond]]<fparam.maxPoints){
            //atomicMakeBond(neighbours[bondInxMap[bond]],  new_particle_Idx, neighboursBondIdx[bondInxMap[bond]], bond, bond_type[bond]); 
                                                                                                                // does not need to be atomic
            //atomicMakeBond(uint thisParticleIndx,  uint otherParticleIdx, uint bondIdx, uint otherParticleBondIndex, uint bond_type)
            int _otherParticleBondIndex = new_particle_Idx*2*BONDS_PER_PARTICLE + new_particle_Idx*2;
            makeBond (neighbours[bondInxMap[bond]],  _otherParticleBondIndex, neighboursBondIdx[bondInxMap[bond]], bond, bond_type[bond]);
        }
    }
    
    /*
    for (int bond=1/_*0*_/; bond<6; bond++){
        if (neighboursBondIdx[bondInxMap[bond]]<BONDS_PER_PARTICLE 
            && neighbours[bondInxMap[bond]]<fparam.maxPoints
            && neighbours2[bondInxMap[bond]]<fparam.maxPoints){                                            // suitable bond to redistribute was found)
            ret1 = atomicMakeBond(neighbours[bondInxMap[bond]],  new_particle_Idx, neighboursBondIdx[bondInxMap[bond]], bond, bond_type[bond]);   // new outging bond 
            if (ret1 == 0){                                                                                     // NB ret == 0 : success
                int _otherParticleBondIndex = neighbours2[bondInxMap[bond]]*2*BONDS_PER_PARTICLE + otherParticleBondIndex*2;
                makeBond ( new_particle_Idx, neighbours2[bondInxMap[bond]], bond, _otherParticleBondIndex, bond_type[bond]);
                printf("\ninsertNewParticle2:  new_particle_Idx=%u, neighbours2[bondInxMap[bond]]=%u, bond=%u, _otherParticleBondIndex=%u, bond_type[bond]=%u   ",
                    new_particle_Idx, neighbours2[bondInxMap[bond]], bond, _otherParticleBondIndex, bond_type[bond] );

            }
            if (ret1 || ret2) ret3++;
        }else ret3++;
        printf ("\ninsertNewParticle2.1: bond=%u, new_particle_Idx=%u ret1=%i,ret2=%i,ret3=%i",bond,new_particle_Idx,ret1,ret2,ret3);
    }
    */
    
    printf ("\ninsertNewParticle3: new_particle_Idx=%u, ,ret3=%i", new_particle_Idx, ret3);
    return ret3;                                                                                                //NB causes incoming bonds to fluid particles -> non-adherent surface.
}

extern "C" __global__ void cleanBonds (int pnum){                                   // Called by CleanBondsCUDA (); for use after ComputeParticleChangesCUDA (); Only in Run(), not Run(...)?
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                         // particle index
    if ( i >= pnum ) return;
    uint gc = fbuf.bufI(FGCELL)[ i ];                                               // Get search cell	
    if ( gc == GRID_UNDEF ) return;                                                 // particle out-of-range

    gc -= (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
    /*
    register float3 force, eterm, dist;                                             // request to compiler to store in a register for speed.
    force = make_float3(0,0,0);    eterm = make_float3(0,0,0);     dist  = make_float3(0,0,0);
    
    float dsq, abs_dist;                                                            // elastic force // new version computes here using particle index rather than ID.
    uint bondsToFill = 0;
    uint bonds[BONDS_PER_PARTICLE][2];                                              // [0] = index of other particle, [1] = bond_index
    float bond_dsq[BONDS_PER_PARTICLE];                                             // length of bond, for potential new bonds
    for (int a=0; a<BONDS_PER_PARTICLE;a++) {
        bonds[a][0]= UINT_MAX;
        bonds[a][1]= UINT_MAX;
        bond_dsq[a]= fparam.rd2;                                                    // NB if ( dsq < fparam.rd2 && dsq > 0) is the cut off for fluid interaction range
    } 
    */                                                                              // Check for broken incomming bonds //////////////////
    for (int a=0; a<BONDS_PER_PARTICLE;a++){                                        // loop round this particle's list of _incomming_ bonds /////
        bool intact = false;
        uint k = fbuf.bufI(FPARTICLEIDX)[i*BONDS_PER_PARTICLE*2 + a*2];
        uint b = fbuf.bufI(FPARTICLEIDX)[i*BONDS_PER_PARTICLE*2 + a*2 +1];          // chk bond intact. nb short circuit evaluation of if conditions.
        // k is a particle, bond_idx is in range, AND k's reciprocal record matches i's record of the bond
        if(k<pnum && b<BONDS_PER_PARTICLE 
            && i==fbuf.bufI(FELASTIDX)[k*BOND_DATA + b*DATA_PER_BOND] 
            && a==fbuf.bufI(FELASTIDX)[k*BOND_DATA + b*DATA_PER_BOND +6] 
            && 0.0<fbuf.bufF(FELASTIDX)[k*BOND_DATA + b*DATA_PER_BOND +2])  intact=true;   
        if(i==k)intact=false;
        if(k<pnum && i!=fbuf.bufI(FELASTIDX)[k*BOND_DATA + b*DATA_PER_BOND] ) printf("\ncleanBonds1: incomming bond not intact : i!=fbuf.bufI(FELASTIDX)[k*BOND_DATA + b*DATA_PER_BOND] ");
        if(k<pnum && a!=fbuf.bufI(FELASTIDX)[k*BOND_DATA + b*DATA_PER_BOND +6]  ) printf("\ncleanBonds1: incomming bond not intact : a!=fbuf.bufI(FELASTIDX)[k*BOND_DATA + b*DATA_PER_BOND +6] ");
        if(k<pnum && 0.0>=fbuf.bufF(FELASTIDX)[k*BOND_DATA + b*DATA_PER_BOND +2] ) printf("\ncleanBonds1: incomming bond not intact : 0.0>=fbuf.bufF(FELASTIDX)[k*BOND_DATA + b*DATA_PER_BOND +2] ");
        if(k<pnum)for(int j=0;j<BONDS_PER_PARTICLE;j++){                                       // check for double bonds, and remove one of them.
           if(k<i && k== fbuf.bufI(FELASTIDX)[i*BOND_DATA + j*DATA_PER_BOND])   intact=false;  // check for reciprocal bonds
           if(j>a && k== fbuf.bufI(FPARTICLEIDX)[i*BONDS_PER_PARTICLE*2 + a*2]) intact=false;  // bonds in the same direction 
        }
        if(intact==false){                                                          // remove broken/missing _incomming_ bonds
            //fbuf.bufI(FPARTICLEIDX)[i*BONDS_PER_PARTICLE*2 + a*2] = UINT_MAX;     // particle NB retain bond direction info
            fbuf.bufI(FPARTICLEIDX)[i*BONDS_PER_PARTICLE*2 + a*2 +1] = UINT_MAX;    // bond index
        }
    }// FELASTIDX //# currently [0]current index, [1]elastic limit, [2]restlength, [3]modulus, [4]damping coeff, [5]particle ID, [6]bond index [7]stress integrator [8]change-type binary indicator
    for (int a=0; a<BONDS_PER_PARTICLE;a++){                                        // loop round this particle's list of _outgoing_ bonds /////
        bool intact = false;
        uint j = fbuf.bufI(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND];
        uint bond_idx = fbuf.bufI(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND + 6];      // chk bond intact nb short circuit evaluation of if conditions.
        // j is a particle, bond_idx is in range, AND j's reciprocal record matches i's record of the bond
        if(j<pnum 
            && bond_idx<BONDS_PER_PARTICLE 
            && i==fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bond_idx*2] 
            && a==fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bond_idx*2 +1])  intact=true; 
        if(i==j){
            fbuf.bufI(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND]   =UINT_MAX;
            fbuf.bufF(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND+1] =0;                 // bond to self not allowed, 
            printf("\ncleanBonds2: i=j=%u ",i );
        }
        if(j<pnum && bond_idx>=BONDS_PER_PARTICLE)
            printf("\ncleanBonds3: outgoing bond not intact (bond_idx>=BONDS_PER_PARTICLE) i=%u, j=%u, a=%u bond_idx=%u \t",
                   i,j,a, bond_idx);
        if(j<pnum && i!=fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bond_idx*2])
            printf("\ncleanBonds3: outgoing bond not intact (i!=fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bond_idx*2]) i=%u, \t\tfbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bond_idx*2]=%u, \tj=%u, a=%u bond_idx=%u \t",
                   i,fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bond_idx*2], j,a, bond_idx);
        if(j<pnum && a!=fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bond_idx*2 +1])
            printf("\ncleanBonds3: outgoing bond not intact (a!=fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bond_idx*2 +1]) i=%u, j=%u, a=%u, \t\tfbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bond_idx*2 +1]=%u, \tbond_idx=%u \t",
                   i,j,a, fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bond_idx*2 +1], bond_idx);
        
        if(intact==false)fbuf.bufF(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND+2] =0.0;  // [2]rest_length  // remove missing _outgoing_ bonds
    }
}


extern "C" __device__ void contribFindBonds ( int i, float3 ipos, int cell, int bond, uint _bondToIdx[BONDS_PER_PARTICLE], float*_bond_dsq, float*_best_theta, uint _pnum)
{
    if ( fbuf.bufI(FGRIDCNT)[cell] == 0 ) return;                                                   // If the cell is empty, skip it.
    uint    j;
    float   dsq;
    float3  dist    = make_float3(0,0,0); 
    int     clast   = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];                        // index of last particle in this cell
    
    for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++ ) {                            // For particles in this cell.
        j       = fbuf.bufI(FGRID)[ cndx ];
        dist    = ( ipos - fbuf.bufF3(FPOS)[ j ] );                                                 // dist in cm (Rama's comment)
        dsq     = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);                                  // scalar distance squared
        
        if ( dsq < fparam.rd2 && dsq > 0) {                                                         // IF in-range && not the same particle
            float3 u,v;
            u               = dist;
            float theta     = 0;
            uint  bondCount = 0;
            for(int b=0;b<BONDS_PER_PARTICLE;b++){                                                  // Loop round existing outgoing bonds
                uint k      = fbuf.bufI(FELASTIDX)[i*BOND_DATA + b*DATA_PER_BOND];
                if(j==k){theta=FLT_MAX;break;}
                if(k<_pnum){
                    bondCount++;
                    v        = ipos - fbuf.bufF3(FPOS)[ k ];
                    theta    += abs(1.91 - acos( dot(u,v) / (length(u)*length(v)) ) );//*dsq;          // 1.91rad=109.5deg, ideal tetrahedral bond angle.
                }                                                                                   // theta = sum (differences from ideal bond angle)
            }
            if (bond==0) theta      = dsq;
            if (theta<*_best_theta){                                                                // if better than best candidate so far
                *_best_theta        = theta;
                _bondToIdx[bond]    = j;
                *_bond_dsq          = dsq;
            }
        }
    }
    return;
}


extern "C" __global__ void initialize_bonds (int ActivePoints, uint list_length, int gene) {        // Bond angle based search for new bonds.
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if ( particle_index >= list_length ) return;                                                   
    uint i = fbuf.bufII(FDENSE_LISTS)[gene][particle_index];                                        // call for dense list of solid particles (gene==1)
    if ( i >= ActivePoints ) return;
    
    //printf("\ninitialize_bonds()1: i=%u,  ",i);
    
    uint buf_length = fbuf.bufI(FDENSE_BUF_LENGTHS)[gene];
    uint gc = fbuf.bufI(FGCELL)[ i ];
    uint bondToIdx[BONDS_PER_PARTICLE]; for(int bond=0; bond<BONDS_PER_PARTICLE; bond++) bondToIdx[bond]=UINT_MAX;
    
    //printf("\ninitialize_bonds()2: i=%u,  ",i);
    
    float3 tpos         = fbuf.bufF3(FPOS)[ i ];
    uint  * uintptr     = &fbuf.bufI(FELASTIDX)[i*BOND_DATA];
    float * floatptr    = &fbuf.bufF(FELASTIDX)[i*BOND_DATA];
    
    //printf("\ninitialize_bonds()3: i=%u,  ",i);
    
    uint  elastin       = fgenome.elastin;
    float damping       = fgenome.param[elastin][fgenome.default_damping];
    float modulus       = fgenome.param[elastin][fgenome.default_modulus];
    float rest_length   = fgenome.param[elastin][fgenome.default_rest_length];
    float elastLim      = fgenome.param[elastin][fgenome.elastLim];
    
    //printf("\ninitialize_bonds()4: i=%u,  ",i);
    
    for (int bond=0; bond<BONDS_PER_PARTICLE; bond++){
        float best_theta    = FLT_MAX, bond_dsq = fparam.rd2;                                      // used to compare potential bonds
        //printf("\ninitialize_bonds()4.1: i=%u,  bond=%u",i,bond);
        
        for (int c=0; c < fparam.gridAdjCnt; c++) contribFindBonds ( i, tpos, gc + fparam.gridAdj[c], bond, bondToIdx, &bond_dsq, &best_theta, fparam.maxPoints);
        //if(bondToIdx[bond]>=ActivePoints)printf("\ninitialize_bonds()4.2: i=%u, bond=%u,  bondToIdx[bond]=%u      ",i,bond, bondToIdx[bond] );
        if(bondToIdx[bond]<ActivePoints){ 
            uintptr [bond*DATA_PER_BOND +0] = bondToIdx[bond];
            floatptr[bond*DATA_PER_BOND +1] = elastLim;
            floatptr[bond*DATA_PER_BOND +2] = rest_length;
            floatptr[bond*DATA_PER_BOND +3] = modulus;
            floatptr[bond*DATA_PER_BOND +4] = damping;
            uintptr [bond*DATA_PER_BOND +5] = fbuf.bufI(FPARTICLE_ID)[bondToIdx[bond]];
            uintptr [bond*DATA_PER_BOND +6] = 0;
            uintptr [bond*DATA_PER_BOND +7] = 0;
        }
    }
    
    //printf("\ninitialize_bonds()5: i=%u,  ",i);
}    


extern "C" __global__ void heal (int ActivePoints, uint list_length, int change_list, uint startNewPoints, uint mMaxPoints) { 
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if ( particle_index >= list_length ) return;                                                   // pnum should be length of list.
    uint i = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index];                        // call for dense list of broken bonds
    if ( i >= ActivePoints ) return;
    uint buf_length = fbuf.bufI(FDENSE_BUF_LENGTHS_CHANGES)[change_list];
    uint bond = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index+buf_length];             //bondIdx
    if (bond>BONDS_PER_PARTICLE)return;
    
    // Bond angle based search for new bond.
    uint gc = fbuf.bufI(FGCELL)[ i ];
    uint bondToIdx[BONDS_PER_PARTICLE]; for(int bond=0; bond<BONDS_PER_PARTICLE; bond++) bondToIdx[bond]=UINT_MAX;
    
    float best_theta= FLT_MAX, bond_dsq = fparam.rd2;                                               // used to compare potential bonds
    
    float3 tpos = fbuf.bufF3(FPOS)[ i ];
    float3 ipos = tpos;
    uint rnd_nmbr = hiprand(&fbuf.bufCuRNDST(FCURAND_STATE)[i]);                                     // NB bitshift and mask to get rand bool to choose bond
    float max_len = sqrt(fparam.rd2);
    
    tpos.x += max_len/float(4+(rnd_nmbr&7))     *(-1*float(1&(rnd_nmbr>>3))  );                     // shift tpos by a random step < max_len, randomises bond.
    tpos.y += max_len/float(4+((rnd_nmbr>>4)&7))*(-1*float(1&(rnd_nmbr>>7))  );
    tpos.z += max_len/float(4+((rnd_nmbr>>8)&7))*(-1*float(1&(rnd_nmbr>>11)) );
    
 //   printf("\nheal: i=%u, max_len=%f, ipos=(%f,%f,%f), tpos=(%f,%f,%f)",i,max_len, ipos.x,ipos.y,ipos.z, tpos.x,tpos.y,tpos.z);
    
    for (int c=0; c < fparam.gridAdjCnt; c++) contribFindBonds ( i, tpos, gc + fparam.gridAdj[c], bond, bondToIdx, &bond_dsq, &best_theta, fparam.maxPoints);
    /*
    for (int c=0; c < fparam.gridAdjCnt; c++) {                                     // Call contributeForce(..) for fluid forces AND potential new bonds
        contribFindBonds ( i, tpos, gc + fparam.gridAdj[c], bondToIdx, &bond_dsq, &best_theta, fparam.maxPoints);
    }
    */
    if(bondToIdx[bond]<fparam.maxPoints){
        // many are made in 1 step because each broken bond calls heal.
        uint    j_ID         = fbuf.bufI(FPARTICLE_ID)[bondToIdx[bond]];
        float   bond_length  = sqrt(bond_dsq);
        float   modulus      = 100000;       // 100 000 000                                              // 1000000 = min for soft matter integrity // 
        uint *  uintptr      = &fbuf.bufI(FELASTIDX)[i*BOND_DATA + bond*DATA_PER_BOND +0];
        float*  floatptr     = &fbuf.bufF(FELASTIDX)[i*BOND_DATA + bond*DATA_PER_BOND +0];
        
        uintptr[0]  = bondToIdx[bond];                        // [0]current index,
        floatptr[1] = 2 * bond_length ;                 // [1]elastic limit  = 2x restlength i.e. %100 strain
        floatptr[2] = 0.5*bond_length;                  // [2]restlength = initial length  
        floatptr[3] = modulus;                          // [3]modulus
        floatptr[4] = 2*sqrt(fparam.pmass*modulus);     // [4]damping_coeff = optimal for mass-spring pair.
        uintptr[5]  = j_ID;                             // [5]save particle ID of the other particle NB for debugging
        uintptr[6]  = 0;                                // [6]bond index at the other particle 'j's incoming bonds // TODO remove [6] deprecated 
        uintptr[7]  = 0;                                // [7]stress integrator
        uintptr[8]  = 0;                                // [8]change-type binary indicator
    }
}


extern "C" __global__ void lengthen_muscle ( int ActivePoints, int list_length, int change_list, uint startNewPoints, uint mMaxPoints) { //Only for Bond[0] collagen chain //NB elastic tissues (yellow ligments) are non-innervated muscle 
    // TODO consider divergently and convergently branching cases of lengthen_muscle ( int pnum )
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                             // particle index
    if ( particle_index >= list_length ) return; // pnum should be length of list.
    uint i = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index]; // call for dense list of lengthen_muscle  // NB can come from multiple bonds of same particle.
    
    if (fparam.debug>2 /*&& (threadIdx.x==0 || particle_index==0)*/ ) printf("\nlengthen_muscle()1:  particle_index=%u, list_length=%u, i=%u, ActivePoints=%u, fparam.frame=%u \t", particle_index, list_length, i, ActivePoints, fparam.frame);
    
return;  // suspend use of this kernel for now.
    if ( i >= ActivePoints ) return; 
    //if (fparam.debug>2 /*&& (threadIdx.x==0 || particle_index==0)*/ ) printf("\nlengthen_muscle()1.1:  particle_index=%u  ",particle_index);
    uint buf_length = fbuf.bufI(FDENSE_BUF_LENGTHS_CHANGES)[change_list];
    uint bondIdx            = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index+buf_length]; // bondIdx, NB FDENSE_LISTS_CHANGES [2][list_length] 
    uint secondParticleIdx  = fbuf.bufI(FELASTIDX)[i*BOND_DATA+bondIdx*DATA_PER_BOND];
    uint bondIdx_reciprocal = fbuf.bufI(FELASTIDX)[i*BOND_DATA+bondIdx*DATA_PER_BOND+6]; //[6]bond index // i.e. the incoming bondindx on next_particle_Idx
    // NB (bondIdx_reciprocal > bondIdx)  => convergent branching, (bondIdx_reciprocal < bondIdx) => divergent branching , (0==bondIdx_reciprocal == bondIdx)=> nonbranching, else => error
    
    if (bondIdx!=0)printf("\nlengthen_muscle(): (bondIdx!=0) particle_index=%u ", particle_index);
    
    if (bondIdx_reciprocal>BONDS_PER_PARTICLE || bondIdx>BONDS_PER_PARTICLE || secondParticleIdx>fparam.maxPoints){                                     // corrupt data.
        printf("\nlengthen_muscle, corrupt: bondIdx_reciprocal=%u, bondIdx=%u, secondParticleIdx=%u ", bondIdx_reciprocal, bondIdx, secondParticleIdx ); 
        return; 
    }
/*
    //if (fparam.debug>2 /_*&& (threadIdx.x==0 || particle_index==0)*_/ ) printf("\nlengthen_muscle()1.2:  particle_index=%u, i=%u, bondIdx=%u, bondIdx_reciprocal=%u  ",
    //    particle_index, i, bondIdx, bondIdx_reciprocal);
*/
    uint new_particle_Idx  =  startNewPoints + particle_index;
/*
    // addParticle(i, new_particle_Idx);
*/
    
    float3 newParticlePos =  fbuf.bufF3(FPOS)[i] - 0.5*(fbuf.bufF3(FPOS)[i] - fbuf.bufF3(FPOS)[secondParticleIdx]); // placed near second particle to ensure selection of this bond
    fbuf.bufF3(FPOS)[new_particle_Idx] = newParticlePos;
    
    printf("\nlengthen_muscle:  bondIdx_reciprocal=%u, newParticlePos=(%f,%f,%f)  ",bondIdx_reciprocal, newParticlePos.x, newParticlePos.y, newParticlePos.z );
    
    addParticle(i, new_particle_Idx);   
    uint bond_type[BONDS_PER_PARTICLE] = {0};  bond_type[0] = 1;        //  0=elastin, 1=collagen, 2=apatite
/*
    //makeBond (uint thisParticleIdx, uint otherParticleIdx, uint bondIdx, uint otherParticleBondIdx, uint bondType /_* elastin, collagen, apatite *_/)
    //atomicMakeBond(uint thisParticleIndx,  uint otherParticleIdx, uint bondIdx, uint otherParticleBondIndex, uint bond_type)
*/
    makeBond (i,                new_particle_Idx,  bondIdx, bondIdx_reciprocal, bond_type[bondIdx] ); // making the collagen chain
    makeBond (new_particle_Idx, secondParticleIdx, bondIdx, bondIdx_reciprocal, bond_type[bondIdx] );
    
    float bondRestLength = 0.5 * fbuf.bufI(FELASTIDX)[i*BOND_DATA+bondIdx*DATA_PER_BOND+6];
    fbuf.bufI(FELASTIDX)[i*BOND_DATA                +bondIdx*DATA_PER_BOND+6]   = bondRestLength;
    fbuf.bufI(FELASTIDX)[new_particle_Idx*BOND_DATA +bondIdx*DATA_PER_BOND+6]   = bondRestLength;

    if (fparam.debug>2 /*&& (threadIdx.x==0 || particle_index==0)*/ ) printf("\nlengthen_muscle()1.3:  particle_index=%u, i=%u, bondIdx=%u, bondIdx_reciprocal=%u  ",
        particle_index, i, bondIdx, bondIdx_reciprocal);

    
    // Re-organize bonds to make a contractile chain. Bond[1] must link to next particle but one in the chain. 
    uint Particle[5];
    Particle[0] =  fbuf.bufI(FPARTICLEIDX)[i*2*BONDS_PER_PARTICLE];                                                     // i.e. ParticleIdx of incoming bond[0] 
    Particle[1] =  i ;
    Particle[2] =  new_particle_Idx;
    Particle[3] =  secondParticleIdx;
    Particle[4] =  fbuf.bufI(FELASTIDX)[secondParticleIdx*BOND_DATA];                                                   // i.e. ParticleIdx of outgoing bond[0]
    
    if (fparam.debug>2 /*&& (threadIdx.x==0 || particle_index==0)*/ ) printf("\nlengthen_muscle()1.3.1:  particle_index=%u, i=%u, bondIdx=%u, bondIdx_reciprocal=%u  ",
        particle_index, i, bondIdx, bondIdx_reciprocal);
    
    int start = 1 - (Particle[0]<fparam.maxPoints);
    int stop  = 2 + (Particle[4]<fparam.maxPoints);
    for (int j=start; j<stop; j++){
        uint oldTargetIdx  = fbuf.bufI(FELASTIDX)[Particle[j]*BOND_DATA  +1*DATA_PER_BOND  +bondIdx*DATA_PER_BOND    ]; // i.e. ParticleIdx of outgoing bond[1]
        uint oldTargetBond = fbuf.bufI(FELASTIDX)[Particle[j]*BOND_DATA  +1*DATA_PER_BOND  +bondIdx*DATA_PER_BOND  +1];
        if (oldTargetIdx < fparam.maxPoints && oldTargetBond<BONDS_PER_PARTICLE){
            fbuf.bufI(FPARTICLEIDX)[oldTargetIdx*2*BONDS_PER_PARTICLE + 2*oldTargetBond   ] = UINT_MAX;  
            fbuf.bufI(FPARTICLEIDX)[oldTargetIdx*2*BONDS_PER_PARTICLE + 2*oldTargetBond +1] = UINT_MAX;
        }
        
        if (fparam.debug>2 /*&& (threadIdx.x==0 || particle_index==0)*/ ) printf("\nlengthen_muscle()1.3.2:  particle_index=%u, i=%u, bondIdx=%u, bondIdx_reciprocal=%u  ",
        particle_index, i, bondIdx, bondIdx_reciprocal);
        
        oldTargetIdx  = fbuf.bufI(FPARTICLEIDX)[Particle[j+2]*2*BONDS_PER_PARTICLE + 2*1     ];                         // i.e. ParticleIdx of incoming bond[1]
        oldTargetBond = fbuf.bufI(FPARTICLEIDX)[Particle[j+2]*2*BONDS_PER_PARTICLE + 2*1  +1 ];
        if (oldTargetIdx < fparam.maxPoints && oldTargetBond<BONDS_PER_PARTICLE){
            fbuf.bufI(FELASTIDX)[oldTargetIdx*BOND_DATA  +oldTargetBond*DATA_PER_BOND    ] = UINT_MAX;                      // [0] particle index
            fbuf.bufI(FELASTIDX)[oldTargetIdx*BOND_DATA  +oldTargetBond*DATA_PER_BOND  +6] = UINT_MAX;                      // [6] bond index
        }
        if (fparam.debug>2 /*&& (threadIdx.x==0 || particle_index==0)*/ ) printf("\nlengthen_muscle()1.3.3:  particle_index=%u, i=%u, bondIdx=%u, bondIdx_reciprocal=%u  ",
        particle_index, i, bondIdx, bondIdx_reciprocal);
        
        makeBond (Particle[j], Particle[j+2],  1, 1, 0 ); // making the elastin chain.
        //makeBond(thisParticleIdx, otherParticleIdx, bondIdx, otherParticleBondIdx, bondType /*0 elastin, 1 collagen, 2 apatite*/)
        // ## TODO connect nerves for actuation and sensation:
        // ...
        
    }
    // ## TODO connect new_particle_Idx bonds[2-5].  Need to specify bondsto fill OR change to check for existing bonds.
    //insertNewParticle(new_particle_Idx, newParticlePos, i, bondIdx, secondParticleIdx, bondIdx_reciprocal, bond_type);


    
    if (fparam.debug>2 /*&& (threadIdx.x==0 || particle_index==0)*/ ) printf("\nlengthen_muscle()1.4:  particle_index=%u, i=%u, bondIdx=%u, bondIdx_reciprocal=%u  ",
        particle_index, i, bondIdx, bondIdx_reciprocal);

    // colour particles to indicate replacement of original bond
    uint increment                                      = fparam.frame*10000 + particle_index*10;
    for (int n=0;n<5;n++){fbuf.bufI(FCLR)[Particle[n]]  = increment + n;}
/*    
    fbuf.bufI(FCLR)[new_particle_Idx]   = increment + 4;
    fbuf.bufI(FCLR)[i]                  = increment + 1;
    fbuf.bufI(FCLR)[secondParticleIdx]  = increment + 7;
*/
 if (fparam.debug>2 /*&& (threadIdx.x==0 || particle_index==0)*/ ) printf("\nlengthen_muscle()end:  particle_index=%u,  i=%u, ",  particle_index, i );
}



extern "C" __global__ void lengthen_tissue ( int ActivePoints, int list_length, int change_list, uint startNewPoints, uint mMaxPoints) { //TODO lengthen_tissue ( int pnum )  // add particle in bond
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                             // particle index
    if ( particle_index >= list_length ) return; // pnum should be length of list.
    uint i = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index]; // call for dense list of lengthen_tissue
    
    //if (fparam.debug>2 /* && (threadIdx.x==0 || particle_index==0) */ ) printf("\nlengthen_tissue() i=%u \t",i);
  return;  
    if ( i >= ActivePoints ) return; 
    uint bondIdx = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index+list_length]; 
    uint secondParticleIdx  = fbuf.bufI(FELASTIDX)[i*BOND_DATA+bondIdx*DATA_PER_BOND];
    // Insert 1 particle on axis of strecthed bond & share existing/new lateral bonds
    // It would help to know which bond. => where to add new particle
    
    uint next_particle_Idx  = fbuf.bufI(FELASTIDX)[i*BOND_DATA+bondIdx*DATA_PER_BOND];
    uint bondIdx_reciprocal = fbuf.bufI(FELASTIDX)[i*BOND_DATA+bondIdx*DATA_PER_BOND+6]; //[6]bond index // i.e. the incoming bondindx on next_particle_Idx
    
    if (next_particle_Idx>fparam.maxPoints) return; 
    
    //create new particle at mid point of bond => uint new_particle_Idx[3] 
    uint new_particle_Idx  =  startNewPoints + particle_index;
    if (new_particle_Idx>fparam.maxPoints)return; // i.e. if run out of spare particles.
    addParticle(i, new_particle_Idx);
    if (new_particle_Idx>fparam.maxPoints)return; // i.e. if addParticle() failed.
    
    printf("\nlengthen_tissue chk0:  i=%u,  next_particle_Idx=%u, fbuf.bufF3(FPOS)[i]=(%f,%f,%f) ",
            i ,next_particle_Idx, fbuf.bufF3(FPOS)[i].x, fbuf.bufF3(FPOS)[i].y, fbuf.bufF3(FPOS)[i].z );
    __syncthreads;
    
    printf("\nlengthen_tissue chk0.1:  i=%u, fbuf.bufF3(FPOS)[next_particle_Idx]=(%f,%f,%f) ",
            i, fbuf.bufF3(FPOS)[next_particle_Idx].x, fbuf.bufF3(FPOS)[next_particle_Idx].y, fbuf.bufF3(FPOS)[next_particle_Idx].z );
    __syncthreads;
    
    //fbuf.bufF3(FPOS)[new_particle_Idx]          = fbuf.bufF3(FPOS)[i] + (fbuf.bufF3(FPOS)[i] - fbuf.bufF3(FPOS)[next_particle_Idx])/2;
    float3 newParticlePos  = fbuf.bufF3(FPOS)[i] + (fbuf.bufF3(FPOS)[next_particle_Idx]  -  fbuf.bufF3(FPOS)[i])/2;
    
    printf("\nlengthen_tissue chk0.2:  i=%u, next_particle_Idx=%u, newParticlePos=(%f,%f,%f) ",
            i, next_particle_Idx, newParticlePos.x, newParticlePos.y, newParticlePos.z );
    __syncthreads;
    
    // Determine bond type from binary change-type indicator
    uint * fbufFEPIGEN = &fbuf.bufI(FEPIGEN)[i]; //*fparam.maxPoints  *NUM_GENES
    uint bond_type[BONDS_PER_PARTICLE] = {0};                          //  0=elastin, 1=collagen, 2=apatite
    
    //printf("\nlengthen_tissue chk1:  i=%u ",i );
    //__syncthreads;
    
    // Calculate material type for bond
    if (fbufFEPIGEN[9*fparam.maxPoints]/*bone*/) for (int bond=0; bond<BONDS_PER_PARTICLE; bond++) bond_type[bond] = 2;
    else if (fbufFEPIGEN[6*fparam.maxPoints]/*tendon*/||fbufFEPIGEN[7*fparam.maxPoints]/*muscle*/||fbufFEPIGEN[10*fparam.maxPoints]/*elast lig*/) {bond_type[0] = 1; bond_type[3] = 1;}
    //NB muscle& elast should not occur here, they have their own list & kernel.
    else if (fbufFEPIGEN[8*fparam.maxPoints]/*cartilage*/)for (int bond=0; bond<BONDS_PER_PARTICLE; bond++) bond_type[bond] = 1;
    
    //printf("\nlengthen_tissue chk2:  i=%u ",i );
    //__syncthreads;
    
    
    int ret = insertNewParticle(new_particle_Idx, newParticlePos, i, bondIdx, secondParticleIdx, bondIdx_reciprocal,  bond_type);
    
    
    
    
    
    
    //__device__ int  insertNewParticle(uint new_particle_Idx, float3 newParticlePos, uint parentParticleIndx, uint bondIdx, uint secondParticleIdx, uint otherParticleBondIndex, uint bond_type[BONDS_PER_PARTICLE]);
    /*
    
    //makeBond (uint thisParticleIdx, uint otherParticleIdx, uint bondIdx, uint otherParticleBondIdx, uint bondType /_* elastin, collagen, apatite *_/)
    makeBond ( i, new_particle_Idx, bondIdx, bondIdx, bond_type[bondIdx] );
    makeBond ( new_particle_Idx, next_particle_Idx, bondIdx, bondIdx_reciprocal, bond_type[bondIdx] );  // NB these _might_not_ req atomic because New Particle is not in the main list yet
    
    //// NB bonds in neighbourhood will mostly be already taken.
    //// Need to dedistribue bonds via the new particle.
    // This may in fact replace the step above as well.
    
    // fn "insert particle" will redistribute bonds as well.
    
    */
    /*
    // Get default bond params
    FBondParams *params_  = &fgenome.fbondparams[bond_type[bondIdx]];
    float max_rest_length = params_[0].param[params_->max_rest_length];
    float min_rest_length = params_[0].param[params_->min_rest_length];
    float elastLim        = params_[0].param[params_->elastLim];
    float default_length  = params_[0].param[params_->default_rest_length];
    float default_modulus = params_[0].param[params_->default_modulus];
    float default_damping = params_[0].param[params_->default_damping];
    
    /////////////////
    uint*   uint_ptr_parent = &fbuf.bufI(FELASTIDX)[i*BOND_DATA + bondIdx*DATA_PER_BOND + 0];
    float* float_ptr_parent = &fbuf.bufF(FELASTIDX)[i*BOND_DATA + bondIdx*DATA_PER_BOND + 0];
    
    uint*   uint_ptr_new = &fbuf.bufI(FELASTIDX)[new_particle_Idx*BOND_DATA + bondIdx*DATA_PER_BOND + 0];
    float* float_ptr_new = &fbuf.bufF(FELASTIDX)[new_particle_Idx*BOND_DATA + bondIdx*DATA_PER_BOND + 0];
        
    // 1st connect replacement bonds 
    
    // (i) bond with parent particle
    uint_ptr_parent [0] = new_particle_Idx;                            //[0]current index, 
    float_ptr_parent[1] = elastLim;                                    //[1]elastic limit, 
    float_ptr_parent[2] = default_length;                              //[2]restlength, 
    float_ptr_parent[3] = default_modulus;                             //[3]modulus, 
    float_ptr_parent[4] = default_damping;                             //[4]damping coeff, 
    uint_ptr_parent [5] = fbuf.bufI(FPARTICLE_ID)[new_particle_Idx];   //[5]particle ID,   
    uint_ptr_parent [6] = bondIdx;                                     //[6]bond index
    float_ptr_parent[7] = 0;                                           //[7]stress integrator 
    uint_ptr_parent [8] = 0;                                           //[8]change-type 
    
    // (ii) bond with next particle
    uint_ptr_new [0]    = next_particle_Idx;                           //[0]current index, 
    float_ptr_new[1]    = elastLim;                                    //[1]elastic limit, 
    float_ptr_new[2]    = default_length;                              //[2]restlength, 
    float_ptr_new[3]    = default_modulus;                             //[3]modulus, 
    float_ptr_new[4]    = default_damping;                             //[4]damping coeff, 
    uint_ptr_new [5]    = fbuf.bufI(FPARTICLE_ID)[next_particle_Idx];  //[5]particle ID,   
    uint_ptr_new [6]    = bondIdx_reciprocal;                          //[6]bond index
    float_ptr_new[7]    = 0;                                           //[7]stress integrator 
    uint_ptr_new [8]    = 0;                                           //[8]change-type 
    
    // (iii) reciprocal records
    fbuf.bufI(FPARTICLEIDX)[new_particle_Idx*2*BONDS_PER_PARTICLE + 1*2]       = i;                        // particle Idx
    fbuf.bufI(FPARTICLEIDX)[new_particle_Idx*2*BONDS_PER_PARTICLE + 1*2 +1]    = bondIdx;                  // bond Idx
    
    fbuf.bufI(FPARTICLEIDX)[next_particle_Idx*2*BONDS_PER_PARTICLE + 1*2]      = new_particle_Idx;         // particle Idx
    fbuf.bufI(FPARTICLEIDX)[next_particle_Idx*2*BONDS_PER_PARTICLE + 1*2 +1]   = bondIdx;                  // bond Idx
    
    */
    /*
    // 2nd redistribute bonds from parent particles ?  
    
    // 3rd fill in other bonds NB (i) bond angles, (ii) tissue types : bone & cartilage => (nearly) full bonding & 
    // NB structural celltypes by active FEPIGEN:
    // helical      : // 7 muscle  // 10 elastic_lig
    // linear       : // 6 tendon 
    // bilinear     : // 5 fibrocyte 
    // homogeneous  : // default (mesenchyme/loose ct) //3 fat : elastin
                      // 8 cartilage                           : collagen
                      // 9 bone                                : apatite
    
    // exclude bond index ...
    
    // freeze bonds // Octrant lists 
         
    uint    bonds[BONDS_PER_PARTICLE][2];
    float   bond_dsq[BONDS_PER_PARTICLE];
    uint gc = fbuf.bufI(FGCELL)[ i ];                                               // Get search cell	NB new particle not yet inserted in correct cell.
	if ( gc == GRID_UNDEF ) return;                                                 // particle out-of-range
	gc -= (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
    // NB must select correct "float max_rest_length = params_[0].param[params_->max_rest_length];"
    
    for (int c=0; c < fparam.gridAdjCnt; c++) find_potential_bonds ( i, fbuf.bufF3(FPOS)[ i ], gc + fparam.gridAdj[c], bonds, bond_dsq, max_rest_length*max_rest_length); 
                                            //find_potential_bonds (int i, float3 ipos, int cell, uint _bonds[BONDS_PER_PARTICLE][2], float _bond_dsq[BONDS_PER_PARTICLE], float max_len_sq);
        // Add new bonds /////////////////////////////////////////////////////////////////////////////
    for (int a =0; a< BONDS_PER_PARTICLE; a++){
        int otherParticleBondIndex = BONDS_PER_PARTICLE*2*bonds[a][0] + 2*a /_*bonds[a][1]*_/; // fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex]
        
        if (fparam.debug>2 && (uint)bonds[a][0]==i) printf("\n (uint)bonds[a][0]==i, i=%u a=%u",i,a);  // float bonds[BONDS_PER_PARTICLE][3];  [0] = index of other particle, [1] = dsq, [2] = bond_index
                                                                                    // If outgoing bond empty && proposed bond for this quadrant is valid
        if (fparam.debug>2 && fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +1] == 0.0  &&  bonds[a][0] < pnum  && bonds[a][0]!=i  && bond_dsq[a]<3 ){  // ie dsq < 3D diagonal of cube ##### hack #####
                                                                                    // NB "bonds[b][0] = UINT_MAX" is used to indicate no candidate bond found
                                                                                    //    (FELASTIDX) [1]elastic limit = 0.0 isused to indicate out going bond is empty
            //if (fparam.debug>2)printf("\nBond making loop i=%u, a=%i, bonds[a][1]=%u, bond_dsq[a]=%f",i,a,bonds[a][1],bond_dsq[a]);
            
            
            do {} while( atomicCAS(&ftemp.bufI(FPARTICLEIDX)[otherParticleBondIndex], UINT_MAX, 0) );               // lock ////////// ###### //  if (not locked) write zero to 'ftemp' to lock.
            if (fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex]==UINT_MAX)  fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex] = i;    //  if (bond is unoccupied) write to 'fbuf' to assign this bond
            ftemp.bufI(FPARTICLEIDX)[otherParticleBondIndex] = UINT_MAX;                                            // release lock // ######

            if (fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex] == i){                                              // if (this bond is assigned) write bond data
                fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex +1] = a;                                             // write i's outgoing bond_index to j's incoming bonds
                uint i_ID = fbuf.bufI(FPARTICLE_ID)[i];                                                             // retrieve permenant particle IDs for 'i' and 'j'
                uint j_ID = fbuf.bufI(FPARTICLE_ID)[bonds[a][0]];                                                   // uint bonds[BONDS_PER_PARTICLE][2];[0]=index of other particle,[1]=bond_index
                float bond_length = sqrt(bond_dsq[a]);
                float modulus = 100000;       // 100 000 000                                                        // 1000000 = min for soft matter integrity // 
                fbuf.bufI(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND]    = bonds[a][0];                               // [0]current index,
                fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +1] = 2 * bond_length ;                          // [1]elastic limit  = 2x restlength i.e. %100 strain
                fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +2] = 0.5*bond_length;                               // [2]restlength = initial length  
                fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +3] = modulus;                                   // [3]modulus
                fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +4] = 2*sqrt(fparam.pmass*modulus);              // [4]damping_coeff = optimal for mass-spring pair.
                fbuf.bufI(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +5] = j_ID;                                      // [5]save particle ID of the other particle NB for debugging
                fbuf.bufI(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +6] = bonds[a][1];                               // [6]bond index at the other particle 'j's incoming bonds
                //if (fparam.debug>2)printf("\nNew Bond a=%u, i=%u, j=%u, bonds[a][1]=%u, fromPID=%u, toPID=%u,, fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex]=%u, otherParticleBondIndex=%u",
                //       a,i,bonds[a][0],bonds[a][1],i_ID,j_ID, fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex], otherParticleBondIndex);
            }            
        }// end if 
        __syncthreads();    // NB applies to all threads _if_ the for loop runs, i.e. if(freeze==true)
    }           // TODO make this work with incoming & outgoing bonds, NB preserve existing bonds                    // end loop around FELASTIDX bonds

    */
}


extern "C" __global__ void shorten_muscle ( int ActivePoints, int list_length, int change_list, uint startNewPoints, uint mMaxPoints) { //TODO shorten_muscle ( int pnum )  // remove particle in chain & update contractile bonds
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                             // particle index
    if ( particle_index >= list_length ) return; // pnum should be length of list.
    uint i = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index]; // call for dense list of shorten_muscle
    
    if (fparam.debug>2 && (threadIdx.x==0 || particle_index==0) ) printf("\nshorten_muscle() i=%u \t",i);
    if ( i >= ActivePoints ) return; 
    uint bondIdx = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index+list_length]; 
    // Need to remove 3 particles, and close the gap.
    
    
    
}

extern "C" __global__ void shorten_tissue ( int ActivePoints, int list_length, int change_list, uint startNewPoints, uint mMaxPoints) { //TODO shorten_tissue ( int pnum )  // remove particle and connect bonds along their axis
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                             // particle index
    if ( particle_index >= list_length ) return; // pnum should be length of list.
    uint i = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index]; // call for dense list of shorten_tissue
    
    if (fparam.debug>2 && (threadIdx.x==0 || particle_index==0) ) printf("\nshorten_tissue() i=%u \t",i);
    if ( i >= ActivePoints ) return; 
    uint bondIdx = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index+list_length]; 
    // Need to remove 1 particle and close the gap
    // It would help to know which bond. => how to close the gap
    
    
}

extern "C" __global__ void strengthen_muscle ( int ActivePoints, int list_length, int change_list, uint startNewPoints, uint mMaxPoints) { //TODO strengthen_muscle ( int pnum )  // NB Y branching etc
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                             // particle index
    if ( particle_index >= list_length ) return; // pnum should be length of list.
    uint i = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index]; // call for dense list of strengthen_muscle
    
    if (fparam.debug>2 && (threadIdx.x==0 || particle_index==0) ) printf("\nstrengthen_muscle() i=%u \t",i);
    if ( i >= ActivePoints ) return; 
    uint bondIdx = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index+list_length]; 
    // Need to doulble up the helix i.e. add particles and contractile bonds in parallel.
    // Q Induced by ?
    // Q double up How ? 
    // NB difference between a helix and a zig-zag is only that the contractile bonds reach 2 particles ahead.
    
    
    
}

extern "C" __global__ void strengthen_tissue ( int ActivePoints, int list_length, int change_list, uint startNewPoints, uint mMaxPoints) { //TODO strengthen_tissue ( int pnum )  // add particle and bonds in parallel AND reduce original bon's modulus
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                             // particle index
    if ( particle_index >= list_length ) return; // pnum should be length of list.
    uint i = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index]; // call for dense list of strengthen_tissue
    
    if (fparam.debug>2 && (threadIdx.x==0 || particle_index==0) ) printf("\nstrengthen_tissue() i=%u \t",i);
    if ( i >= ActivePoints ) return; 
    uint bondIdx = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index+list_length]; 
    // Need to double up articles and bonds in parallel wrt the affected bond
    // It would help to know which bond. => where to place the new particle i.e. orthogonal to the bond NB place where there is space in the plane.
    
    
}

extern "C" __global__ void weaken_muscle ( int ActivePoints, int list_length, int change_list, uint startNewPoints, uint mMaxPoints) { //TODO weaken_muscle ( int pnum )  // NB Y branching etc
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                             // particle index
    if ( particle_index >= list_length ) return; // pnum should be length of list.
    uint i = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index]; // call for dense list of weaken_muscle
    
    if (fparam.debug>2 && (threadIdx.x==0 || particle_index==0) ) printf("\nweaken_muscle() i=%u \t",i);
    if ( i >= ActivePoints ) return; 
    uint bondIdx = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index+list_length]; 
    // Need to remove a row of particles in parallel - i.e. form/propagate a branch 
    // 
    
    
}

extern "C" __global__ void weaken_tissue ( int ActivePoints, int list_length, int change_list, uint startNewPoints, uint mMaxPoints) { //TODO weaken_tissue ( int pnum )  // remove particle & transfer bonds laterally  
    uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                             // particle index
    if ( particle_index >= list_length ) return; // pnum should be length of list.
    uint i = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index]; // call for dense list of weaken_tissue
    
    if (fparam.debug>2 && (threadIdx.x==0 || particle_index==0) ) printf("\nweaken_tissue() i=%u \t",i);
    if ( i >= ActivePoints ) return; 
    uint bondIdx = fbuf.bufII(FDENSE_LISTS_CHANGES)[change_list][particle_index+list_length]; 
    // Need to remove a particle and close the gap by transfering load laterally
    // It would help to know which bond. => how to close the gap
    
    
}

// NB aim to set particles to their 'correct' bond pattern for their tissue type(s)
// What happens if different bonds cause a particle to be repeatedly created and deleted ? When/how could this happen ? 



    
    
/*   
//     #pragma unroll
//     for(int j=0;j<BONDS_PER_PARTICLE;j++)
//         modulus = (j==0 &&(FEPIGEN[6]/_*tendon*_/||FEPIGEN[7]/_*muscle*_/||FEPIGEN[10]/_*elast_lig*_/) 
//                     ||(j<3&&FEPIGEN[5]/_*fibrocyte*_/)
//                     ||FEPIGEN[8]/_*cartilage*_/)
//                 *collagen_mod   
//                 + FEPIGEN[9]*bone_mod
//                 + elastin_mod; 
//         
//         FELASTIDX[j*DATA_PER_BOND +3]/_*modulus*_/ = FELASTIDX[j*DATA_PER_BOND +3] +  modulus * (FELASTIDX[j*DATA_PER_BOND+7]/_*stress_integrator*_/ - stregthThreshold);  
//         
//         if(FELASTIDX[3]>modulus) FCONC[0] = duplicate  // i.e. need to grow more tissue to dissipate the stress
//     }
//     
//     for(int j=0;j<BONDS_PER_PARTICLE;j++){
//         Adjust rest_length according to stress_integrator
//         
//         if too short, mark particle to remove
//         if too long, du[licate particle along axis of stretch
//     }
//     
//     
//     if (FEPIGEN[6]/_*tendon*_/||FEPIGEN[7]/_*muscle*_/||FEPIGEN[10]/_*elast_lig*_/){
//         NB FELASTIDX[0*DATA_PER_BOND +3] i.e. bond[0] is the principal collaen fibre
//         In muscle & elast lig, bond[1] reaches ahead 2 steps in the chain 
//         In muscle bond[1] will contract iff motor nerve is firing
//         Elast lig has no motor nerve & does not contract
//         
//         moduli & lengths are fixed. 
//         Strength => delete/duplicate particle in parallel.
//         Length => delete/duplicate along bond[0] axis
//         
//         if((FEPIGEN[6]/_*tendon*_/){                              // danger that every particle in chain will elongate. ? elongate at myotendinous jxn?
//             if ( integrator[0] > elongate_threshold ){
//                 new_particle = atomicAdd( &particles_inuse  ); //e.g. atomicAdd ( &fbuf.bufI(FGRIDCNT)[ gs ], 1 ); 
//             }
//             if ( integrator[0] >   
//         }
//     }
//     if (FEPIGEN[8]/_*cartilage*_/||FEPIGEN[9]/_*bone*_/){
//     }
*/   
    
/*    
    // Equation to modify mass/radius:
    // (i)   mass & radius  * TF[0] 
    
    // (ii)  if (radius > 1.36) split : make two new particles, position orthogonal to two strongest bonds 
    
    // (iii) if (radius < 1) combine : find another particle to combine with. closest particle with same cell type(s)
    
    
    
    // Equation to break/make springs, due to:
    // (i) particle split/combine
    // (ii) spring breakage
    // (iii) particle migration  // modify rest length -> stress, break if too short
    // For each unused bond:
    //      Find new attachment in desired direction, at desired distance, and of desired cell type
      
    
    // genes: (expect 64 to get from zygote-> hand)
    
    // -- special genes, for simulation efficiency
    // 0 active particles
    // 1 solid  (has springs)
    // 2 living/telomere (has genes)
    
    // -- behavioural celltypes
    // 3 fat
    // 4 nerves 
    // 5 fibrocyte
    // 6 tendon
    // 7 muscle
    // 8 cartilage
    // 9 bone
    // 10 elastic_lig
    
    
    // -- regulatory genes
    // 11 pluripotency
    // 12 Trophoblast
    // 13 Amniotic Trophoblast
    // 14 Epiblast/ectoderm
    // 15 Hypoblast/endoderm
    // 16 Primitive node
    // 17 Primitive streak
    // 18 Mesoderm
    // 19 Notocord
    // 20 Pre-somitic mesoderm → ‘clock’ 
    // 21 Dorsal ectoderm
    // 22 Somite
    // 23 Hox 1
    // 24 Hox 4
    // 25 Hox 5
    // 26 Hox 6
    // 27 ZPA – zone of polarizing activity
    // 28 AER – apical epidermal ridge
    // 29
    // 30
    // 31
    
    
    // Non-diffusible TFs 
    // 0 growth - treat as int, INT_MIN => die
    // 1 # not used: spring stress integrator_1 slow addition, fast decay, prolonged stress indicator // rather use per spring integrator
    // 2 # not used: spring stress integrator_2 fast addition, slow decay, cyclical peak indicator    // rather use current stress?  
    // 3 clock_1
    // 4 clock_2
    // 5 
    // 6
    // 7
    
    // matrix stiffness 
    // hyaline & apatite => all bonds triangulated and stiff
    // fribrocyte => 2 stiff bonds
    // tenocyte & muscle => only one stiff bond
    // elastin => medium stiffness & long yield strain
    
                                        
    //(ii)make/break bonds - run on dense list of particles, requires (i)search of neighbours, (ii)spare bonds on both 
    
    
    
    //(iii)add/remove particles from/to reserve list. NB ideally don't process particles in reserve list. 
    
    
    
    // Remodelling rules
    // Different cell types respond by secreting or resorbing their characteristic materials. 
    // The general pattern of response is 
    // (i) prolonged tension causes lengthening,                    i.e. slow addition, fast decay integrator_1.
    // (ii) cyclical loading causes strengthening,                  i.e. fast addition, slow decay integrator_2.
    // (iii) low peak strain causes shortening of fibrous tissue,  (low value on integrator_1)
    // (iv) low peak stress causes weakening.                      (low value on integrator_2)
    
    // peak stress & strain can differ for muscle, not for other tissues.
*/    
    
 

/*
    // what is current epigenetic state 
    // which genes should run. 
    
    ////////////////////////////////
    
    // For a given cell (i.e. particle), there is a list of active genes – bit mask on an "active genes" uint. This could be a uint array if >32 genes are required.
    // Genes may contain bit masks for activating other genes.
    // These are equivalent to "Long non-coding RNA" transcription activators →  which help to form the "promotor initiator complex"
    // (As opposed to general silencer/supressor/enhancer binding transcription factors.) NB silencing is permanent.
    // This leads to dense lists for particles on which to execute each gene.
    // For each gene : active/inactive & silenced/not_yet
    
    // Efficient tracking of particles for active gene lists:
    // Copy bond tracking, 
    // Use "update genes" flag to make dense list of changes at particle sorting time.
    // Run update genes kernel on "changes" list -> add to 
    
    // Making dense lists: 
    // # should hold particles in the same order as the general list
    // # should require only processing of (i) existing dense list PLUS (ii) changes list
    
    // Have separate "reinitiate dense lists" kernel - to check / limit error propagation (rarely run).
    // See section on Optimisation below.
    
    
    ///////////////////////////////
    
    // Points from "New Biological Morphogenetic Methods..."
    // 2.1) Mutation of Mutability - this is genome modification, outside of the simulation. 
    //      However it constrains how genes can be implemented.
    //
    // 2.2.1) Epigenetic Cell Lines—Morphogenetic and Histological Identities
    //
    //  Epigenetic variables: 
    //      (i) (float)current_activation, - 'phosphorylation'
    //      (ii) (bool)available/silenced, - 'methylation' => epigenetic type.
    //      (iii) (uint) spread count down - of silencing - e.g. Hox Genes stop spreading -> epigenetic type
    //                                          NB requires sequence of genes on chromosome.
    //      (iv) (bool) stop spreading
    //      (v)  (bool) "Not yet activated"
    //      => bool, bool, bool, uint, float. Could bitshift the uint to get the three bools.
    //
    //  Genetic parameters: 
    //      (i) Mutability, - NB probability of mutation at all, not magnitude of change.
    //                      - types: (a) cis-regulatory (I)degree of sensitivity, (II)to what - morphogen/stress
    //                               (b) change of gene action (not req for morphogenesis, i.e. protien change)
    //                               (c) genome architecture - (I) duplication/relocation of gene, 
    //                                                         (II)repartition of chromosomes
    //
    //      (ii) Delay/insulator, barrier to spread of inactivation.
    //
    //      (iii) Cis-regulatory sensitivities (morphogens, stress & strain cycles) altering current activation.
    //
    //      (iv)  Cell actions - secrete morphogens, move, adhere, divide, secrete/resorb material 
    //                          - dependent on current activation.
    //                          - Q: how to implement ?
    //                                  - move - NB risk of adding energy... 
    //                                  - adhere - make/break springs
    //                                  - divide - add particle from reserve.
    //                                  - secrete/resorb - change particle mass, viscosity, fluid stiffness, 
    //                                                   - spring length/stiffness -> anisotropy
    //
    // 2.2.2) Local Anatomical Coordinates—From Morphogen Gradients
    //                         - implemented via secretion, diffusion, & breakdown of transcription factors
    //                         - need genetic codes for:
    //                                  (I&II) symmetry breaking - establish orthogonality of poles & layers 
    //                                          (blastulation, primitive streak, gastrulation)
    //                                          blastomere->cyst->embryonic disk->primitive streak....
    //                                  
    //                                  (III) clock & wave front -> Hox genes
    //                                  (IV) gap and pair
    //                                  (V) tissue layer co-growth
    //                                  (VI) limb bud location
    //                                  (VII) limb growth & segmentation
    //                                  (VIII) digital ray lateral & lognitudinal segmentation
    //                                  (IX) reuse of synovial joint 'module'
    //                                  (X) location, migration & connection of of muscle-tendon
    //                                  (XI) location & connection of ligaments - articular, retinacular, dermal
    //                                  (XII) dermal specialization - palmar pads, nails, claws, hooves
    //                                  (XIII) nervous system connection & construction 
    //                          - NB epi-genetic branching tree, local coords, repeated patterns 
    //                                                                  -> reuse of cell types & modules 
    //
    // 2.2.3) Remodeling - implemented through cell actions 2.2.1(iv), regulated through 2.2.1(iii).
    // 
    //
    ///////////////////////////////////////////////////////////////////////////
    
    // Required genes: NB #define NUM_GENES  16.  NB tractability + evolvability
    //
    // Basic actions ://////  4+ ... but are these functions of celltype ?
    //
    // Add/remove particles - function of mass/radius
    //
    // Add remove mass/radius
    //
    // Increase/decrease matrix modulus & viscosity
    //
    // Incr/decr spring length, stiffness & damping - which springs? -> anisotropy & adhesion
    // 
    // 
    
    // Tissue modification rules 
    //
    // (i) prolonged tension causes lengthening, 
    // (ii) cyclical loading causes strengthening, 
    // (iii) low peak strain causes shortening of fibrous tissue, 
    // (iv) low peak stress causes weakening.
    // 
    // ? rolling integrators for spring stress & stress^3 -> mean vs peak
    // NB for muscles stress & strain are independent. => need to track strain for lengthening, stress for strengthening
    
    // Bone growth is regulated by morphogen diffusion at the growth plates and articular cartilage.
    // Bone itself is shaped by 
    //      * passive deformation of the bone primordia in the formation of the joints
    //      * active remodeling in response to forces to form the ridges and protrusions where major muscles and tendons attach
    //
    // 
    
    //
    // Cell type genes: 9+2
    //
    // bone, cartilage, tendon, muscle, ligament/fascia, loose tissue, dermis, epidermis, horn, 
    //
    // myotendinous junction, enthesis - poss double-expression
    //
    // 
    //
    // General anatomical modules:///////
    // 
    // articulation/synovial joint
    // 
    // bone primordia
    // 
    // endo/meso/ectoderm
    //
    // secretory
    //
    //
    // 
    // Epigenetic labels:
    // 
    // Hox genes - axial zoning - (6 used, 13x4 available)
    //
    // Limb bud fore/hind  (poss due to Hox combination)
    // 
    // Other homeobox genes - autopod, zeugoopod, stylopod, scapula/pelvis
    //
    // digit rays, carpus, metacarpus, phalanges, plalanx number, nail/claw/hoof
    
    
    
 */   
    
    // Data structures (fluid.h)
    //
    /*
        struct FGenome{   
                        // ## currently using fixed size genome for efficiency. 
                                            // NB Particle data size depends on genome size.
            uint mutability[NUM_GENES];
            uint delay[NUM_GENES];
            uint sensitivity[NUM_GENES][NUM_GENES]; // for each gene, its sensitivity to each TF or morphogen
            uint difusability[NUM_GENES][2];// for each gene, the diffusion and breakdown rates of its TF.
            //uint *function[NUM_GENES];    
                        // Hard code a case-switch that calls each gene's function iff the gene is active.
        };                                  // NB gene functions need to be in fluid_system_cuda.cu
    */
    
    /*
        struct FBufs {  // holds an array of pointers,
            ..
            char*				mcpu[ MAX_BUF ];
            ..
        }
    */
    
    /*
            #define FFORCE		3       //# 3DF        force 
            #define FPRESS		4       //# F      32  pressure
            #define FDENSITY	5       //# F          density

            #define FELASTIDX   14      //# currently [0]current index, [1]elastic limit, [2]restlength, 
                                                    [3]modulus, [4]damping coeff, [5]particle ID, [6]bond index 
            #define FPARTICLEIDX 29     //# uint[BONDS_PER_PARTICLE *2]  
                                                    list of other particles' bonds connecting to this particle AND their indices 
            #define FPARTICLE_ID 30     //# uint original pnum,
            #define FMASS_RADIUS 31     //# uint holding modulus 16bit and limit 16bit.  
            #define FNERVEIDX   15      //# uint
            #define FCONC       16      //# float[NUM_TF]        NUM_TF = num transcription factors & morphogens
            #define FEPIGEN     17      //# uint[NUM_GENES]
    */
/*    ///////////////////////////////////////////////////////////////////////////////
    
    
    // Code Optimisation:
    // Making dense lists
    //           part of particle sorting, build particle index arrays for 
    //                  (i) available genes (ii) active genes (iii) diffusion particles (iv) active/reserve particles. 
    //                   NB sequence of kernels called bu fluid_system::run()
    //                   InsertParticlesCUDA - sort particles into bins
    //                   PrefixSumCellsCUDA - count particles in bins - need to count (i&ii) above. (NB FUNC_FPREFIXSUM & FUNC_FPREFIXFIXUP)
    //                   CountingSortFullCUDA - build arrays - need (NB TransferToTempCUDA(..) for each fbuf array )
    
    // NB bitonic merge sort _may_ be useful to sort particles in active gene lists wrt to their location in the main particle list.
    // This is sorting particles in gene bins, on their particle bin FGNDX.
    // _Alternatively_, could loop on particles in bin and write each to gene bin. Max time : max num particles/bin. 
    
    // could write to a list of particles per gene, then run dense blocks for each gene.
    // NB all particles in block execute identical code.
    // Build active gene arrays during sorting
    // NB sequence : InsertParticles, PrefixSumCells, CountingSortFullCUDA  

    // NB generally in C/C++/Cuda types <32bit have to be converted to 32bit for processing. 
    // => use 32bit int/float, except where there is explicit support.
    
    // ############ convert to FP16 - NB Minimum spec: SM 5.3, so on Tesla-P100 (SM 6.0), NOT on GTX970m (SM 5.2)
    // NB P100 has GP100 with FP16, but the GTX 10xx series have GP104 with INT8 instead.
    // see https://docs.nvidia.com/cuda/pascal-tuning-guide/index.html#arithmetic-primitives
    // see https://github.com/tpn/cuda-samples/tree/master/v8.0/0_Simple/fp16ScalarProduct
    // 
    
    // ############ convert to BFLOAT16 - only available on RTX cores,  i.e. not Tesla-P100
    // see  https://mc.ai/fp64-fp32-fp16-bfloat16-tf32-and-other-members-of-the-zoo/ 
    // https://docs.nvidia.com/cuda/cuda-math-api/modules.html
    // also https://medium.com/@prrama/an-introduction-to-writing-fp16-code-for-nvidias-gpus-da8ac000c17f
    // NB this also applies to much of the data in Morphogenesis.
    ///////////////////////////////////////////////////////////////////////////////
    
    
    
    // Epigenetic data uint[NUM_GENES], per gene :  1st bit -> available/silenced
    //                                              2nd portion -> spread/stop
    //                                              3rd portion -> current activation
    
    // Genome data (once for all particles):    uint sensitivity[NUM_GENES][NUM_GENES]; // cis-regulatory sensitivity to each TF, register for automata kernel 
    //
    //                                          uint difusability[NUM_GENES][2];        // diffusion rates of FCONC, register for diffusion kernel
    //                                          
    //                                          uint delay[NUM_GENES];                  // sets intial spread/stop
    //                                          uint mutability[NUM_GENES];             // used only for mutation
    
    // Epigenetics kernel
    // for each (available) gene: read epigenetic data, compute spread of silencing,  
    
    // Gene execution kernels
    // for each (active) gene:
    // for each particle in dense list of particles where that gene is active:
    //                          read current epigenetic activation, read FCONC, FNERVEIDX, FELASTIDX strain, FPRESS, FDENSITY
    //                          compute current activity of gene, & change in epigenetic activation
    //                          run gene function
    //                              (i)modify  FCONC, FNERVEIDX, FPRESS, FDENSITY, FMASS_RADIUS,
    //                                         FELASTIDX [1]elastic limit, [2]restlength, [3]modulus, [4]damping coeff
    //                                      
    //                              (ii)make/break bonds    - requires (i)search of neighbours, (ii)spare bonds on both 
    //                                                      - run on dense list of particles
    //
    //                              (iii)add/remove particles from/to reserve list. NB ideally don't process particles in reserve list. 
    
    
    // NB difference between sparse representation of gene sensitivities for efficient simulation
    // vs dense representation for mutation.
*/  

    // New plan:
    // 1) make _densely_packed_lists_ 
    //      (i) for each gene
    //      NB needs mods to prefixSum to count for bins of each gene.
    //      Run only on: existing list + changes list. NB most particles run only a few genes at a time.
    // 
    //      i.e. for each particle in list atomic_add to bin count
    //      Then for each bin in main list, write particle index to dense list of each active gene. 
    //      NB running this kernel on bins avoids the need to sort or atomic add.
    //
    //      (ii) Likewise make dense list(s) for epigenetics.
    //      (iii) also for elastic vs fluid,  diffusion vs non-diffusion, reserve vs in use.
    //      NB cuda malloc for list sizes, when enlargement req.
    //
    // 2) call epigenetics kernel for dense list. 
    //          - Used for Hox genes in somites, and probably limb segments.
    // 
    // 3) call gene kernel for each dense list
    //      NB each 'gene' is equivalent to a biological gene cluster under common cis regulatory control.
    //      Each gene has a list of operations :
    //      (i) packed sparse list of sensitivities to TFs.
    //      (ii) gene action(s) ?
    //          - modify params for property update kernel
    //          - activate other genes
    //          - silence self
    //          - secrete a few TFs
    //          - move cell i.e. change which particles it is bonded to.
    //      (iii) nervous interactions
    //          - send sensory data - to nervous system
    //          - contract muscle - on nerve stimulus, NB temporary change of stiffness & rest ln.
    //           
    // 4) call property update kernel for all particles 
    //      reads parameters for each particle to modify properties - tissue type: ectoderm, mesoderm, cartilage, bone, muscle, tendon, fascia, fat, horn.
    //      avoids multiple calculations & edits.
    //          - mass/radius, divide/combine/delete
    //          - bonds - form/break/length/stiffness/elastic limit
    //          - fluid - stiffness & viscosity
    //
    // Automatic genotype Optimisation: (In lieu of full differentiability).  
    // ? Record gradients ? wrt to what ? parameters of genome - but select which. 
    // Record snap shots. Replay to find _when_ change most affected desired result. 
    // i.e. gradient of result wrt time.
    // Replay from snap shot - which genes are active? 
    // Find gradients of result wrt to genes
    // For most significant genes, What are they sensitive to? -> gradient of gene wrt factor
    // Options (1) adjust sensitivity, (2) increase the source of the stimulus
    // 
    
 
 
 
//////////////////////////////////////////////////////////

    // Diffusion kernel: (i) read & use uint difusability[NUM_GENES][2];    // for each gene, the diffusion and breakdown rates of its TF.
    //                   (ii) non-difusability of morphogens outside body, yet we may want fluid & womb fluid-elastic simulation
    //                   (iii) non-difusibility of internal transcription factors
    //                   (iv) breakdown rate of morphogens and transcription factors, ?
//! constant diffusion rate (as a percentage, 0.0 to 1.0) of chemical exchanged per step. change this in future!
#define DIFFUSE_RATE 10.0   // - replace with: FGenome->difusability[NUM_GENES][2] (above)

//! loops over all the chemicals in the given particle and exchanges chemicals
extern "C" __device__ void contributeDiffusion(uint i, float3 p, int cell, const float currentConc[NUM_TF], float newConc[NUM_TF], uint diffusability[NUM_TF]){  
    // if the cell is empty, skip it
    if (fbuf.bufI(FGRIDCNT)[cell] == 0) return;

    // this is all standard setup stuff, borrowed from contributePressure()
    register float d2 = fparam.psimscale * fparam.psimscale; // (particle simulation scale), not PSI
    register float r2 = fparam.r2 / d2;

    // offset of particle in particle list, and number of particles in cell?
    int clast = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];

    // iterate over particles in cell
    for (int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++) {
        int pndx = fbuf.bufI(FGRID)[cndx];

        // distance between this particle and considered particle (scalar distance squared, to save time I presume)
        float3 dist = p - fbuf.bufF3(FPOS) [pndx];
        float dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);

        // if the particle is in range, and not ourselves
        if (dsq < r2 && dsq > 0.0) {
            // distance falloff, diffusion rate scalar
            float c = (r2 - dsq) * d2;

            // chemical loop
            // for each chemical in this neighbour particle, exchange an amount relative to the diffusion rate with us
            // get the j'th chemical from this particle and exchange
            #pragma unroll
            for (int j = 0; j < NUM_TF; j++) 
                if(diffusability[j]) newConc[j] += diffusability[j] * c * (fbuf.bufF(FCONC)[pndx * NUM_TF + j] - currentConc[j]);
        }
    }
    // method:
    // add to ourselves 1% of what they have, and give away 1% of what we have
    // compute calls contribute once per bin
    // contribute will loop over particles for the bin
    // therefore it needs to loop over each 16 chemical per particle
    // space -> bin -> particle -> chemical in particle

    // this function returns nothing because all arguments are passed to it
    return;
}

//! main function to handle calculating diffusion, visits bins, then particles, then chemicals in particles
extern "C" __global__ void computeDiffusion(int pnum){
    // get particle index
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    // if the particle is outside the simulation, quit processing
    if (i >= pnum) return;

    // collect current concentration: copy from global memory to thread memory
    float newConc[NUM_TF] = {0};
    float currentConc[NUM_TF] = {0};
    // TODO maybe memcpy?
    for (int j = 0; j < NUM_TF; j++){
        currentConc[j] = fbuf.bufF(FCONC)[i * NUM_TF + j];
    }
    uint diffusability[NUM_TF];
    for (int j = 0; j < NUM_TF; j++) diffusability[j]=fgenome.tf_diffusability[j];

    // Get search cell
    int nadj = (1 * fparam.gridRes.z + 1) * fparam.gridRes.x + 1;
    uint gc = fbuf.bufI(FGCELL) [i];
    if (gc == GRID_UNDEF) return;
    gc -= nadj;

    // Now we work to exchange diffusion, by adding our neighbours chemicals and subtracting our own chemicals
    float3 pos = fbuf.bufF3(FPOS) [i];
    // bin loop: visit the bins
    for (int c = 0; c < fparam.gridAdjCnt; c++) {
        contributeDiffusion(i, pos, gc + fparam.gridAdj[c], currentConc, newConc, diffusability);
    }
    __syncthreads();

    // for this particular particle, loop over chemicals and write to global memory
    // TODO could also be memcpy
    for (int j = 0; j < NUM_TF; j++){
        fbuf.bufF(FCONC)[i * NUM_TF + j] = fgenome.tf_breakdown_rate[j] * (currentConc[j] + newConc[j]);
    }
}

/*
Wendland C2 kernel for 3D,  

    Phi_(1,3) := ((1-r)**4)*(1+4r),     for range 0<=r<=1, where r is the distance between two particles.

Used to prevent particle clumping/pairing instability under tension. 

for which:
    C = 21/2PI,              C := "normalization constant"
    sigma**2/H**2 = 1/15,    sigma := std dev,    sigma**2 := variance,     H := kernel basis,   h := 2*sigma
    
    1st differential wrt r,         Del.Phi_(1,3)   =   4(1-r)**4   -  4(1-r)**3 * (1+4r)
    2nd differential wrt r,     Del.Del.Phi_(1,3)   = -32(1-r)**3   + 12(1-r)**2 * (1+4r)  
    
    (i.e. the Laplacian differential. Not to be confused with the Laplacian integral transform.)

    From (Dehnen & Al 2012) "Improving convergence in smoothed particle hydrodynamics simulations without pairing instability",
    Differentials computed with Sympy.

##
    
Construction of SPH equations from (Muller et al 2003), 

Navier-Stokes eq for conservation of momentum, in SPH becomes:

    Rho(delta.v/delta.t + v.del.v) = -del.p + rho.g + mu.del.del.v      where p:=pressure, g:=gravity, mu:=coeff viscosity, v:=relative velocity between particles
    
    particle accel:= a_i = dv_i/dt = ( -del.p + rho.g + mu.del.del.v + surface_tension ) / rho_i 
    
    Forces:
    
    f_i^surface_tension := sigma.k.n = -sigma.del.del.Cs.n/|n|                  where n = del.Cs,  Cs(r):= SUM_j{  m_j.(1/rho_j).W(r-r_j,h) }
    
                                                                                so n/|n| is the direction vector, of the gradient of the density field (??)
    
                                     = -sigma.del.del. SUM_j{  m_j.(1/rho_j).W(r-r_j,h) } .n/|n| 

                                                                                where sigma:=surf_tension const.  W():= smoothing kernel.
                                                                        
    f_i^viscosity := mu.SUM_j{ m_j .((v_j-v_i)/rho_j).del.del.W(r_i-r_j,h) }    where mu:= coeff viscosity.
    
                                                                                where del.del.W(..)  is the Laplacian differential of the smoothing kernel.
    
    f_i^pressure  := -SUM_j{ m_j.((p_i+p_j)/2rho_j) .del.W(r_i-r_j,h) }         NB Muller uses W_poly6kern for pressure and W_spiky for force due to pressure. 
                                                                                We use Wendland C^2 for pressure, viscosity and surface tension.
                                                                                
                                                                                NB Fluids-v3 used +ve atmospheric pressure and no surface tension.
                                                                                
    
Notes on possible meanings of the Del operator:

gradient of scalar,          grad_f := del.f
divergence vector field,     div_v  := del.v
curl vector field,           curl_v := del cross v
directoinal derivative     a.grad_f := a.(del.f)
Laplacian                    Delta  := del.del = del^2 
Hessian                      H      := del^2 = del.del^T
Tensor derivative                   := del circle_cross v
    
*/
extern "C" __device__ float3 contributeForce ( int i, float3 ipos, float3 iveleval, float ipress, float idens, int cell)
{			
	if ( fbuf.bufI(FGRIDCNT)[cell] == 0 ) return make_float3(0,0,0);                                        // If the cell is empty, skip it.
	float  dsq, sdist, c, r, sr=1.0;//fparam.psmoothradius;
    float3 pterm= make_float3(0,0,0), sterm= make_float3(0,0,0), vterm= make_float3(0,0,0), forcej= make_float3(0,0,0), delta_v= make_float3(0,0,0);                                                              // pressure, surface tension and viscosity terms.
	float3 dist     = make_float3(0,0,0),      eterm = make_float3(0,0,0),    force = make_float3(0,0,0);
	uint   j;
	int    clast    = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];                                // index of last particle in this cell
    
    for (int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++ ) {                                     // For particles in this cell.
		j           = fbuf.bufI(FGRID)[ cndx ];
		dist        = ( ipos - fbuf.bufF3(FPOS)[ j ] );                                                     // dist in cm (Rama's comment)
		dsq         = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);                                      // scalar distance squared
		r           = sqrt(dsq);
        
        // From https://github.com/DualSPHysics/DualSPHysics/wiki/3.-SPH-formulation#31-smoothing-kernel 
        /*
         * q=r/h, where r=dist between particles, h=smoothing length
         * 
         * W(r,h) = alpha_D(1-q/2)**4 *(2q+1) for 0<=q<=2
         * 
         * where alpha_D = 21/(16*Pi*h**3)  , the normalization kernel in 3D,
         * i.e. 1/integral_(0,2){kernel * area of a sphere}dr 
         * 
         */
        /* My new kernels
         * sr=1
         * # NB gives equi-pressure radius = 0.5.
         * 
         * Continuous Pressure, i.e. hyrostatic + surface tension or vapour pressure
         * CPkern = (sr - r )**3 - (1/2)*(sr - r )**2
         * 
         * Viscous kernel
         * vkern = (1/2)*(sr - r )**3
         */
        
        if ( dsq < 1 /*fparam.rd2*/ && dsq > 0) {                                                                 // IF in-range && not the same particle
            float kern = pow((sr - r),3);
            pterm = 1000.0* (dist/r) *(kern - (0.5)*pow((sr - r),2));       // 1000 = hydroststic stiffness      
            delta_v = fbuf.bufF3(FVEVAL)[j] - iveleval;
            vterm =  100000.0* delta_v * kern;// (1/2)*pow((sr - r),3) ;
            
            /*
             sdist   = sqrt(dsq * fparam.d2);                                                                // smoothing distance = sqrt(dist^2 * sim_scale^2))
             c       = ( fparam.psmoothradius - sdist );
             pterm   = (dist/sdist) * pow((fparam.psmoothradius - sqrt(dsq)), 3) * (fparam.psmoothradius - dsq) ;
             * fparam.psimscale * -0.5f * c * fparam.spikykern   * ( ipress + fbuf.bufF(FPRESS)[ j ] )/ sdist )  ;       // pressure term
            //sterm   = (dist/dsq) * fparam.sterm * cos(3*HIP_PI_F*r/(2*fparam.psmoothradius));  // can we use sdist in placeof r ?  or in place od dsq? What about pressure?
			//vterm   =  fparam.vterm * ( fbuf.bufF3(FVEVAL)[ j ] - iveleval );  // make_float3(0,0,0);//
			forcej  += ( pterm + sterm + vterm) * c * idens * (fbuf.bufF(FDENSITY)[ j ] );  // fluid force
            */
            force   +=  vterm + pterm ;
            /*if(i<10)  printf("\ncontribForce : i=%u, r=,%f, sr=,%f, (sr-r)^3=,%f, delta_v=,(%f,%f,%f), vterm=(%f,%f,%f), pterm(%f,%f,%f)  ",i, r, sr, kern, delta_v.x,delta_v.y,delta_v.z, vterm.x,vterm.y,vterm.z, pterm.x,pterm.y,pterm.z);*/
            /*
            if(i<10) printf("\ncontribForce() : i=,%u, ,cell=,%u,  ,cndx=,%u, ,r=,%f, ,sqrt(fparam.rd2)=r_basis=,%f, ,fparam.psmoothradius=,%f,,sdist=,%f, ,(fparam.psmoothradius-sdist)= c =,%f, \t,ipress=,%f, ,jpress=,%f, ,idens=,%f, ,jdens=,%f,       \t ,pterm=(,%f,%f,%f,),  ,sterm=(,%f,%f,%f,), ,vterm=(,%f,%f,%f,), ,forcej=(,%f,%f,%f,) ,  ,fparam.vterm=,%f, ,fbuf.bufF3(FVEVAL)[ j ]=(,%f,%f,%f,), ,iveleval=(,%f,%f,%f,) ", 
                i, cell, cndx, r, sqrt(fparam.rd2), fparam.psmoothradius, sdist, c,  ipress, fbuf.bufF(FPRESS)[j], idens, fbuf.bufF(FDENSITY)[j],    pterm.x,pterm.y,pterm.z, sterm.x,sterm.y,sterm.z, vterm.x,vterm.y,vterm.z, forcej.x,forcej.y,forcej.z, 
                fparam.vterm, fbuf.bufF3(FVEVAL)[j].x, fbuf.bufF3(FVEVAL)[j].y, fbuf.bufF3(FVEVAL)[j].z, iveleval.x, iveleval.y, iveleval.z
            );
            */
        }                                                                                                   // end of: IF in-range && not the same particle
    }                                                                                                       // end of loop round particles in this cell
    //if(i<10)  printf("\ncontribForce : i=%u, force=(%f,%f,%f)  ",i, force.x,force.y,force.z  );
    return force;                                                                                           // return fluid force && list of potential bonds fron this cell
}

extern "C" __global__ void computeForce ( int pnum, bool freeze, uint frame)
{			
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                         // particle index
	if ( i >= pnum ) return;
	uint gc = fbuf.bufI(FGCELL)[ i ];                                               // Get search cell	
	if ( gc == GRID_UNDEF ) return;                                                 // particle out-of-range

	gc -= (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	register float3 force, eterm, dist;                                             // request to compiler to store in a register for speed.
	force = make_float3(0,0,0);    eterm = make_float3(0,0,0);     dist  = make_float3(0,0,0);
    float dsq, abs_dist;                                                            // elastic force // new version computes here using particle index rather than ID.
    uint bondsToFill = 0;
    uint bonds[BONDS_PER_PARTICLE][2];                                              // [0] = index of other particle, [1] = bond_index
    float bond_dsq[BONDS_PER_PARTICLE];                                             // length of bond, for potential new bonds
    for (int a=0; a<BONDS_PER_PARTICLE;a++) {
        bonds[a][0]= UINT_MAX;
        bonds[a][1]= UINT_MAX;
        bond_dsq[a]= fparam.rd2;                                                    // NB if ( dsq < fparam.rd2 && dsq > 0) is the cut off for fluid interaction range
    } 
    
    //if(fbuf.bufI(FPARTICLE_ID)[i]<10) printf("\ncomputeForce() chk2: ParticleID=%u  ",fbuf.bufI(FPARTICLE_ID)[i] );  
    //__syncthreads();
    
    float3  pvel = {fbuf.bufF3(FVEVAL)[ i ].x,  fbuf.bufF3(FVEVAL)[ i ].y,  fbuf.bufF3(FVEVAL)[ i ].z}; // copy i's FEVAL to thread memory
    for (int a=0;a<BONDS_PER_PARTICLE;a++){                                         // compute elastic force due to bonds /////////////////////////////////////////////////////////
        uint bond                   = i*BOND_DATA + a*DATA_PER_BOND;                // bond's index within i's FELASTIDX 
        uint j                      = fbuf.bufI(FELASTIDX)[bond];                   // particle IDs   i*BOND_DATA + a
        float restlength        = fbuf.bufF(FELASTIDX)[bond + 2];                   // NB fbuf.bufF() for floats, fbuf.bufI for uints.
        if(j<pnum && restlength>0){                                                 // copy FELASTIDX to thread memory for particle i.
            float elastic_limit     = fbuf.bufF(FELASTIDX)[bond + 1];               // [0]current index, [1]elastic limit, [2]restlength, [3]modulus, [4]damping_coeff, [5]particle ID, [6]bond index 
            
            float modulus           = fbuf.bufF(FELASTIDX)[bond + 3];
            float damping_coeff     = fbuf.bufF(FELASTIDX)[bond + 4];
            uint  other_particle_ID = fbuf.bufI(FELASTIDX)[bond + 5];
            uint  bondIndex         = fbuf.bufI(FELASTIDX)[bond + 6];
            
            float3 j_pos = make_float3(fbuf.bufF3(FPOS)[ j ].x,  fbuf.bufF3(FPOS)[ j ].y,  fbuf.bufF3(FPOS)[ j ].z); // copy j's FPOS to thread memory
        
            dist            = ( fbuf.bufF3(FPOS)[ i ] - j_pos  );                   // dist in cm (Rama's comment)  /*fbuf.bufF3(FPOS)[ j ]*/
            dsq             = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);      // scalar distance squared
            abs_dist        = sqrt(dsq) + FLT_MIN;                                  // FLT_MIN adds minimum +ve float, to prevent division by abs_dist=zero
            float3 rel_vel  = fbuf.bufF3(FVEVAL)[ j ] - pvel;                       // add optimal damping:  -l*v , were v is relative velocity, and l= 2*sqrt(m*k)  
                                                                                    // where k is the spring stiffness.
                                                                                    // eterm = (bool within elastic limit) * (spring force + damping)
            float spring_strain = /* modulus * */ (abs_dist-restlength)/restlength; // NB this is now a strain accumulator, because stress is too large a number > FLT_MAX
            #define DECAY_FACTOR 0.8                                                                                   // could be a gene.
            fbuf.bufF(FELASTIDX)[bond + 7] = (fbuf.bufF(FELASTIDX)[bond + 7] + spring_strain) * DECAY_FACTOR;           // spring strain integrator
          
          //if(fbuf.bufI(FPARTICLE_ID)[i]<10) printf("\ncomputeForce() chk3: ParticleID=%u, bond=%u, restlength=%f, modulus=%f , abs_dist=%f , spring_strain=%f , strain_integrator=%f  ",fbuf.bufI(FPARTICLE_ID)[i], a, restlength , modulus , abs_dist , spring_strain , fbuf.bufF(FELASTIDX)[bond + 7]  );  
            
            eterm = ((float)(abs_dist < elastic_limit)) * ( ((dist/abs_dist) * spring_strain * modulus) - damping_coeff*rel_vel); // Elastic force due to bond ####
            
            //if(i<10) printf("\ncomputeForce() : i=,%u, bond=,%u, eterm=(,%f,%f,%f,) ",i, a, eterm.x,eterm.y,eterm.z);
            
            force -= eterm;                                                         // elastic force towards other particle, if (rest_len -abs_dist) is -ve
            atomicAdd( &fbuf.bufF3(FFORCE)[ j ].x, eterm.x);                        // NB Must send equal and opposite force to the other particle
            atomicAdd( &fbuf.bufF3(FFORCE)[ j ].y, eterm.y);
            atomicAdd( &fbuf.bufF3(FFORCE)[ j ].z, eterm.z);                        // temporary hack, ? better to write a float3 attomicAdd using atomicCAS  #########

            if (abs_dist >= elastic_limit){                                         // If (out going bond broken)
                fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +2]=0.0;         // remove broken bond by setting rest length to zero.
                //fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +3]=0;         // set modulus to zero
                
                uint bondIndex_ = fbuf.bufI(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +6];
                //fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bondIndex_+1] = UINT_MAX ;// set the reciprocal bond index to UINT_MAX, but leave the old particle ID for bond direction.
                //fbuf.bufI(FELASTIDX)[bond] = UINT_MAX;
                if (fparam.debug>2)printf("\n#### Set to broken, i=,%i, j=,%i, b=,%i, fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + b]=UINT_MAX\t####",i,j,bondIndex_);
                bondsToFill++;
            }
        }
        //__syncthreads();    // when is this needed ? ############
    }   

    //if (fparam.debug>2)printf("\nComputeForce chk4: i=%u, bondsToFill=%u,  gc=%u,  fparam.gridTotal=%u", i, bondsToFill, gc, fparam.gridTotal);  // was always zero . why ?
    //__syncthreads();
    
    //if(i<10) printf("\n computeForce()1: i=,%u, elastic force=(,%f,%f,%f,) ",i, force.x,force.y,force.z);
	
    bondsToFill=BONDS_PER_PARTICLE; // remove and use result from loop above ? ############
    for (int c=0; c < fparam.gridAdjCnt; c++) {                                 // Call contributeForce(..) for fluid forces AND potential new bonds /////////////////////////
        
        float3 fluid_force = make_float3(0,0,0);
        fluid_force = contributeForce ( i, fbuf.bufF3(FPOS)[ i ], fbuf.bufF3(FVEVAL)[ i ], fbuf.bufF(FPRESS)[ i ], fbuf.bufF(FDENSITY)[ i ], gc + fparam.gridAdj[c]); 
        //if (freeze==true) fluid_force *=0.1;                                        // slow fluid movement while forming bonds
        force += fluid_force;
    }
    //if(i<10) printf("\nComputeForce 2: i=,%u, force=(,%f,%f,%f,) ", i,force.x,force.y,force.z);
    //printf(".\n");
    //__syncthreads();
    
    
    //if (fparam.debug>2)printf("\ni=%u, bond_dsq=(%f,%f,%f,%f,%f,%f),",i,bond_dsq[0],bond_dsq[1],bond_dsq[2],bond_dsq[3],bond_dsq[4],bond_dsq[5]);

	//__syncthreads();   // when is this needed ? ############
    atomicAdd(&fbuf.bufF3(FFORCE)[ i ].x, force.x);                                 // atomicAdd req due to other particles contributing forces via incomming bonds. 
    atomicAdd(&fbuf.bufF3(FFORCE)[ i ].y, force.y);                                 // NB need to reset FFORCE to zero in  CountingSortFull(..)
    atomicAdd(&fbuf.bufF3(FFORCE)[ i ].z, force.z);                                 // temporary hack, ? better to write a float3 atomicAdd using atomicCAS ?  ########

}                                                                                   // end computeForce (..)

extern "C" __global__ void randomInit ( int seed, int numPnts )                                                                 // NB not currently used
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= numPnts ) return;

	// Initialize particle random generator	
	hiprandState_t* st = (hiprandState_t*) (fbuf.bufC(FSTATE) + i*sizeof(hiprandState_t));
	hiprand_init ( seed + i, 0, 0, st );		
}

#define CURANDMAX		2147483647

extern "C" __global__ void emitParticles ( float frame, int emit, int numPnts )                                                 // NB not currently used, may be a useful template
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= emit ) return;

	hiprandState_t* st = (hiprandState_t*) (fbuf.bufC(FSTATE) + i*sizeof(hiprandState_t));
	uint v = hiprand( st);
	uint j = v & (numPnts-1);
	float3 bmin = make_float3(-170,10,-20);
	float3 bmax = make_float3(-190,60, 20);

	float3 pos = make_float3(0,0,0);	
	pos.x = float( v & 0xFF ) / 256.0;
	pos.y = float((v>>8) & 0xFF ) / 256.0;
	pos.z = float((v>>16) & 0xFF ) / 256.0;
	pos = bmin + pos*(bmax-bmin);	
	
	fbuf.bufF3(FPOS)[j] = pos;
	fbuf.bufF3(FVEVAL)[j] = make_float3(0,0,0);
	fbuf.bufF3(FVEL)[j] = make_float3(5,-2,0);
	fbuf.bufF3(FFORCE)[j] = make_float3(0,0,0);	
}

__device__ uint getGridCell ( float3 pos, uint3& gc )                                                                           // NB not currently used
{	
	gc.x = (int)( (pos.x - fparam.gridMin.x) * fparam.gridDelta.x);			// Cell in which particle is located
	gc.y = (int)( (pos.y - fparam.gridMin.y) * fparam.gridDelta.y);
	gc.z = (int)( (pos.z - fparam.gridMin.z) * fparam.gridDelta.z);		
	return (int) ( (gc.y*fparam.gridRes.z + gc.z)*fparam.gridRes.x + gc.x);	
}

extern "C" __global__ void sampleParticles ( float* brick, uint3 res, float3 bmin, float3 bmax, int numPnts, float scalar )     // NB not currently used
{
	float3 dist;
	float dsq;
	int j, cell;	
	register float r2 = fparam.r2;
	register float h2 = 2.0*r2 / 8.0;		// 8.0=smoothing. higher values are sharper

	uint3 i = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;
	if ( i.x >= res.x || i.y >= res.y || i.z >= res.z ) return;
	
	float3 p = bmin + make_float3(float(i.x)/res.x, float(i.y)/res.y, float(i.z)/res.z) * (bmax-bmin);
	//float3 v = make_float3(0,0,0);
	float v = 0.0;

	// Get search cell
	int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	uint3 gc;
	uint gs = getGridCell ( p, gc );
	if ( gc.x < 1 || gc.x > fparam.gridRes.x-fparam.gridSrch || gc.y < 1 || gc.y > fparam.gridRes.y-fparam.gridSrch || gc.z < 1 || gc.z > fparam.gridRes.z-fparam.gridSrch ) {
		brick[ (i.y*int(res.z) + i.z)*int(res.x) + i.x ] = 0.0;
		return;
	}

	gs -= nadj;	

	for (int c=0; c < fparam.gridAdjCnt; c++) {
		cell = gs + fparam.gridAdj[c];		
		if ( fbuf.bufI(FGRIDCNT)[cell] != 0 ) {				
			for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell]; cndx++ ) {
				j = fbuf.bufI(FGRID)[cndx];
				dist = p - fbuf.bufF3(FPOS)[ j ];
				dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
				if ( dsq < fparam.rd2 && dsq > 0 ) {
					dsq = sqrt(dsq * fparam.d2);					
					//v += fbuf.mvel[j] * (fparam.gausskern * exp ( -(dsq*dsq)/h2 ) / fbuf.mdensity[ j ]);
					v += fparam.gausskern * exp ( -(dsq*dsq)/h2 );
				}
			}
		}
	}
	__syncthreads();

	brick[ (i.z*int(res.y) + i.y)*int(res.x) + i.x ] = v * scalar;
	//brick[ (i.z*int(res.y) + i.y)*int(res.x) + i.x ] = length(v) * scalar;
}

extern "C" __global__ void computeQuery ( int pnum )                                                                            // NB not currently used
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	// Get search cell
	int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	uint gc = fbuf.bufI(FGCELL) [i];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= nadj;

	// Sum Pressures
	float sum = 0.0;
	for (int c=0; c < fparam.gridAdjCnt; c++) {
		sum += 1.0;
	}
	__syncthreads();
	
}

extern "C" __global__ void advanceParticles ( float time, float dt, float ss, int numPnts )
{		
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index
	if ( i >= numPnts ) return;
	
	if ( fbuf.bufI(FGCELL)[i] == GRID_UNDEF ) {
		fbuf.bufF3(FPOS)[i] = make_float3(fparam.pboundmin.x,fparam.pboundmin.y,fparam.pboundmin.z-2*fparam.gridRes.z);
		fbuf.bufF3(FVEL)[i] = make_float3(0,0,0);
		return;
	}
			
	// Get particle vars
	register float3 accel, norm;
	register float diff, adj, speed;
	register float3 pos = fbuf.bufF3(FPOS)[i];
	register float3 veval = fbuf.bufF3(FVEVAL)[i];

	// Leapfrog integration						
	accel = fbuf.bufF3(FFORCE)[i];
	accel *= fparam.pmass;	
		
	// Boundaries
	// Y-axis
	diff = fparam.pradius - (pos.y - (fparam.pboundmin.y + (pos.x-fparam.pboundmin.x)*fparam.pground_slope )) * ss;
	if ( diff > EPSILON ) {
		norm = make_float3( -fparam.pground_slope, 1.0 - fparam.pground_slope, 0);
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	diff = fparam.pradius - ( fparam.pboundmax.y - pos.y )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3(0, -1, 0);
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	// X-axis
	//diff = fparam.pradius - (pos.x - (fparam.pboundmin.x + (sin(time*fparam.pforce_freq)+1)*0.5 * fparam.pforce_min))*ss;  //wave machine NB fparam.pforce_freq
	diff = fparam.pradius - (pos.x - fparam.pboundmin.x ) * ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 1, 0, 0);
		adj = (fparam.pforce_min+1) * fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	//diff = fparam.pradius - ( (fparam.pboundmax.x - (sin(time*fparam.pforce_freq)+1)*0.5*fparam.pforce_max) - pos.x)*ss;  //wave machine
	diff = fparam.pradius - ( fparam.pboundmax.x - pos.x )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3(-1, 0, 0);
		adj = (fparam.pforce_max+1) * fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	// Z-axis
	diff = fparam.pradius - (pos.z - fparam.pboundmin.z ) * ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 0, 0, 1 );
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	diff = fparam.pradius - ( fparam.pboundmax.z - pos.z )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 0, 0, -1 );
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	
	// Shield for particle store at fparam.pboundmax . ? does this exist implicitly due to the other boundaries ? 
	float3 dist = fparam.pboundmax - pos;
	diff = 2*fparam.pradius - (dist.x + dist.y + dist.z) * ss;                  // use Manhatan norm for speed & 2*pradius for safety
	if ( diff > EPSILON ) {
        norm = make_float3( 1, 1, 1 );                                          // NB planar norm for speed, not spherical
        adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
    }
	
		
	// Gravity
	accel += fparam.pgravity;
    
    // NB Accel & Vel limits prevent visible instability, but produce thoroughly non-physical behaviour.
    // For quasi-physical simulations we want to avoid triggering these limits. 
    
	// Accel Limit
	speed = accel.x*accel.x + accel.y*accel.y + accel.z*accel.z;
    if(i<10)printf("\nadvanceParticles()1: i=,%u,  mass=,%f,  accel=(,%f,%f,%f,),\t  accel^2=,%f,\t fparam.AL2=,%f,\t  fparam.pgravity=,(,%f,%f,%f,) ",
        i, fparam.pmass, accel.x,accel.y,accel.z, speed, fparam.AL2, fparam.pgravity.x, fparam.pgravity.y, fparam.pgravity.z
    );
	if ( speed > fparam.AL2 ) {
		accel *= fparam.AL / sqrt(speed);     // reduces accel to fparam.AL, while preserving direction. 
	}

	// Velocity Limit
	float3 vel = fbuf.bufF3(FVEL)[i];
    
	speed = vel.x*vel.x + vel.y*vel.y + vel.z*vel.z;
    if(i<10)printf("\nadvanceParticles()2: i=,%u, accel=(,%f,%f,%f,),  vel=(,%f,%f,%f,),  vel^2=,%f,  fparam.VL2=,%f, ",
        i, accel.x,accel.y,accel.z, vel.x,vel.y,vel.z,  speed, fparam.VL2
    );
	if ( speed > fparam.VL2 ) {
		speed = fparam.VL2;
		vel *= fparam.VL / sqrt(speed);       // reduces vel to fparam.VL , while preerving direction.
	}
	
	// Leap-frog Integration                                                    // Write to ftemp.buf*(FEVEL/FVEL/FPOS)
                                                                                // Allows specialParticles() to read old values.
	float3 vnext = accel*dt + vel;                                              // v(t+1/2) = v(t-1/2) + a(t) dt		
	ftemp.bufF3(FVEVAL)[i] = (vel + vnext) * 0.5;                               // v(t+1) = [v(t-1/2) + v(t+1/2)] * 0.5			
	ftemp.bufF3(FVEL)[i] = vnext;
	ftemp.bufF3(FPOS)[i] += vnext * (dt/ss);                                    // p(t+1) = p(t) + v(t+1/2) dt		
    
    
    if (i<10 ){  // fparam.debug>2 && i==0
        printf("\nadvanceParticles()3: i=,%u, accel.x==(,%f,%f,%f,),  vel=(,%f,%f,%f,),  dt==%f, vnext.x==(,%f,%f,%f,), ss==%f",
              i,  accel.x,accel.y,accel.z,  vel.x,vel.y,vel.z,    dt,   vnext.x,vnext.y,vnext.z,   ss
              );
/*
         * printf("\naccel.x==%f",accel.x);
        printf("\ndt==%f",dt);
        printf("\nvnext.x==%f",vnext.x);
        printf("\nss==%f",ss);
*/
    }
    
}


extern "C" __global__ void externalActuation (uint list_len,  float time, float dt, float ss, int numPnts )
{		
	uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index
	if ( particle_index >= list_len ) return;
    uint i = fbuf.bufII(FDENSE_LISTS)[12][particle_index];
	if ( i >= numPnts ) return;
  //if (fparam.debug>2)printf("\nexternalActuation(): i=%u\t",i);
    
    // Get particle vars
	register float3 accel;//, norm;
	register float speed; //diff, adj, 
	register float3 pos = fbuf.bufF3(FPOS)[i];
	register float3 veval = fbuf.bufF3(FVEVAL)[i];

	// Leapfrog integration						
	accel = fbuf.bufF3(FFORCE)[i];
	accel *= fparam.pmass;	

    // // Gravity
    // accel += fparam.pgravity;
    
    // External force   // How best to define this ?
    // For now, take sine wave on time.
    // Later, 
    // 1) take cmdln input for period & force vector
    // 2) model input
    // 3) simulation interaction (SOFA)
    
#define FACTOR  10
#define PERIOD  20
    accel +=  fparam.pgravity * FACTOR * sin(time/PERIOD) ;
    
    
	// Accel Limit
	speed = accel.x*accel.x + accel.y*accel.y + accel.z*accel.z;
	if ( speed > fparam.AL2 ) {
		accel *= fparam.AL / sqrt(speed);
	}

	// Velocity Limit
	float3 vel = fbuf.bufF3(FVEL)[i];
    
	speed = vel.x*vel.x + vel.y*vel.y + vel.z*vel.z;
	if ( speed > fparam.VL2 ) {
		speed = fparam.VL2;
		vel *= fparam.VL / sqrt(speed);
	}
	
	// Leap-frog Integration                                                    // Write to ftemp.buf*(FEVEL/FVEL/FPOS)
                                                                                // Allows specialParticles() to read old values.
	float3 vnext = accel*dt + vel;					// v(t+1/2) = v(t-1/2) + a(t) dt		
	ftemp.bufF3(FVEVAL)[i] = (vel + vnext) * 0.5;	// v(t+1) = [v(t-1/2) + v(t+1/2)] * 0.5			
	ftemp.bufF3(FVEL)[i] = vnext;
	ftemp.bufF3(FPOS)[i] += vnext * (dt/ss);		// p(t+1) = p(t) + v(t+1/2) dt		
    //if (fparam.debug>2)printf("\nexternalActuation(): i=%u (FVEL)[i]=(%f,%f,%f), (FPOS)[i]=(%f,%f,%f) ",
    //       i, ftemp.bufF3(FVEL)[i].x, ftemp.bufF3(FVEL)[i].y, ftemp.bufF3(FVEL)[i].z,
    //       ftemp.bufF3(FPOS)[i].x, ftemp.bufF3(FPOS)[i].y, ftemp.bufF3(FPOS)[i].z  
    //      );
}


extern "C" __global__ void fixedParticles (uint list_len, int numPnts )
{		
	uint particle_index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index
	if ( particle_index >= list_len ) return;
    uint i = fbuf.bufII(FDENSE_LISTS)[11][particle_index];
	if ( i >= numPnts ) return;
  //if (fparam.debug>2)printf("\nfixedParticles(): i=%u\t",i);

	ftemp.bufF3(FVEVAL)[i] = fbuf.bufF3(FVEVAL)[i];
	ftemp.bufF3(FVEL)[i]   = fbuf.bufF3(FVEL)[i];
	ftemp.bufF3(FPOS)[i]   = fbuf.bufF3(FPOS)[i];
}


/*
 * NB Mechanism of atomicCAS :
 * 
int  atomicCAS( int  *p,  intcmp,  intv )
{
	exclusive_single_thread
	{
		int  old = *p;
		if (cmp== old) *p = v;
	}
	return old;
}

*/
