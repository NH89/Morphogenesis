#include "hip/hip_runtime.h"
//----------------------------------------------------------------------------------
//
// FLUIDS v.3 - SPH Fluid Simulator for CPU and GPU
// Copyright (C) 2012-2013. Rama Hoetzlein, http://fluids3.com
//
// BSD 3-clause:
// Redistribution and use in source and binary forms, with or without modification, 
// are permitted provided that the following conditions are met:
// 1. Redistributions of source code must retain the above copyright notice, this 
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this 
//    list of conditions and the following disclaimer in the documentation and/or 
//    other materials provided with the distribution.
// 3. Neither the name of the copyright holder nor the names of its contributors may 
//    be used to endorse or promote products derived from this software without specific 
//   prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY 
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES 
// OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT 
// OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) 
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
// TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
// EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//----------------------------------------------------------------------------------

#define CUDA_KERNEL
#include "fluid_system_cuda.cuh"
#include <cfloat>
#include <cstdint>
#include "cutil_math.h"			// cutil32.lib
#include <string.h>
#include <assert.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__constant__ FParams		fparam;			// CPU Fluid params
__constant__ FBufs			fbuf;			// GPU Particle buffers (unsorted). An FBufs struct holds an array of pointers. 
__constant__ FBufs			ftemp;			// GPU Particle buffers (sorted)
__constant__ FGenome		fgenome;		// GPU Genome for particle automata behaviour. Also holds morphogen diffusability.
__constant__ uint			gridActive;

#define SCAN_BLOCKSIZE		512
//#define FLT_MIN  0.000000001                // set here as 2^(-30)
//#define UINT_MAX 65535

extern "C" __global__ void insertParticles ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index
	if ( i >= pnum ) return;
for (int a=0;a<BONDS_PER_PARTICLE;a++){                                          // The list of bonds from other particles 
            uint j = fbuf.bufI(FPARTICLEIDX) [i*BONDS_PER_PARTICLE*2 + a];       // NB j is valid only in ftemp.*
            uint k = ftemp.bufI(FPARTICLEIDX) [i*BONDS_PER_PARTICLE*2 + a];
//if(i<34)printf("\nAA(i=%i,a=%i,j=%u,k=%u)\t",i,a,j,k);
}
	//-- debugging (pointers should match CUdeviceptrs on host side)
	// printf ( " pos: %012llx, gcell: %012llx, gndx: %012llx, gridcnt: %012llx\n", fbuf.bufC(FPOS), fbuf.bufC(FGCELL), fbuf.bufC(FGNDX), fbuf.bufC(FGRIDCNT) );

	register float3 gridMin =	fparam.gridMin;                                  // "register" is a compiler 'hint', to keep this variable in thread register
	register float3 gridDelta = fparam.gridDelta;                                //  even if other variable have to be moved to slower 'local' memory  
	register int3 gridRes =		fparam.gridRes;                                  //  in the streaming multiprocessor's cache.
	register int3 gridScan =	fparam.gridScanMax;

	register int		gs;
	register float3		gcf;
	register int3		gc;	

	gcf = (fbuf.bufF3(FPOS)[i] - gridMin) * gridDelta; 
	gc = make_int3( int(gcf.x), int(gcf.y), int(gcf.z) );
	gs = (gc.y * gridRes.z + gc.z)*gridRes.x + gc.x;

	if ( gc.x >= 1 && gc.x <= gridScan.x && gc.y >= 1 && gc.y <= gridScan.y && gc.z >= 1 && gc.z <= gridScan.z ) {
		fbuf.bufI(FGCELL)[i] = gs;											     // Grid cell insert.
		fbuf.bufI(FGNDX)[i] = atomicAdd ( &fbuf.bufI(FGRIDCNT)[ gs ], 1 );		 // Grid counts.
		//gcf = (-make_float3(poff,poff,poff) + fbuf.bufF3(FPOS)[i] - gridMin) * gridDelta;
		//gc = make_int3( int(gcf.x), int(gcf.y), int(gcf.z) );
		//gs = ( gc.y * gridRes.z + gc.z)*gridRes.x + gc.x;
	} else {
		fbuf.bufI(FGCELL)[i] = GRID_UNDEF;		
	}
}

// Counting Sort - Full (deep copy)
extern "C" __global__ void countingSortFull ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;            // particle index
	if ( i >= pnum ) return;

	// Copy particle from original, unsorted buffer (msortbuf),
	// into sorted memory location on device (mpos/mvel)
	uint icell = ftemp.bufI(FGCELL) [ i ];                             // icell is bin into which i is sorted in fbuf.*

	if ( icell != GRID_UNDEF ) {	  
		// Determine the sort_ndx, location of the particle after sort		
        uint indx =  ftemp.bufI(FGNDX)  [ i ];                         // indx is off set within new cell
        int sort_ndx = fbuf.bufI(FGRIDOFF) [ icell ] + indx ;          // global_ndx = grid_cell_offet + particle_offset	
		float3 zero; zero.x=0;zero.y=0;zero.z=0;
		// Transfer data to sort location
		fbuf.bufI (FGRID) [ sort_ndx ] =	sort_ndx;                  // full sort, grid indexing becomes identity		
		fbuf.bufF3(FPOS) [sort_ndx] =		ftemp.bufF3(FPOS) [i];
		fbuf.bufF3(FVEL) [sort_ndx] =		ftemp.bufF3(FVEL) [i];
		fbuf.bufF3(FVEVAL)[sort_ndx] =		ftemp.bufF3(FVEVAL) [i];
		fbuf.bufF3(FFORCE)[sort_ndx] =      zero;                      // fbuf.bufF3(FFORCE)[ i ] += force; in contributeForce() requires value setting to 0 // old:	ftemp.bufF3(FFORCE) [i];  
		fbuf.bufF (FPRESS)[sort_ndx] =		ftemp.bufF(FPRESS) [i];
		fbuf.bufF (FDENSITY)[sort_ndx] =	ftemp.bufF(FDENSITY) [i];
		fbuf.bufI (FCLR) [sort_ndx] =		ftemp.bufI(FCLR) [i];
		fbuf.bufI (FGCELL) [sort_ndx] =		icell;
		fbuf.bufI (FGNDX) [sort_ndx] =		indx;
        float3 pos = ftemp.bufF3(FPOS) [i];
        // add extra data for morphogenesis
        // track the sort index of the other particle
        
        for (int a=0;a<BONDS_PER_PARTICLE;a++){         // [0]current index, [1]elastic limit, [2]restlength, [3]modulus, [4]damping coeff, [5]particle ID, [6]bond index 
            uint j = ftemp.bufI(FELASTIDX) [i*BOND_DATA + a*DATA_PER_BOND];             // NB j is valid only in ftemp.*
            uint j_sort_ndx = UINT_MAX;
            uint jcell = GRID_UNDEF;
            if (j<pnum){
                jcell = ftemp.bufI(FGCELL) [ j ];                                       // jcell is bin into which j is sorted in fbuf.*
                uint jndx = UINT_MAX;
                if ( jcell != GRID_UNDEF ) {                                            // avoid out of bounds array reads
                    jndx =  ftemp.bufI(FGNDX)  [ j ];      
                    if((fbuf.bufI(FGRIDOFF) [ jcell ] + jndx) <pnum){
                        j_sort_ndx = fbuf.bufI(FGRIDOFF) [ jcell ] + jndx ;             // new location in the list of the other particle
                    }
                }                                                                       // set modulus and length to zero if ( jcell != GRID_UNDEF ) 
            }
            fbuf.bufI (FELASTIDX) [sort_ndx*BOND_DATA + a*DATA_PER_BOND]  = j_sort_ndx; // NB if (j<pnum) j_sort_ndx = UINT_MAX; preserves non-bonds
            for (int b=1;b<DATA_PER_BOND;b++){                                          // copy [1]elastic limit, [2]restlength, [3]modulus, [4]damping coeff, iff unbroken
                fbuf.bufI (FELASTIDX) [sort_ndx*BOND_DATA + a*DATA_PER_BOND +b] = ftemp.bufI (FELASTIDX) [i*BOND_DATA + a*DATA_PER_BOND + b]  * ( jcell != GRID_UNDEF ) ; 
            }                                                                           // old: copy the modulus & length
        }
        
        for (int a=0;a<BONDS_PER_PARTICLE;a++){                                         // The list of bonds from other particles 
            uint k = ftemp.bufI(FPARTICLEIDX) [i*BONDS_PER_PARTICLE*2 + a*2];           // NB j is valid only in ftemp.*
            uint b = ftemp.bufI(FPARTICLEIDX) [i*BONDS_PER_PARTICLE*2 + a*2 +1];
            uint ksort_ndx = UINT_MAX; 
            uint kndx, kcell;
            if (k<pnum){                                                                //(k>=pnum) => bond broken // crashes when j=0 (as set in demo), after run().
                kcell = ftemp.bufI(FGCELL) [ k ];                                       // jcell is bin into which j is sorted in fbuf.*
                if ( kcell != GRID_UNDEF ) {
                    kndx =  ftemp.bufI(FGNDX)  [ k ];  
                    ksort_ndx = fbuf.bufI(FGRIDOFF) [ kcell ] + kndx ;            
                }
            }
            fbuf.bufI(FPARTICLEIDX) [sort_ndx*BONDS_PER_PARTICLE*2 + a*2] =  ksort_ndx; // ftemp.bufI(FPARTICLEIDX) [i*BONDS_PER_PARTICLE + a]
            fbuf.bufI(FPARTICLEIDX) [sort_ndx*BONDS_PER_PARTICLE*2 + a*2 +1] =  b;
            ftemp.bufI(FPARTICLEIDX) [i*BONDS_PER_PARTICLE*2 + a*2] = UINT_MAX;         // set ftemp copy for use as a lock when inserting new bonds in ComputeForce(..)
        }
        fbuf.bufI (FPARTICLE_ID) [sort_ndx] =	ftemp.bufI(FPARTICLE_ID) [i];
        fbuf.bufI (FMASS_RADIUS) [sort_ndx] =	ftemp.bufI(FMASS_RADIUS) [i];
        fbuf.bufI (FNERVEIDX)    [sort_ndx] =	ftemp.bufI(FNERVEIDX) [i];
        
        for (int a=0;a<NUM_TF;a++){fbuf.bufF (FCONC)   [sort_ndx * NUM_TF + a]      =	ftemp.bufF(FCONC) [i * NUM_TF + a]    ;}
        for (int a=0;a<NUM_TF;a++){fbuf.bufI (FEPIGEN) [sort_ndx * NUM_GENES + a]   =	ftemp.bufI(FEPIGEN) [i * NUM_GENES + a];}
	}
} 

extern "C" __device__ float contributePressure ( int i, float3 p, int cell )  
// pressure due to particles in 'cell'. NB for each particle there are 27 cells in which interacting particles might be.
{			
	if ( fbuf.bufI(FGRIDCNT)[cell] == 0 ) return 0.0;                       // If the cell is empty, skip it.

	float3 dist;
	float dsq, c, sum = 0.0;
	register float d2 = fparam.psimscale * fparam.psimscale;
	register float r2 = fparam.r2 / d2;
	
	int clast = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];      // off set of this cell in the list of particles,  PLUS  the count of particles in this cell.

	for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++ ) {    // For particles in this cell.
		int pndx = fbuf.bufI(FGRID) [cndx];                                       // index of this particle
		dist = p - fbuf.bufF3(FPOS) [pndx];                                       // float3 distance between this particle, and the particle for which the loop has been called.
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);                    // scalar distance squared
		if ( dsq < r2 && dsq > 0.0) {                                             // IF in-range && not the same particle.
			c = (r2 - dsq)*d2;                                                    //(NB this means all unused particles can be stored at one point)
			sum += c * c * c;
		}
	}
	return sum;                                                             // NB a scalar value for pressure contribution, at the current particle, due to particles in this cell.
}
			
extern "C" __global__ void computePressure ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                 // particle index
	if ( i >= pnum ) return;

	// Get search cell
	int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	uint gc = fbuf.bufI(FGCELL) [i];                                        // get grid cell of the current particle.
	if ( gc == GRID_UNDEF ) return;                                         // IF particle not in the simulation
	gc -= nadj;

	// Sum Pressures
	float3 pos = fbuf.bufF3(FPOS) [i];
	float sum = 0.0;
	for (int c=0; c < fparam.gridAdjCnt; c++) {                                    
		sum += contributePressure ( i, pos, gc + fparam.gridAdj[c] );
	}
	__syncthreads();
		
	// Compute Density & Pressure
	sum = sum * fparam.pmass * fparam.poly6kern;
	if ( sum == 0.0 ) sum = 1.0;
	fbuf.bufF(FPRESS)  [ i ] = ( sum - fparam.prest_dens ) * fparam.pintstiff;
	fbuf.bufF(FDENSITY)[ i ] = 1.0f / sum;
}

extern "C" __device__ float contributeDiffusion(int i, float3 p, int cell){
    // if the cell is empty, skip it
    if (fbuf.bufI(FGRIDCNT)[cell] == 0) return 0.0f;

    float3 dist;
    float dsq, c, sum = 0.0;
    register float d2 = fparam.psimscale * fparam.psimscale;
    register float r2 = fparam.r2 / d2;

    // process will be something like:
    // - look at neighbours around me, add their chemicals to this particle, and subtract some from myself as well
    // - return that

    // USE FCONC - should be float

    // add to neighbours, subtract from myself

    return 1.0f;
}


extern "C" __global__ void computeDiffusion(int pnum){
    // get particle index
    uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    // if the particle is outside the simulation, quit processing
    if (i >= pnum) return;

    // Get search cell
    // TODO - what does this block do?
    int nadj = (1 * fparam.gridRes.z + 1) * fparam.gridRes.x + 1;
    uint gc = fbuf.bufI(FGCELL) [i];
    if (gc == GRID_UNDEF) return;
    gc -= nadj;

    // Sum diffusion? (or in this case subtract it?)
    __syncthreads();

    // Compute diffusion?
}

extern "C" __device__ float3 contributeForce ( int i, float3 ipos, float3 iveleval, float ipress, float idens, int cell, uint _bondsToFill, uint _bonds[BONDS_PER_PARTICLE][2], float _bond_dsq[BONDS_PER_PARTICLE], bool freeze)
{			
	if ( fbuf.bufI(FGRIDCNT)[cell] == 0 ) return make_float3(0,0,0);                // If the cell is empty, skip it.
	float dsq, sdist, c, pterm;
	float3 dist = make_float3(0,0,0), eterm  = make_float3(0,0,0), force = make_float3(0,0,0);
	uint j;
	int clast = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];              // index of last particle in this cell
    for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++ ) {            // For particles in this cell.
		j = fbuf.bufI(FGRID)[ cndx ];
		dist = ( ipos - fbuf.bufF3(FPOS)[ j ] );                                    // dist in cm (Rama's comment)
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);                      // scalar distance squared
		if ( dsq < fparam.rd2 && dsq > 0) {                                         // IF in-range && not the same particle
            sdist = sqrt(dsq * fparam.d2);                                          // smoothing distance = sqrt(dist^2 * sim_scale^2))
			c = ( fparam.psmoothradius - sdist ); 
			pterm = fparam.psimscale * -0.5f * c * fparam.spikykern * ( ipress + fbuf.bufF(FPRESS)[ j ] ) / sdist;                       // pressure term
			force += ( pterm * dist + fparam.vterm * ( fbuf.bufF3(FVEVAL)[ j ] - iveleval )) * c * idens * (fbuf.bufF(FDENSITY)[ j ] );  // fluid force
            if (_bondsToFill >0 && dist.x+dist.y+dist.z > 0.0 && freeze==true){                             // collect particles, in the x+ve hemisphere, for potential bond formation 
                bool known = false;
                uint bond_index = UINT_MAX;

                for (int a=0; a<BONDS_PER_PARTICLE; a++){                                                   // chk if known, i.e. already bonded 
                    if (fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + a*2] == i        ) known = true;   // particle 'j' has a bond to particle 'i'
                    if (fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + a*2] == UINT_MAX ) bond_index = a; // patricle 'j' has an empty bond 'a' : picks last empty bond
                    //if (_bonds[a][0] == j )known = true;                                                  // particle 'i' already has a bond to particle 'j'  // not req, _bonds starts empty && only touch 'j' once
                }
                if (known == false && bond_index<UINT_MAX){       
                    //int bond_direction = 1*(dist.x-dist.y+dist.z>0.0) + 2*(dist.x+dist.y-dist.z>0.0);       // booleans divide bond space into quadrants of x>0.
                    float approx_zero = 0.02*fparam.rd2;
                    int bond_direction = ((dist.x+dist.y+dist.z)>0) * (1*(dist.x*dist.x>approx_zero) + 2*(dist.y*dist.y>approx_zero) + 4*(dist.z*dist.z>approx_zero)) -1; // booleans select +ve quadrant x,y,z axes and their planar diagonals
                    printf("\ni=%u, bond_direction=%i, dist=(%f,%f,%f), dsq=%f, approx_zero=%f", i, bond_direction, dist.x, dist.y, dist.z, dsq, approx_zero);
                    if(0<=bond_direction && bond_direction<BONDS_PER_PARTICLE && dsq<_bond_dsq[bond_direction]){ //if new candidate bond is shorter, for this quadrant. 
                                                                                                                //lacks a candidate bond _bonds[bond_direction][1]==0
                        _bonds[bond_direction][0] = j;                                                      // index of other particle
                        _bonds[bond_direction][1] = bond_index;                                             // FPARTICLEIDX vacancy index of other particle
                        _bond_dsq[bond_direction] = dsq;                                                    // scalar distance squared 
                    }
                }
            }                                                                                               // end of collect potential bonds
        }                                                                                                   // end of: IF in-range && not the same particle
    }                                                                                                       // end of loop round particles in this cell
    return force;                                                                                           // return fluid force && list of potential bonds fron this cell
}

extern "C" __global__ void computeForce ( int pnum, bool freeze, uint frame)
{			
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;                         // particle index
	if ( i >= pnum ) return;
	uint gc = fbuf.bufI(FGCELL)[ i ];                                               // Get search cell	
	if ( gc == GRID_UNDEF ) return;                                                 // particle out-of-range

	gc -= (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	register float3 force, eterm, dist;                                             // request to compiler to store in a register for speed.
	force = make_float3(0,0,0);    eterm = make_float3(0,0,0);     dist  = make_float3(0,0,0);
    float dsq, abs_dist;                                                            // elastic force // new version computes here using particle index rather than ID.
    uint bondsToFill = 0;
    uint bonds[BONDS_PER_PARTICLE][2];                                               // [0] = index of other particle, [1] = bond_index
    float bond_dsq[BONDS_PER_PARTICLE];                                             // length of bond, for potential new bonds
    for (int a=0; a<BONDS_PER_PARTICLE;a++) {
        bonds[a][0]= UINT_MAX;
        bonds[a][1]= UINT_MAX;
        bond_dsq[a]= fparam.rd2;                                                    // NB if ( dsq < fparam.rd2 && dsq > 0) is the cut off for fluid interaction range
    } 
    if (freeze==true){                                                              // If we are going to make new bonds, first check for broken incomming bonds //////////////////
        for (int a=0; a<BONDS_PER_PARTICLE;a++){                                    // loop round this particle's list of _incomming_ bonds /////
            bool intact = false;
            uint k = fbuf.bufI(FPARTICLEIDX)[i*BONDS_PER_PARTICLE*2 + a*2];
            uint b = fbuf.bufI(FPARTICLEIDX)[i*BONDS_PER_PARTICLE*2 + a*2 +1];      // chk bond intact. nb short circuit evaluation of if conditions.
            // k is a particle, bond_idx is in range, AND k's reciprocal record matches i's record of the bond
            if(k<pnum && b<BONDS_PER_PARTICLE && i==fbuf.bufI(FELASTIDX)[k*BOND_DATA + b*DATA_PER_BOND] && a==fbuf.bufI(FELASTIDX)[k*BOND_DATA + b*DATA_PER_BOND +6] )intact=true;   
            if(i==k)intact=false;
            //if(intact==true)printf("\ncomputeForce: incomming bond intact  i=%u, k=%u, a=%u, b=%u",i,k,a,b);
            if(intact==false){                                                      // remove broken/missing _incomming_ bonds
                fbuf.bufI(FPARTICLEIDX)[i*BONDS_PER_PARTICLE*2 + a*2] = UINT_MAX;   // particle
                fbuf.bufI(FPARTICLEIDX)[i*BONDS_PER_PARTICLE*2 + a*2 +1] = UINT_MAX;// bond index
            }
        }
        
        for (int a=0; a<BONDS_PER_PARTICLE;a++){                                    // loop round this particle's list of _outgoing_ bonds /////
            bool intact = false;
            uint j = fbuf.bufI(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND];
            uint bond_idx = fbuf.bufI(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND + 6];  // chk bond intact nb short circuit evaluation of if conditions.
            // j is a particle, bond_idx is in range, AND j's reciprocal record matches i's record of the bond
            if(j<pnum && bond_idx<BONDS_PER_PARTICLE && i==fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bond_idx*2] && a==fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bond_idx*2 +1])intact=true; 
            if(i==j)fbuf.bufI(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND]=false;
            if(intact==false){                                                      // remove missing _outgoing_ bonds 
                fbuf.bufI(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND]=UINT_MAX;         // [0]current index, [1]elastic limit, [2]restlength, [3]modulus, [4]damping_coeff, [5]particle ID, [6]bond index 
                fbuf.bufF(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND+1]=0.0;
                fbuf.bufF(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND+2]=1.0;
                fbuf.bufF(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND+3]=0.0;
                fbuf.bufF(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND+4]=0.0;
                fbuf.bufI(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND+5]=UINT_MAX;
                fbuf.bufI(FELASTIDX)[i*BOND_DATA+a*DATA_PER_BOND+6]=UINT_MAX;
            }
        }
    }
    float3  pvel = {fbuf.bufF3(FVEVAL)[ i ].x,  fbuf.bufF3(FVEVAL)[ i ].y,  fbuf.bufF3(FVEVAL)[ i ].z}; // copy i's FEVAL to thread memory
    for (int a=0;a<BONDS_PER_PARTICLE;a++){                                         // compute elastic force due to bonds /////////////////////////////////////////////////////////
        uint bond = i*BOND_DATA + a*DATA_PER_BOND;                                  // bond's index within i's FELASTIDX 
        uint j                      = fbuf.bufI(FELASTIDX)[bond];                   // particle IDs   i*BOND_DATA + a
        if(j<pnum){                                                                 // copy FELASTIDX to thread memory for particle i.
            float elastic_limit     = fbuf.bufF(FELASTIDX)[bond + 1];               // [0]current index, [1]elastic limit, [2]restlength, [3]modulus, [4]damping_coeff, [5]particle ID, [6]bond index 
            float restlength        = fbuf.bufF(FELASTIDX)[bond + 2];               // NB fbuf.bufF() for floats, fbuf.bufI for uints.
            float modulus           = fbuf.bufF(FELASTIDX)[bond + 3];
            float damping_coeff     = fbuf.bufF(FELASTIDX)[bond + 4];
            uint  other_particle_ID = fbuf.bufI(FELASTIDX)[bond + 5];
            uint  bondIndex         = fbuf.bufI(FELASTIDX)[bond + 6];
            
            float3 j_pos = make_float3(fbuf.bufF3(FPOS)[ j ].x,  fbuf.bufF3(FPOS)[ j ].y,  fbuf.bufF3(FPOS)[ j ].z); // copy j's FPOS to thread memory
        
            dist = ( fbuf.bufF3(FPOS)[ i ] - j_pos  );                              // dist in cm (Rama's comment)  /*fbuf.bufF3(FPOS)[ j ]*/
            dsq  = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);                 // scalar distance squared
            abs_dist = sqrt(dsq) + FLT_MIN;                                         // FLT_MIN adds minimum +ve float, to prevent division by abs_dist=zero
            float3 rel_vel = fbuf.bufF3(FVEVAL)[ j ] - pvel;                        // add optimal damping:  -l*v , were v is relative velocity, and l= 2*sqrt(m*k)  where k is the spring stiffness.
                                                                                    // eterm = (bool within elastic limit) * (spring force + damping)
                                                                                     
            eterm = ((float)(abs_dist < elastic_limit)) * ( ((dist/abs_dist) * modulus * (abs_dist-restlength)/restlength) - damping_coeff*rel_vel); // Elastic force due to bond ####
            force -= eterm;                                                         // elastic force towards other particle, if (rest_len -abs_dist) is -ve
            atomicAdd( &fbuf.bufF3(FFORCE)[ j ].x, eterm.x);                        // NB Must send equal and opposite force to the other particle
            atomicAdd( &fbuf.bufF3(FFORCE)[ j ].y, eterm.y);
            atomicAdd( &fbuf.bufF3(FFORCE)[ j ].z, eterm.z);                        // temporary hack, ? better to write a float3 attomicAdd using atomicCAS  #########

            if (abs_dist >= elastic_limit){                                         // If (out going bond broken)
                fbuf.bufI(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +1]=0;           // remove broken bond by setting elastic limit to zero.
                fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +3]=0;           // set modulus to zero
                
                uint bondIndex_ = fbuf.bufI(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +6];
                fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + bondIndex_] = UINT_MAX ;
                fbuf.bufI(FELASTIDX)[bond] = UINT_MAX;
                printf("\n#### Set to broken, i=%i, j=%i, b=%i, fbuf.bufI(FPARTICLEIDX)[j*BONDS_PER_PARTICLE*2 + b]=UINT_MAX\t####",i,j,bondIndex_);
                bondsToFill++;
            }
        }
        __syncthreads();    // when is this needed ? ############
    }   
//printf("\tComputeForce: i=%u, bondsToFill=%u", i, bondsToFill);  // was always zero . why ?
	bondsToFill=BONDS_PER_PARTICLE; // remove and use result from loop above ? ############
    for (int c=0; c < fparam.gridAdjCnt; c++) {                                 // Call contributeForce(..) for fluid forces AND potential new bonds /////////////////////////
        float3 fluid_force = make_float3(0,0,0);
        fluid_force = contributeForce ( i, fbuf.bufF3(FPOS)[ i ], fbuf.bufF3(FVEVAL)[ i ], fbuf.bufF(FPRESS)[ i ], fbuf.bufF(FDENSITY)[ i ], gc + fparam.gridAdj[c], bondsToFill, bonds ,bond_dsq, freeze); 
        if (freeze==true) fluid_force *=0.1;                                        // slow fluid movement while forming bonds
        force += fluid_force;
    }
    
    //printf("\ni=%u, bond_dsq=(%f,%f,%f,%f,%f,%f),",i,bond_dsq[0],bond_dsq[1],bond_dsq[2],bond_dsq[3],bond_dsq[4],bond_dsq[5]);

	__syncthreads();   // when is this needed ? ############
    atomicAdd(&fbuf.bufF3(FFORCE)[ i ].x, force.x);                                 // atomicAdd req due to other particles contributing forces via incomming bonds. 
    atomicAdd(&fbuf.bufF3(FFORCE)[ i ].y, force.y);                                 // NB need to reset FFORCE to zero in  CountingSortFull(..)
    atomicAdd(&fbuf.bufF3(FFORCE)[ i ].z, force.z);                                 // temporary hack, ? better to write a float3 attomicAdd using atomicCAS ?  ########

    // Add new bonds /////////////////////////////////////////////////////////////////////////////
    int a = BONDS_PER_PARTICLE * (int)(freeze!=true);                               // if (freeze!=true) skip for loop, else a=0
    for (; a< BONDS_PER_PARTICLE; a++){
        int otherParticleBondIndex = BONDS_PER_PARTICLE*2*bonds[a][0] + 2*a /*bonds[a][1]*/; // fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex]
        
        if((uint)bonds[a][0]==i) printf("\n (uint)bonds[a][0]==i, i=%u a=%u",i,a);  // float bonds[BONDS_PER_PARTICLE][3];  [0] = index of other particle, [1] = dsq, [2] = bond_index
                                                                                    // If outgoing bond empty && proposed bond for this quadrant is valid
        if( fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +1] == 0.0  &&  bonds[a][0] < pnum  && bonds[a][0]!=i  && bond_dsq[a]<3 ){  // ie dsq < 3D diagonal of cube ##### hack #####
                                                                                    // NB "bonds[b][0] = UINT_MAX" is used to indicate no candidate bond found
                                                                                    //    (FELASTIDX) [1]elastic limit = 0.0 isused to indicate out going bond is empty
            printf("\nBond making loop i=%u, a=%i, bonds[a][1]=%u, bond_dsq[a]=%f",i,a,bonds[a][1],bond_dsq[a]);
            
            do {} while( atomicCAS(&ftemp.bufI(FPARTICLEIDX)[otherParticleBondIndex], UINT_MAX, 0) );               // lock ///////////////// ###### //  if (not locked) write zero to 'ftemp' to lock.
            if (fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex]==UINT_MAX)  fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex] = i;                     //  if (bond is unoccupied) write to 'fbuf' to assign this bond
            ftemp.bufI(FPARTICLEIDX)[otherParticleBondIndex] = UINT_MAX;                                            // release lock ///////// ######

            
            if (fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex] == i){                                              // if (this bond is assigned) write bond data
                fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex +1] = a;                                             // write i's outgoing bond_index to j's incoming bonds
                uint i_ID = fbuf.bufI(FPARTICLE_ID)[i];                                                             // retrieve permenant particle IDs for 'i' and 'j'
                uint j_ID = fbuf.bufI(FPARTICLE_ID)[bonds[a][0]];
                float bond_length = sqrt(bond_dsq[a]);
                float modulus = 100000;       // 100 000 000                                                                    // 1000000 = min for soft matter integrity // 
                fbuf.bufI(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND]    = bonds[a][0];                               // [0]current index,
                fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +1] = 2 * bond_length ;                          // [1]elastic limit  = 2x restlength i.e. %100 strain
                fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +2] = 0.5*bond_length;                               // [2]restlength = initial length  
                fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +3] = modulus;                                   // [3]modulus
                fbuf.bufF(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +4] = 2*sqrt(fparam.pmass*modulus);              // [4]damping_coeff = optimal for mass-spring pair.
                fbuf.bufI(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +5] = j_ID;                                      // [5]save particle ID of the other particle NB for debugging
                fbuf.bufI(FELASTIDX)[i*BOND_DATA + a*DATA_PER_BOND +6] = bonds[a][1];                               // [6]bond index at the other particle 'j's incoming bonds
                printf("\nNew Bond a=%u, i=%u, j=%u, bonds[a][1]=%u, fromPID=%u, toPID=%u,, fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex]=%u, otherParticleBondIndex=%u",
                       a,i,bonds[a][0],bonds[a][1],i_ID,j_ID, fbuf.bufI(FPARTICLEIDX)[otherParticleBondIndex], otherParticleBondIndex);
            }            
        }// end if 
        __syncthreads();    // NB applies to all threads _if_ the for loop runs, i.e. if(freeze==true)
    }                                                                               // end loop around FELASTIDX bonds
}                                                                                   // end computeForce (..)

extern "C" __global__ void randomInit ( int seed, int numPnts )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= numPnts ) return;

	// Initialize particle random generator	
	hiprandState_t* st = (hiprandState_t*) (fbuf.bufC(FSTATE) + i*sizeof(hiprandState_t));
	hiprand_init ( seed + i, 0, 0, st );		
}

#define CURANDMAX		2147483647

extern "C" __global__ void emitParticles ( float frame, int emit, int numPnts )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= emit ) return;

	hiprandState_t* st = (hiprandState_t*) (fbuf.bufC(FSTATE) + i*sizeof(hiprandState_t));
	uint v = hiprand( st);
	uint j = v & (numPnts-1);
	float3 bmin = make_float3(-170,10,-20);
	float3 bmax = make_float3(-190,60, 20);

	float3 pos = make_float3(0,0,0);	
	pos.x = float( v & 0xFF ) / 256.0;
	pos.y = float((v>>8) & 0xFF ) / 256.0;
	pos.z = float((v>>16) & 0xFF ) / 256.0;
	pos = bmin + pos*(bmax-bmin);	
	
	fbuf.bufF3(FPOS)[j] = pos;
	fbuf.bufF3(FVEVAL)[j] = make_float3(0,0,0);
	fbuf.bufF3(FVEL)[j] = make_float3(5,-2,0);
	fbuf.bufF3(FFORCE)[j] = make_float3(0,0,0);	
	
}

__device__ uint getGridCell ( float3 pos, uint3& gc )
{	
	gc.x = (int)( (pos.x - fparam.gridMin.x) * fparam.gridDelta.x);			// Cell in which particle is located
	gc.y = (int)( (pos.y - fparam.gridMin.y) * fparam.gridDelta.y);
	gc.z = (int)( (pos.z - fparam.gridMin.z) * fparam.gridDelta.z);		
	return (int) ( (gc.y*fparam.gridRes.z + gc.z)*fparam.gridRes.x + gc.x);	
}

extern "C" __global__ void sampleParticles ( float* brick, uint3 res, float3 bmin, float3 bmax, int numPnts, float scalar )
{
	float3 dist;
	float dsq;
	int j, cell;	
	register float r2 = fparam.r2;
	register float h2 = 2.0*r2 / 8.0;		// 8.0=smoothing. higher values are sharper

	uint3 i = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;
	if ( i.x >= res.x || i.y >= res.y || i.z >= res.z ) return;
	
	float3 p = bmin + make_float3(float(i.x)/res.x, float(i.y)/res.y, float(i.z)/res.z) * (bmax-bmin);
	//float3 v = make_float3(0,0,0);
	float v = 0.0;

	// Get search cell
	int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	uint3 gc;
	uint gs = getGridCell ( p, gc );
	if ( gc.x < 1 || gc.x > fparam.gridRes.x-fparam.gridSrch || gc.y < 1 || gc.y > fparam.gridRes.y-fparam.gridSrch || gc.z < 1 || gc.z > fparam.gridRes.z-fparam.gridSrch ) {
		brick[ (i.y*int(res.z) + i.z)*int(res.x) + i.x ] = 0.0;
		return;
	}

	gs -= nadj;	

	for (int c=0; c < fparam.gridAdjCnt; c++) {
		cell = gs + fparam.gridAdj[c];		
		if ( fbuf.bufI(FGRIDCNT)[cell] != 0 ) {				
			for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell]; cndx++ ) {
				j = fbuf.bufI(FGRID)[cndx];
				dist = p - fbuf.bufF3(FPOS)[ j ];
				dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
				if ( dsq < fparam.rd2 && dsq > 0 ) {
					dsq = sqrt(dsq * fparam.d2);					
					//v += fbuf.mvel[j] * (fparam.gausskern * exp ( -(dsq*dsq)/h2 ) / fbuf.mdensity[ j ]);
					v += fparam.gausskern * exp ( -(dsq*dsq)/h2 );
				}
			}
		}
	}
	__syncthreads();

	brick[ (i.z*int(res.y) + i.y)*int(res.x) + i.x ] = v * scalar;
	//brick[ (i.z*int(res.y) + i.y)*int(res.x) + i.x ] = length(v) * scalar;
}

extern "C" __global__ void computeQuery ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	// Get search cell
	int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	uint gc = fbuf.bufI(FGCELL) [i];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= nadj;

	// Sum Pressures
	float sum = 0.0;
	for (int c=0; c < fparam.gridAdjCnt; c++) {
		sum += 1.0;
	}
	__syncthreads();
	
}

		
extern "C" __global__ void advanceParticles ( float time, float dt, float ss, int numPnts )
{		
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= numPnts ) return;
	
	if ( fbuf.bufI(FGCELL)[i] == GRID_UNDEF ) {
		fbuf.bufF3(FPOS)[i] = make_float3(fparam.pboundmin.x,fparam.pboundmin.y,fparam.pboundmin.z-2*fparam.gridRes.z);
		fbuf.bufF3(FVEL)[i] = make_float3(0,0,0);
		return;
	}
			
	// Get particle vars
	register float3 accel, norm;
	register float diff, adj, speed;
	register float3 pos = fbuf.bufF3(FPOS)[i];
	register float3 veval = fbuf.bufF3(FVEVAL)[i];

	// Leapfrog integration						
	accel = fbuf.bufF3(FFORCE)[i];
	accel *= fparam.pmass;	
		
	// Boundaries
	// Y-axis
	
	diff = fparam.pradius - (pos.y - (fparam.pboundmin.y + (pos.x-fparam.pboundmin.x)*fparam.pground_slope )) * ss;
	if ( diff > EPSILON ) {
		norm = make_float3( -fparam.pground_slope, 1.0 - fparam.pground_slope, 0);
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	diff = fparam.pradius - ( fparam.pboundmax.y - pos.y )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3(0, -1, 0);
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	// X-axis
	diff = fparam.pradius - (pos.x - (fparam.pboundmin.x + (sin(time*fparam.pforce_freq)+1)*0.5 * fparam.pforce_min))*ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 1, 0, 0);
		adj = (fparam.pforce_min+1) * fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	diff = fparam.pradius - ( (fparam.pboundmax.x - (sin(time*fparam.pforce_freq)+1)*0.5*fparam.pforce_max) - pos.x)*ss;
	if ( diff > EPSILON ) {
		norm = make_float3(-1, 0, 0);
		adj = (fparam.pforce_max+1) * fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	// Z-axis
	diff = fparam.pradius - (pos.z - fparam.pboundmin.z ) * ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 0, 0, 1 );
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	diff = fparam.pradius - ( fparam.pboundmax.z - pos.z )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 0, 0, -1 );
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
		
	// Gravity
	accel += fparam.pgravity;
//    printf(" accel+gravity=%f,%f,%f  gravity=%f,%f,%f\t",accel.x,accel.y,accel.z,fparam.pgravity.x,fparam.pgravity.y,fparam.pgravity.z);

	// Accel Limit
	speed = accel.x*accel.x + accel.y*accel.y + accel.z*accel.z;
	if ( speed > fparam.AL2 ) {
		accel *= fparam.AL / sqrt(speed);
	}

	// Velocity Limit
	float3 vel = fbuf.bufF3(FVEL)[i];
	speed = vel.x*vel.x + vel.y*vel.y + vel.z*vel.z;
	if ( speed > fparam.VL2 ) {
		speed = fparam.VL2;
		vel *= fparam.VL / sqrt(speed);
	}

	// Ocean colors
	/*uint clr = fbuf.bufI(FCLR)[i];
	if ( speed > fparam.VL2*0.2) {
		adj = fparam.VL2*0.2;		
		clr += ((  clr & 0xFF) < 0xFD ) ? +0x00000002 : 0;		// decrement R by one
		clr += (( (clr>>8) & 0xFF) < 0xFD ) ? +0x00000200 : 0;	// decrement G by one
		clr += (( (clr>>16) & 0xFF) < 0xFD ) ? +0x00020000 : 0;	// decrement G by one
		fbuf.bufI(FCLR)[i] = clr;
	}
	if ( speed < 0.03 ) {		
		int v = int(speed/.01)+1;
		clr += ((  clr & 0xFF) > 0x80 ) ? -0x00000001 * v : 0;		// decrement R by one
		clr += (( (clr>>8) & 0xFF) > 0x80 ) ? -0x00000100 * v : 0;	// decrement G by one
		fbuf.bufI(FCLR)[i] = clr;
	}*/
	
	//-- surface particle density 
	//fbuf.mclr[i] = fbuf.mclr[i] & 0x00FFFFFF;
	//if ( fbuf.mdensity[i] > 0.0014 ) fbuf.mclr[i] += 0xAA000000;

	// Leap-frog Integration
	float3 vnext = accel*dt + vel;					// v(t+1/2) = v(t-1/2) + a(t) dt		
	fbuf.bufF3(FVEVAL)[i] = (vel + vnext) * 0.5;	// v(t+1) = [v(t-1/2) + v(t+1/2)] * 0.5			
	fbuf.bufF3(FVEL)[i] = vnext;
	fbuf.bufF3(FPOS)[i] += vnext * (dt/ss);			// p(t+1) = p(t) + v(t+1/2) dt		
    
    
}


extern "C" __global__ void prefixFixup(uint *input, uint *aux, int len)     // merge *aux into *input  
{
	unsigned int t = threadIdx.x;
	unsigned int start = t + 2 * blockIdx.x * SCAN_BLOCKSIZE;
	if (start < len)					input[start] += aux[blockIdx.x];      
	if (start + SCAN_BLOCKSIZE < len)   input[start + SCAN_BLOCKSIZE] += aux[blockIdx.x];
}

extern "C" __global__ void prefixSum(uint* input, uint* output, uint* aux, int len, int zeroff) // sum *input, write to *output
{
	__shared__ uint scan_array[SCAN_BLOCKSIZE << 1];
	unsigned int t1 = threadIdx.x + 2 * blockIdx.x * SCAN_BLOCKSIZE;
	unsigned int t2 = t1 + SCAN_BLOCKSIZE;

	// Pre-load into shared memory
	scan_array[threadIdx.x] = (t1<len) ? input[t1] : 0.0f;
	scan_array[threadIdx.x + SCAN_BLOCKSIZE] = (t2<len) ? input[t2] : 0.0f;
	__syncthreads();

	// Reduction
	int stride;
	for (stride = 1; stride <= SCAN_BLOCKSIZE; stride <<= 1) {
		int index = (threadIdx.x + 1) * stride * 2 - 1;
		if (index < 2 * SCAN_BLOCKSIZE)
			scan_array[index] += scan_array[index - stride];
		__syncthreads();
	}

	// Post reduction
	for (stride = SCAN_BLOCKSIZE >> 1; stride > 0; stride >>= 1) {
		int index = (threadIdx.x + 1) * stride * 2 - 1;
		if (index + stride < 2 * SCAN_BLOCKSIZE)
			scan_array[index + stride] += scan_array[index];
		__syncthreads();
	}
	__syncthreads();

	// Output values & aux
	if (t1 + zeroff < len)	output[t1 + zeroff] = scan_array[threadIdx.x];
	if (t2 + zeroff < len)	output[t2 + zeroff] = (threadIdx.x == SCAN_BLOCKSIZE - 1 && zeroff) ? 0 : scan_array[threadIdx.x + SCAN_BLOCKSIZE];
	if (threadIdx.x == 0) {
		if (zeroff) output[0] = 0;
		if (aux) aux[blockIdx.x] = scan_array[2 * SCAN_BLOCKSIZE - 1];
	}
}

